#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <limits>
#include <algorithm>

#if defined(ENABLE_CUDA) && defined(ENABLE_MULTI_GPU) 
  #include <nccl.h>
#endif

#include "utils/utils.h"
#include "utils/logger.h"

#include "kernel_db/cuda_kernel_db.h"
#include "kernels/cuda_kmmkernel.h"

#ifdef ENABLE_CUDA
  //Defines ALL_CUDA_KERNELS array
  #include "kernels/cuda/kron-kernels/kernel_decl.inc"
#endif

/**
 * @AllCUDAKernels: An array of All CUDA kernels compiled.
*/
CUDAKMMKernel AllCUDAKernels[] = {
#ifdef ENABLE_CUDA
  ALL_CUDA_KERNELS
#endif
};


CUDAKernelDatabase::CUDAKernelDatabase() : isDistributed_(false) {
  streams.push_back(NULL);
  //Load all CUDA kernels and check if each kernel is valid
  loadKernels<CUDAKMMKernel>(AllCUDAKernels, sizeof(AllCUDAKernels)/sizeof(CUDAKMMKernel));
  for (uint i = 0; i < sizeof(AllCUDAKernels)/sizeof(CUDAKMMKernel); i++) {
    CUDAKMMKernel& info = AllCUDAKernels[i];
    if (!info.isValid()) abort();
    CUDA_CHECK(info.setSharedMemAttr());
  }
}

CUDAKernelDatabase::~CUDAKernelDatabase() {
  streams.clear();
  if (isDistributed_) {
#ifdef ENABLE_MULTI_GPU
    for (uint g = 0; g < gpusInM_; g++) {
      int s = pthread_barrier_destroy(&barriers_[g]);
      PTHREAD_BARRIER_CHECK(s);
    }

    delete threads_;
    delete barriers_;

    if (distComm_ == DistComm::NCCL) {
      for (int i=0; i<ncclComms.size(); i++)
        ncclCommDestroy((ncclComm_t)ncclComms[i]);
    }
#endif
  }
}

fastKronError CUDAKernelDatabase::init(void* ptrToStream, int gpus,
                                       int gpusInM, int gpusInK, int gpuKrons) {
  numGPUs_ = gpus;
  setCUDAStream(ptrToStream);
  if (numGPUs_ > numDevices()) {
    return fastKronInvalidArgument;
  }

  isDistributed_ = gpus > 1;

  for (int i = 0; i < numGPUs_; i++) {
    //Get information about each GPU
    auto detail = new CUDAArchDetails(i);
    hardware.push_back(detail);
    Logger(LogLevel::Info) << "Detected GPU " << i << std::endl << (*detail);
  }

  if (isDistributed_) {
#ifdef ENABLE_MULTI_GPU
    bool allP2PAccess = true;
    for (int g1 = 0; g1 < gpus; g1++) {
      for (int g2 = 0; g2 < gpus; g2++) {
        if (g1 == g2) continue;
        int p2pAccess = -1;
        CUDA_CHECK(hipDeviceCanAccessPeer(&p2pAccess, g1, g2));
        if (p2pAccess == 0) {allP2PAccess = false; break;}
        CUDA_CHECK(hipSetDevice(g1));
        CUDA_CHECK(hipDeviceEnablePeerAccess(g2, 0));
      }
      if (!allP2PAccess) break;
    }

    distComm_ = env::getDistComm();

    if (distComm_ == DistComm::P2P) {
      if (!allP2PAccess) {
        Logger(LogLevel::Debug) << "P2P Access among GPUs is available" << std::endl;
        distComm_ = DistComm::DistCommNone;
      }
    } else if (distComm_ == DistComm::NCCL) {
      int devs[gpus];
      distComm_ = DistComm::NCCL;
      ncclUniqueId ncclId;
      ncclGetUniqueId(&ncclId);
      Logger(LogLevel::Debug) << "Initializing NCCL"<<std::endl;
      for (int i = 0; i < gpus; i++) {
        CUDA_CHECK(hipSetDevice(i));
        ncclComms.push_back(nullptr);
        devs[i] = i;
      }
      NCCLCHECK(ncclCommInitAll((ncclComm_t*)&ncclComms[0], gpus, devs));
    }

    if (distComm_ == DistComm::DistCommNone) {
      if (allP2PAccess) {
        distComm_ = DistComm::P2P;
      } else {
        int devs[gpus];
        distComm_ = DistComm::NCCL;
        ncclUniqueId ncclId;
        ncclGetUniqueId(&ncclId);
        Logger(LogLevel::Debug) << "Initializing NCCL"<<std::endl;
        for (int i = 0; i < gpus; i++) {
          CUDA_CHECK(hipSetDevice(i));
          ncclComms.push_back(nullptr);
          devs[i] = i;
        }
        NCCLCHECK(ncclCommInitAll((ncclComm_t*)&ncclComms[0], gpus, devs));
      }
    }

    Logger(LogLevel::Info) << "Using " << distComm_ << 
                                  " for distributed communication" <<
                                  std::endl;

    if (gpusInK >= 1)
      gpusInK_ = gpusInK;
    else
      gpusInK_ = 2;//ilog2(gpus);
    
    if (gpusInM >= 1)
      gpusInM_ = gpusInM;  
    else
      gpusInM_ = 1;//ilog2(gpus);
      
    //TODO: Check that gpuKrons batch is valid, i.e., P1*P2..PBatch <= gpusInK
    if (gpuKrons > 0)
      perGPUKronBatch_ = gpuKrons;
    else 
      perGPUKronBatch_ = 1;

    //TODO: Check if gpusInK_ == 1 then perGPUKronBatch = NumKrons

    Logger(LogLevel::Debug) << "gpusInRows " << gpusInM_ <<
                 " gpusInCols " << gpusInK_ << 
                 " gpuKronBatch " << perGPUKronBatch_ <<
                 std::endl;
    if (gpusInK_ * gpusInM_ != numGPUs_)  {
      Logger(LogLevel::Info) << "gpusInCols * gpusInRows != total gpus (" << 
                   gpusInK_ * gpusInM_ << "!= " << 
                   numGPUs_<< ")" << std::endl;
      abort();
    }
    //TODO: Check that localKrons <= log (gpuK_)_P
    
    //All gpus with same row shares the same barrier
    //TODO: free
    barriers_ = new pthread_barrier_t[gpusInM_];
    threads_ = new thread_pool<ThreadArgs*>(numGPUs_);

    for (int i = 0; i < gpusInM_; i++) {
      int s = pthread_barrier_init(&barriers_[i], NULL, gpusInK_);
      PTHREAD_BARRIER_CHECK(s);
    }
#endif
  }

  return fastKronSuccess;
}

void CUDAKernelDatabase::setCUDAStream(void* ptrToStream) {
  streams.clear();
  hipStream_t* t = new hipStream_t;
  *t = 0;
  for (int i = 0; i < numGPUs_; i++) {
    if (ptrToStream != NULL) {
      hipStream_t* s = new hipStream_t;
      *s = *(((hipStream_t*)ptrToStream) + i);
	    streams.push_back((void*)s);
    }
    else
	    streams.push_back(t);
  }
}

int CUDAKernelDatabase::numDevices() {
  int devs;
  CUDA_CHECK(hipGetDeviceCount(&devs));
  return devs;
}

CUDAArchDetails CUDAKernelDatabase::getCUDADeviceProperties() {
  return *(dynamic_cast<CUDAArchDetails*>(hardware[0]));
}

fastKronError CUDAKernelDatabase::procMemset(uint32_t proc, Matrix& m, float val) {
  CUDA_CHECK(hipSetDevice(proc));
  float* host = new float[m.numel()];
  memset<float>(host, m.numel(), val);
  CUDA_CHECK(hipMemcpy(m.data(), host, m.numel()*sizeof(float),
                        hipMemcpyHostToDevice));
  delete host;
  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::procMalloc(uint32_t proc, size_t size, void*& ptr) {
  CUDA_CHECK(hipSetDevice(proc));
  CUDA_CHECK(hipMalloc(&ptr, size));
  CUDA_CHECK(hipMemset(ptr, 1, size));

  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::procFree(uint32_t proc, void* ptr) {
  CUDA_CHECK(hipSetDevice(proc));
  CUDA_CHECK(hipFree(ptr));
  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::initTune() {
  CUDA_CHECK(hipSetDevice(0));
  return fastKronSuccess;
}


/**
 * invoke() - Invoke a CUDA kernel.
 * @FusedFacs: The number of fusion in the CUDA kernel
 * @kernel: kernel to invoke.
 * @problem: KMMProblem to compute.
 * @fidx: Factor index in the KMMProblem.
 * @distParams: Parameters for Distributed 
 * @eplogueParams: Parameter for Epilogue (alpha, beta, and Y)
 * @execMode: Execution mode
 */
template<typename KMMProblemT, typename EpilogueParamsT>
fastKronError invoke(CUDAKMMKernel& kernelInfo, KMMProblemT problem,
                     const uint fidx, 
                     DistributedParams distParams,
                     EpilogueParamsT epilogueParams,
                     KernelMode execMode,
                     hipStream_t stream) {
  hipError_t status;

  KernelParams<KMMProblemT> params (problem, nullptr, 
                                  kernelInfo.getTileX(problem), 
                                  kernelInfo.getTileF(problem), 
                                  fidx, execMode);

  FusedParams<KMMProblemT> fusedParams (problem, kernelInfo.getMaxTileX().n());
  dim3 g = kernelInfo.grid(problem);
  std::cout <<fidx << "  " << g.x << " " << g.y << " " << g.z << std::endl;
  //TODO: Change this to kernelInfo.invoke
  typedef void (*KronMatmulKernelTy)(KernelParams<KMMProblemT>&, FusedParams<KMMProblemT>&, 
                                     DistributedParams&, EpilogueParams&, 
                                     dim3, dim3, uint32_t, hipStream_t);
  KronMatmulKernelTy(kernelInfo.kernelInvoker)(params, fusedParams, distParams, 
                                               epilogueParams, 
                                               kernelInfo.grid(problem),
                                               kernelInfo.block(),
                                               kernelInfo.getSharedMemSize(problem),
                                               stream);
  status = hipGetLastError();
  CUDA_CHECK(status);

  return fastKronSuccess;
}

template<typename KMMProblem, typename EpilogueParams>
fastKronError CUDAKernelDatabase::invokeKernel(KMMKernel* kernel,
                                               KMMProblem problem,
                                               const uint fidx,
                                               EpilogueParams epilogueParams,
                                               KernelMode execMode) {
  DistributedParams distParams;
  hipStream_t stream = *(hipStream_t*)streams[0];
  CUDAKMMKernel& cudaKernel = dynamic_cast<CUDAKMMKernel&>(*kernel);

  switch(problem.n()) {
    case 1:
      return invoke(cudaKernel, problem.template factorSlice<1>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 2:
      return invoke(cudaKernel, problem.template factorSlice<2>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 3:
      return invoke(cudaKernel, problem.template factorSlice<3>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 4:
      return invoke(cudaKernel, problem.template factorSlice<4>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 5:
      return invoke(cudaKernel, problem.template factorSlice<5>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 6:
      return invoke(cudaKernel, problem.template factorSlice<6>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    default:
      Logger(LogLevel::Debug) << "Invalid number of fused kernels: " << problem.n() << std::endl;
      return fastKronKernelNotFound;
  }
}


fastKronError CUDAKernelDatabase::invokeKernel(KMMKernel* kernel, KMMProblem problem,
                                    const uint fidx,
                                    EpilogueParams epilogueParams,
                                    KernelMode execMode) {
  return invokeKernel<KMMProblem, EpilogueParams>(kernel, problem, fidx, epilogueParams, execMode);
}

fastKronError CUDAKernelDatabase::invokeKernel(KMMKernel* kernel, KMMProblemStridedBatched problem,
                                    const uint fidx,
                                    EpilogueStridedBatchedParams epilogueParams,
                                    KernelMode execMode) {
  return invokeKernel<KMMProblemStridedBatched, EpilogueStridedBatchedParams>(kernel, problem, fidx, epilogueParams, execMode);
}

fastKronError CUDAKernelDatabase::invokeP2PStoreKernel(KMMKernel* kernel, 
                                                       KMMProblem problem,
                                                       const uint fidx,  
                                                       DistributedParams distParams, 
                                                       EpilogueParams epilogueParams,
                                                       KernelMode execMode) {
  hipStream_t stream = *(hipStream_t*)streams[distParams.proc()];
  CUDAKMMKernel& cudaKernel = dynamic_cast<CUDAKMMKernel&>(*kernel);

  switch (problem.n()) {
    case 1:
      return invoke(cudaKernel, problem.template factorSlice<1>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 2:
      return invoke(cudaKernel, problem.template factorSlice<2>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 3:
      return invoke(cudaKernel, problem.template factorSlice<3>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 4:
      return invoke(cudaKernel, problem.template factorSlice<4>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 5:
      return invoke(cudaKernel, problem.template factorSlice<5>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    case 6:
      return invoke(cudaKernel, problem.template factorSlice<6>(), fidx, distParams, epilogueParams,
                       execMode, stream);
    default:
      Logger(LogLevel::Debug) << "Invalid number of fused kernels: " << problem.n() << std::endl;
  }

  return fastKronKernelNotFound;
}

template<typename KMMProblemT, typename EpilogueParamsT>
fastKronError CUDAKernelDatabase::timeKernel(KMMKernel* kernel,
                                             KMMProblemT problem,
                                             const uint fidx, 
                                             DistributedParams distParams,
                                             EpilogueParamsT epilogueParams,
                                             KernelMode execMode, 
                                             bool useP2PStore,
                                             int warmups, int runs,
                                             float& runtime) {
#ifdef ENABLE_MULTI_GPU
//TODO: Also for FULL_TUNE
  if ((dynamic_cast<CUDAKMMKernel*>(kernel))->getLocalSize() > 0 || 
      (problem.f(0).q() >= 64 && kernel->getMaxTileF().q() <= 32)) {
    //skip probably slow kernels
    runtime = std::numeric_limits<float>::max();
    return fastKronSuccess;
  }
#endif

  hipStream_t stream = *(hipStream_t*)streams[0];
  CUDA_CHECK(hipStreamSynchronize(stream));
  hipEvent_t startEvent, endEvent;
  CUDA_CHECK(hipEventCreate(&startEvent));
  CUDA_CHECK(hipEventCreate(&endEvent));
  fastKronError status = fastKronSuccess;
  for (int r = 0; r < warmups + runs; r++) {
    if (r == warmups) CUDA_CHECK(hipEventRecord(startEvent, stream));
    if (useP2PStore) {
      status = invokeP2PStoreKernel(kernel, problem, fidx,
                                    distParams, epilogueParams, execMode);
    } else {
      status = invokeKernel(kernel, problem, fidx,
                            epilogueParams, execMode);
    }
  }
  CUDA_CHECK(hipEventRecord(endEvent, stream));
  CUDA_CHECK(hipEventSynchronize(endEvent));
  if (status != fastKronSuccess) {
    CUDA_CHECK(hipEventDestroy(startEvent));
    CUDA_CHECK(hipEventDestroy(endEvent));
    Logger(LogLevel::Info) << "Error in CUDA autotuning: "   <<
                              fastKronGetErrorString(status) <<
                              std::endl;
    return status;
  }
  CUDA_CHECK(hipEventElapsedTime(&runtime, startEvent, endEvent));
  runtime = runtime/runs;
  CUDA_CHECK(hipEventDestroy(startEvent));
  CUDA_CHECK(hipEventDestroy(endEvent));
  return status;
}

fastKronError CUDAKernelDatabase::timeKernel(KMMKernel* kernel, KMMProblem problem, 
                                                      const uint fidx, 
                                                      DistributedParams distParams,
                                                      EpilogueParams epilogueParams,
                                                      KernelMode execMode, 
                                                      bool useP2PStore,
                                                      int warmups, int runs,
                                                      float& runtime) {
  return timeKernel<KMMProblem, EpilogueParams>(kernel, problem, fidx, distParams, epilogueParams, execMode, 
                                                useP2PStore, warmups, runs, runtime);

}

fastKronError CUDAKernelDatabase::timeKernel(KMMKernel* kernel, KMMProblemStridedBatched problem, 
                                                     const uint fidx, 
                                                     DistributedParams distParams,
                                                     EpilogueStridedBatchedParams epilogueParams,
                                                     KernelMode execMode, 
                                                     bool useP2PStore,
                                                     int warmups, int runs,
                                                     float& runtime) {
  return timeKernel<KMMProblemStridedBatched, EpilogueStridedBatchedParams>(kernel, problem, fidx, distParams, epilogueParams, 
                                                                            execMode, useP2PStore, warmups, runs, runtime);
}

std::map<uint32_t, std::vector<KMMKernel*>, std::greater<int>>
  CUDAKernelDatabase::filterFastestFusedKernels(const KMMProblem& problem, 
                                                const std::vector<KMMKernel*>& kernels) {
  //TODO: 16 for Ampere and 8 for Volta
  uint32_t MinConsecutiveStoreElems = (getCUDADeviceProperties().smArch == SMArch::ampere) ? 16 : 8;

  //A fused kernel stores logP (TK) consecutive elements.
  //Remove all kernels that stores (< MinConsecutiveStoreElems).
  std::vector<KMMKernel*> validFusedKernels;
  
  {
    auto filter = [problem, MinConsecutiveStoreElems](KMMKernel* kernel) {
      const int PpowerN = (int)powf(problem.f(0).p(), kernel->getFusedFacs());
      const int consecutiveStoreElems = kernel->getMaxTileX().n()/PpowerN;
      return consecutiveStoreElems >= MinConsecutiveStoreElems;
    };

    std::copy_if(kernels.begin(), kernels.end(), std::back_inserter(validFusedKernels), filter);
  }

  return KernelDatabase::filterFastestFusedKernels(problem, validFusedKernels);
}

/**
 * blocksPerSM() - Returns blocks per SM occupied by a CUDA kernel based on occupancy
 */
static float blocksPerSM(const CUDAArchDetails gpu, CUDAKMMKernel* kernel, dim3 grid) {
  uint32_t regOcc = gpu.regsPerSM / (kernel->block().x * kernel->getNumRegs());
  uint32_t shmemOcc = gpu.sharedMemPerSM / kernel->getMaxSharedMemSize();
  return min(min(regOcc, shmemOcc), gpu.maxBlocksPerSM);
}

template<typename KMMProblemT>
KMMKernel* CUDAKernelDatabase::findKernelAtOptLevel(KMMProblemT subProblem, 
                                                    const std::vector<KMMKernel*>& kernelsForOptLevel) {
  if (kernelsForOptLevel.size() > 0) {
    //Find kernels that have either same P or same Q
    std::vector<KMMKernel*> kernelsWithSamePOrQ;
    std::copy_if(kernelsForOptLevel.begin(), kernelsForOptLevel.end(), 
                 std::back_inserter(kernelsWithSamePOrQ),
                 [subProblem](auto& kernel){return kernel->getMaxFactor().p() == subProblem.f(0).p() or 
                                            kernel->getMaxFactor().q() == subProblem.f(0).q();});
    std::vector<KMMKernel*> filteredKernels;
    if (kernelsWithSamePOrQ.size() > 0) {
      filteredKernels = kernelsWithSamePOrQ;
    } else {
      filteredKernels = kernelsForOptLevel;
    }
    //sort kernels in descending order based on the number of thread blocks a kernel invoke
    auto order = [subProblem, this](auto k1, auto k2) {
      return ((CUDAKMMKernel*)k1)->getNumBlocks(subProblem) > 
             ((CUDAKMMKernel*)k2)->getNumBlocks(subProblem);
    };
    std::sort(filteredKernels.begin(), filteredKernels.end(), order);
    for (auto k : filteredKernels) {
      uint blocksm = blocksPerSM(getCUDADeviceProperties(), (CUDAKMMKernel*)k, ((CUDAKMMKernel*)k)->grid(subProblem));
      if (((CUDAKMMKernel*)k)->getNumBlocks(subProblem) <= getCUDADeviceProperties().numSMs * blocksm) {
        return k;
      }
    }

    //If no kernel is found then return the kernel with max reuse
    return filteredKernels[filteredKernels.size() - 1];
  }

  return nullptr;
}

template<typename KMMProblemT>
std::string CUDAKernelDatabase::occupancyDetails(KMMKernel* kernelInfo, KMMProblemT problem) {
  CUDAKMMKernel* cudaKernel = dynamic_cast<CUDAKMMKernel*>(kernelInfo);
  std::stringstream ss;
  dim3 grid = cudaKernel->grid(problem);
  dim3 block = cudaKernel->block();
  std::string indent = "  ";

  ss << indent << "Grid          : {" << grid.x << ", " << grid.y << ", " << grid.z << "}" << std::endl
     << indent << "Block         : {" << block.x << ", " << block.y << ", " << block.z << "}" << std::endl
     << indent << "Shared Mem    : " << cudaKernel->getSharedMemSize(problem) << std::endl 
     << indent << "Reg per Thread: " << cudaKernel->getNumRegs() << std::endl
     << indent << "Blocks Per SM : " << blocksPerSM(getCUDADeviceProperties(), cudaKernel, cudaKernel->grid(problem)) << std::endl
     << indent << "Local Memory  : " << cudaKernel->getLocalSize() << std::endl;

  return ss.str();
}

std::string CUDAKernelDatabase::occupancyDetails(KMMKernel* kernelInfo, KMMProblem problem) {
  return occupancyDetails<KMMProblem>(kernelInfo, problem);
}

std::string CUDAKernelDatabase::occupancyDetails(KMMKernel* kernelInfo, KMMProblemStridedBatched problem) {
  return occupancyDetails<KMMProblemStridedBatched>(kernelInfo, problem);
}

CUDAArchDetails::CUDAArchDetails(int dev) {
  hipDeviceProp_t prop;

  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  numSMs             = prop.multiProcessorCount;
  maxBlocksPerSM     = prop.maxBlocksPerMultiProcessor;
  maxThreadsPerBlock = prop.maxThreadsPerBlock;
  maxThreadsPerSM    = prop.maxThreadsPerMultiProcessor;
  regsPerSM          = prop.regsPerMultiprocessor;
  maxRegsPerThread   = 256; 
  sharedMemPerSM     = prop.sharedMemPerMultiprocessor;
  sharedMemPerBlock  = prop.sharedMemPerBlock;
  name               = std::string(prop.name);
  computeMajor       = prop.major;
  computeMinor       = prop.minor;
  warpSize           = prop.warpSize;
  smArch             = computeCapabilityToSMArch(computeMajor, computeMinor);
}
