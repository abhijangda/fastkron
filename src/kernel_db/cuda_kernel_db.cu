#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <nccl.h>

#include "utils/utils.h"
#include "kernel_db/cuda_kernel_db.h"
#include "kernels/cuda_kernel_info.h"

#ifdef ENABLE_CUDA
  #include "kernels/cuda/kron-kernels/kernel_decl.inc"
#endif

CUDAKernel AllCUDAKernels[] = {
#ifdef ENABLE_CUDA
  ALL_CUDA_KERNELS
#endif
};

CUDAKernelDatabase::CUDAKernelDatabase() {
  streams.push_back(NULL);
  loadKernels<CUDAKernel>(AllCUDAKernels, sizeof(AllCUDAKernels)/sizeof(CUDAKernel));
  for (uint i = 0; i < sizeof(AllCUDAKernels)/sizeof(CUDAKernel); i++) {
    CUDAKernel& info = AllCUDAKernels[i];
    if (!info.isValid()) abort();
    CUDA_CHECK(info.setSharedMemAttr());
  }
  //TODO: Check that if distP2PStore is needed then there is a kernel that can 
  //do it
  //TODO: Add if debug
}

void CUDAKernelDatabase::free() {
  streams.clear();
  if (isDistributed_) {
    for (uint g = 0; g < gpusInM_; g++) {
      int s = pthread_barrier_destroy(&barriers_[g]);
      PTHREAD_BARRIER_CHECK(s);
    }

    delete threads_;
    delete barriers_;

    if (distComm_ == DistComm::NCCL) {
      for (int i=0; i<ncclComms.size(); i++)
        ncclCommDestroy((ncclComm_t)ncclComms[i]);
    }
  }
}

fastKronError CUDAKernelDatabase::initTune() {
  CUDA_CHECK(hipSetDevice(0));
  return fastKronSuccess;
}

//Launch cuda kernels
template<uint FusedFacs>
fastKronError invoke(CUDAKernel& kernelInfo, const uint kronIndex, 
                     KMMProblem problem,
                     DistributedParams distParams,
                     EpilogueParams epilogueParams,
                     KernelMode execMode,
                     hipStream_t stream) {
  hipError_t status;

  //Create the grid and thread block
  KernelParams<FusedFacs> params (problem, kernelInfo.getTileX(problem), 
                                  kernelInfo.getTileF(problem), 
                                  kronIndex, execMode);
  FusedParams<FusedFacs> fusedParams (problem, kernelInfo.tileX.n());

  std::cout << "72: " << kernelInfo.grid(problem).x << " " << kernelInfo.grid(problem).y << std::endl;
  std::cout << "73: " << kernelInfo.getTileX(problem) << std::endl;
  std::cout << "74: " << kernelInfo.getTileF(problem) << std::endl;
  std::cout << "75: " << kernelInfo.sharedMemSize(problem) << std::endl;
  //Call kernel
  typedef void (*KronMatmulKernelTy)(KernelParams<FusedFacs>, FusedParams<FusedFacs>, 
                                     DistributedParams, EpilogueParams, dim3, dim3, uint32_t, hipStream_t);
  KronMatmulKernelTy(kernelInfo.invokerFunc)(params, fusedParams, distParams, 
                                        epilogueParams, kernelInfo.grid(problem), 
                                        kernelInfo.block(), kernelInfo.sharedMemSize(problem), stream);
  status = hipGetLastError();
  CUDA_CHECK(status);

  if (false && kronIndex == 1) {
    printf("80\n");
    CUDA_CHECK(hipDeviceSynchronize());
    float* m = new float[problem.x().numel()];
    hipMemcpy(m, params.problem.y().data(), params.problem.y().numel() * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < problem.y().numel(); i++) {
      if (m[i] != 31) {
        printf("%f %d %d\n", m[i], i/(problem.y().n()), i%(problem.y().n()));
        break;
      }
    }
    exit(EXIT_SUCCESS);
  }

  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::invokeKernel(KernelInfo* kernel, const uint kronIndex, 
                                             KMMProblem problem, EpilogueParams epilogueParams,
                                             KernelMode execMode) {
  DistributedParams distParams;
  hipStream_t stream = *(hipStream_t*)streams[0];
  CUDAKernel& cudaKernel = dynamic_cast<CUDAKernel&>(*kernel);

  switch(problem.n()) {
    case 1:
      return invoke<1>(cudaKernel, kronIndex, problem,
                       distParams, epilogueParams, execMode, stream);
    case 2:
      return invoke<2>(cudaKernel, kronIndex, problem,
                       distParams, epilogueParams, execMode, stream);
    case 3:
      return invoke<3>(cudaKernel, kronIndex, problem,
                       distParams, epilogueParams, execMode, stream);
    case 4:
      return invoke<4>(cudaKernel, kronIndex, problem,
                       distParams, epilogueParams, execMode, stream);
    case 5:
      return invoke<5>(cudaKernel, kronIndex, problem,
                       distParams, epilogueParams, execMode, stream);
    case 6:
      return invoke<6>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    default:
      std::cout << "Invalid number of fused kernels" << std::endl;
      return fastKronKernelNotFound;
  }
}

fastKronError CUDAKernelDatabase::invokeP2PStoreKernel(KernelInfo* kernel, const uint kronIndex, 
                                                     KMMProblem problem, DistributedParams distParams, 
                                                     EpilogueParams epilogueParams,
                                                     KernelMode execMode) {
  hipStream_t stream = *(hipStream_t*)streams[distParams.proc()];
  CUDAKernel& cudaKernel = dynamic_cast<CUDAKernel&>(*kernel);

  switch (problem.n()) {
    case 1:
      return invoke<1>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    case 2:
      return invoke<2>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    case 3:
      return invoke<3>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    case 4:
      return invoke<4>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    case 5:
      return invoke<5>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    case 6:
      return invoke<6>(cudaKernel, kronIndex, problem, 
                       distParams, epilogueParams, execMode, stream);
    default:
      std::cout << "Invalid number of fused kernels" << std::endl;
  }

  return fastKronKernelNotFound;
}

fastKronError CUDAKernelDatabase::timeKernel(KernelInfo* kernel, const uint factorIdx, 
                                           KMMProblem problem, DistributedParams distParams, 
                                           EpilogueParams epilogueParams,
                                           KernelMode execMode, 
                                           bool distP2PStore,
                                           int warmups, int runs,
                                           float& runtime) {
  hipStream_t stream = *(hipStream_t*)streams[0];
  CUDA_CHECK(hipStreamSynchronize(stream));
  hipEvent_t startEvent, endEvent;
  CUDA_CHECK(hipEventCreate(&startEvent));
  CUDA_CHECK(hipEventCreate(&endEvent));
  fastKronError status;
  for (int r = 0; r < warmups + runs; r++) {
    if (r == warmups) CUDA_CHECK(hipEventRecord(startEvent, stream));
    if (distP2PStore) {
      status = invokeP2PStoreKernel(kernel, factorIdx, problem,
                                    distParams, epilogueParams, execMode);
    } else {
      status = invokeKernel(kernel, factorIdx, problem,
                            epilogueParams, execMode);
    }
  }
  
  CUDA_CHECK(hipEventRecord(endEvent, stream));
  CUDA_CHECK(hipEventSynchronize(endEvent));
  if (status != fastKronSuccess) {
    CUDA_CHECK(hipEventDestroy(startEvent));
    CUDA_CHECK(hipEventDestroy(endEvent));
    std::cout << "Error: " << fastKronGetErrorString(status) << std::endl;
    return status;
  }
  CUDA_CHECK(hipEventElapsedTime(&runtime, startEvent, endEvent));
  runtime = runtime/runs;
  CUDA_CHECK(hipEventDestroy(startEvent));
  CUDA_CHECK(hipEventDestroy(endEvent));
  return status;
}

fastKronError CUDAKernelDatabase::procMalloc(uint32_t proc, size_t size, void*& ptr) {
  CUDA_CHECK(hipSetDevice(proc));
  CUDA_CHECK(hipMalloc(&ptr, size));
  CUDA_CHECK(hipMemset(ptr, 1, size));
  
  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::procFree(uint32_t proc, void* ptr) {
  CUDA_CHECK(hipSetDevice(proc));
  CUDA_CHECK(hipFree(ptr));
  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::procMemset(uint32_t proc, Matrix& m, float val) {
  //TODO: call a CUDA kernel for memset
  CUDA_CHECK(hipSetDevice(proc));
  float* host = new float[m.numel()];
  memset<float>(host, m.numel(), val);
  CUDA_CHECK(hipMemcpy(m.data(), host, m.numel()*sizeof(float), hipMemcpyHostToDevice));
  delete host;
  return fastKronSuccess;
}

fastKronError CUDAKernelDatabase::init(void* ptrToStream, int gpus, int gpusInM, int gpusInK, int gpuKrons) {
  streams.clear();
  hipStream_t* t = new hipStream_t;
  *t = 0;
  for (int i = 0; i < gpus; i++) {
    if (ptrToStream != NULL)
	  streams.push_back(((hipStream_t*)ptrToStream) + i);
    else
	    streams.push_back(t);
  }
  numGPUs_ = gpus;
  isDistributed_ = gpus > 1;
  if (isDistributed_) {
    bool allP2PAccess = true;
    for (int g1 = 0; g1 < gpus; g1++) {
      for (int g2 = 0; g2 < gpus; g2++) {
        if (g1 == g2) continue;
        int p2pAccess = -1;
        CUDA_CHECK(hipDeviceCanAccessPeer(&p2pAccess, g1, g2));
        if (p2pAccess == 0) {allP2PAccess = false; break;}
        CUDA_CHECK(hipSetDevice(g1));
        CUDA_CHECK(hipDeviceEnablePeerAccess(g2, 0));
      }
      if (!allP2PAccess) break;
    }

    distComm_ = env::getDistComm();

    if (distComm_ == DistComm::P2P) {
      if (!allP2PAccess) {
        std::cout << "P2P Access among GPUs not available using NCCL" << std::endl;
        distComm_ = DistComm::DistCommNone;
      }
    } else if (distComm_ == DistComm::NCCL) {
      int devs[gpus];
      distComm_ = DistComm::NCCL;
      ncclUniqueId ncclId;
      ncclGetUniqueId(&ncclId);
      std::cout << "Initializing NCCL"<<std::endl;
      for (int i = 0; i < gpus; i++) {
        CUDA_CHECK(hipSetDevice(i));
        ncclComms.push_back(nullptr);
        devs[i] = i;
      }
      NCCLCHECK(ncclCommInitAll((ncclComm_t*)&ncclComms[0], gpus, devs));
    }

    if (distComm_ == DistComm::DistCommNone) {
      if (allP2PAccess) {
        distComm_ = DistComm::P2P;
      } else {
        int devs[gpus];
        distComm_ = DistComm::NCCL;
        ncclUniqueId ncclId;
        ncclGetUniqueId(&ncclId);
        std::cout << "Initializing NCCL"<<std::endl;
        for (int i = 0; i < gpus; i++) {
          CUDA_CHECK(hipSetDevice(i));
          ncclComms.push_back(nullptr);
          devs[i] = i;
        }
        NCCLCHECK(ncclCommInitAll((ncclComm_t*)&ncclComms[0], gpus, devs));
      }
    }

    std::cout << "Using " << distComm_ << " for distributed comm" << std::endl;

    if (gpusInK >= 1)
      gpusInK_ = gpusInK;
    else
      gpusInK_ = 2;//ilog2(gpus);
    
    if (gpusInM >= 1)
      gpusInM_ = gpusInM;  
    else
      gpusInM_ = 1;//ilog2(gpus);
      
    //TODO: Check that gpuKrons batch is valid, i.e., P1*P2..PBatch <= gpusInK
    if (gpuKrons > 0)
      perGPUKronBatch_ = gpuKrons;
    else 
      perGPUKronBatch_ = 1;

    //TODO: Check if gpusInK_ == 1 then perGPUKronBatch = NumKrons

    std::cout << "gpusInRows " << gpusInM_ <<
                 " gpusInCols " << gpusInK_ << 
                 " gpuKronBatch " << perGPUKronBatch_ <<
                 std::endl;
    if (gpusInK_ * gpusInM_ != numGPUs_)  {
      std::cout << "gpusInCols * gpusInRows != total gpus (" << 
                   gpusInK_ * gpusInM_ << "!= " << 
                   numGPUs_<< ")" << std::endl;
      abort();
    }
    //TODO: Check that localKrons <= log (gpuK_)_P
    
    //All gpus with same row shares the same barrier
    //TODO: free
    barriers_ = new pthread_barrier_t[gpusInM_];
    threads_ = new thread_pool<ThreadArgs*>(numGPUs_);

    for (int i = 0; i < gpusInM_; i++) {
      int s = pthread_barrier_init(&barriers_[i], NULL, gpusInK_);
      PTHREAD_BARRIER_CHECK(s);
    }
  }

  return fastKronSuccess;
}
