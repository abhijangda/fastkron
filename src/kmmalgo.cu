#include <stdio.h>

#include "kmmalgo.h"

hipError_t executeGeKMM(const KMMProblem problem, void* temp1,
                         void* temp2, 
                         std::function<uint (const KMMProblem, void*, void*, hipError_t&)> func) {
  uint k = problem.k;
  size_t l = k;
  int nextF = 1;
  for (int i = problem.shape.n - 1; i >= 0; i = i - nextF) {
    l = (k/problem.shape.ps[i])*problem.shape.qs[i];
    printf("nextF %d i %d\n", nextF, i);
    KMMProblem subProblem(problem, i, i+1, k, l);

    hipError_t e = hipSuccess;
    nextF = func(subProblem, temp1, temp2, e);
    
    if (e != hipSuccess) return e;
    k = l;
  }

  return hipSuccess;
}