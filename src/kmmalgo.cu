#include <stdio.h>

#include "kmmalgo.h"

hipError_t executeGeKMM(const KMMProblem problem, void* temps[2],
                         void* result,
                         std::function<uint (const KMMProblem)> next,
                         std::function<hipError_t (const KMMProblem, void*[2], void*)> func) {
  uint k = problem.k;
  size_t l = k;
  int nextF = 1;
  GeKMMPtrs ptrs = problem.ptrs;
  hipError_t err;

  for (int i = problem.shape.n - 1; i >= 0; i = i - nextF) {
    nextF = next(problem);
    for (int f = i; f > i - nextF; f--) {
      l = (l/problem.shape.ps[f])*problem.shape.qs[f];
    }
    uint qs[problem.shape.n];
    uint ps[problem.shape.n];
    void* fs[problem.shape.n];
    if(i < nextF) {
      ptrs = GeKMMPtrs(ptrs.x, ptrs.fs, result);
    }
    auto subProblem = problem.rsub(ptrs, ps, qs, fs, i, nextF, k, l);
    err = func(subProblem, temps, result);
    if (err != hipSuccess) break;
    k = l;
    if (temps != nullptr)
      ptrs = ptrs.swap(temps[0], temps[1]);
  }

  return hipSuccess;
}