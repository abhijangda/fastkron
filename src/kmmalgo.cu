#include <stdio.h>

#include "kmmalgo.h"

hipError_t executeGeKMM(const KMMProblem problem, void* temp1,
                         void* temp2, 
                         std::function<uint (const KMMProblem, void*, void*)> func) {
  uint k = problem.k;
  size_t l = k;
  int nextF = 1;
  for (int i = problem.shape.n - 1; i >= 0; i = i - nextF) {
    l = (k/problem.shape.ps[i])*problem.shape.qs[i];
    KMMProblem subProblem(problem, i, i+1, k, l);
    nextF = func(subProblem, temp1, temp2);
    k = l;
  }

  return hipSuccess;
}