#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <type_traits>

#include "kron.h"

#define CUDA_CHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define MIN(x,y) (((x) < (y)) ? (x) : (y))
#define MAX(x,y) (((x) > (y)) ? (x) : (y))
#define DIVUP(x, y) (((x) + (y) - 1)/((y)))

#define C_IN_REG
#define EVAL

//utils.h
static constexpr int log2(uint n) {return 31 - __builtin_clz(n);}
static constexpr int log2(int n) {return 31 - __builtin_clz(n);}

struct KernelInfo {
  void* kernel;
  uint NumThreads;
  uint KronCols;
  uint KronRows;
  uint KP_N_TILE_;
  uint MaxColsA;
  uint CRegRows;
  uint CRegCols;
};

enum RowParallelismTy {
  Low = 0,
  Medium,
  High,
  Num = 3,
};


#define N_THREADS 256

#include "kernel_decl.inc" 

#define TYPE_KERNELS(T, VecT) \
  KERNEL_DECL(T, VecT, 0, 0),\
  KERNEL_DECL(T, VecT, 1, 0),\
  KERNEL_DECL(T, VecT, 0, 1),\
  KERNEL_DECL(T, VecT, 1, 1),


//Three type kernels float/float4, int/int4, and double/double4
#define NUM_TYPE_KERNELS 2
// #define MIN_K 16
// #define MAX_K 4096
#define NUM_MAX_K_KERNELS (log2(MAX_K)-log2(MIN_K) + 1)

// #define MIN_KP_K 2
// #define MAX_KP_K 64
#define NUM_KP_N_K_KERNELS (log2(MAX_KP_K)-log2(MIN_KP_K) + 1)

#define NUM_K_EQUALS_VAR 2
#define NUM_KPK_EQUALS_VAR 1
#define NUM_ROWS_MOD_TILE_IS_ZERO 2
#define EXTERNAL_KP_K_TILE_ MAX_K

#include "kron_device.cu"

static KernelInfo KronGemmKernels[NUM_TYPE_KERNELS][RowParallelismTy::Num][NUM_K_EQUALS_VAR][NUM_ROWS_MOD_TILE_IS_ZERO][NUM_MAX_K_KERNELS][NUM_KP_N_K_KERNELS][NUM_KPK_EQUALS_VAR] = {
  // KP_N_K_KERNELS(8, 1024, 32)
  TYPE_KERNELS(float,  float4)
  TYPE_KERNELS(int,    int4)
  // TYPE_KERNELS(double, double4)
    // COARSE_TB_KERNELS(1)
    // COARSE_TB_KERNELS(2)
    // COARSE_TB_KERNELS(4)
  };

static_assert(sizeof(KronGemmKernels)/sizeof(KernelInfo) == NUM_TYPE_KERNELS * RowParallelismTy::Num * NUM_ROWS_MOD_TILE_IS_ZERO * NUM_KP_N_K_KERNELS * NUM_MAX_K_KERNELS*NUM_K_EQUALS_VAR*NUM_KPK_EQUALS_VAR);

template<typename T>
static int typeKernelIndex(T x) {
  if (std::is_same<T, float>::value)
    return 0;
  if (std::is_same<T, int>::value)
    return 1;
  if (std::is_same<T, double>::value)
    return 2;
}

/**Library entry points to launch cuda kernels**/

//Check N and K is a multiplication of KronMatCols and KronMatRows
static bool checkKronMatrixSizes(const uint NumKronMats, 
                                 const uint M, const uint N, const uint K, 
                                 const uint KronMatCols[], const uint KronMatRows[]) {
  uint n=1,k=1;
  for (uint i = 0; i < NumKronMats; i++) {
    k *= KronMatRows[i];
    n *= KronMatCols[i];
  }
  if (n != N || k != K) {
    printf("Invalid Kron product sizes %d != %d, %d != %d\n", n, N, k, K);
    return false;
  }

  return true;
}

//Launch cuda kernels
template<typename T, typename VecT>
hipError_t generalKronGemm(const uint NumKronMats, 
                            T* kronGemmResults[], T* x, T* kronMats[], T** kronGemmResult,
                            const uint M, const uint N, const uint K, 
                            const uint KronMatCols[], const uint KronMatRows[], 
                            hipStream_t stream) {
  typedef int (*KronGemmKernel)(const uint, const uint, const uint, const uint, const uint, T*, T*, T*);
  hipError_t status;

  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return hipErrorInvalidValue;

  //Only row major layout of all matrics is supported.
  
  //Use double buffering for writing result and using output 
  //of previous iteration as input to current
  
  *kronGemmResult = kronGemmResults[0];
  T* prevResult = x;
  RowParallelismTy rowParallelism = RowParallelismTy::Low;
  for (uint i = 0; i < NumKronMats; i++) {
    KronGemmKernel cuda_gemm_func = NULL;
    dim3 grid;
    dim3 block;
    const uint kronMat = NumKronMats-i-1;

    const int KP_K_BATCH = 1;
    int N_COARSE_TB = 1; //(M > 100) ? 2 : 1;
    int max_k;
    int min_k;
    int max_k_kernel = 1;
    int row_mod_tile_zero = 0;
    // if (min_k/KronMatRows[0] >= 256) {
    //   //K dimension is very high. Divide it in different threadblocks to have better parallelism
    //   min_k = min_k/KronMatRows[0];
    //   k_equals_var = 0;
    // }
    // printf("min_k %d\n", min_k);
    uint typeKernelIdx = typeKernelIndex((T)0);

    if (KronMatCols[kronMat] >= 64) {
      //Go through all MaxColsA starting from MAX_K and select the relevant
      min_k = K; //TODO: find MAX_K lower than K
      while (KronGemmKernels[typeKernelIdx][rowParallelism][0][0][log2(min_k)-log2(MIN_K)][log2(KronMatRows[0])-log2(MIN_KP_K)][0].kernel == NULL)
        min_k = min_k / 2;
    } else {
      while (max_k_kernel < MIN_K) {
        max_k_kernel *= KronMatCols[0];
      }
      while (max_k_kernel < MAX_K && KronGemmKernels[typeKernelIdx][rowParallelism][0][0][log2(max_k_kernel)-log2(MIN_K)][log2(KronMatRows[0])-log2(MIN_KP_K)][0].kernel != NULL) {
        // printf("max_k_kernel %d KronMatCols[0] %d\n", max_k_kernel, KronMatCols[0]);
        max_k_kernel *= KronMatCols[0];
      }

      // printf("max_k_kernel %d\n", max_k_kernel);

      if (max_k_kernel > MAX_K || KronGemmKernels[typeKernelIdx][rowParallelism][0][0][log2(max_k_kernel)-log2(MIN_K)][log2(KronMatRows[0])-log2(MIN_KP_K)][0].kernel == NULL)
        max_k_kernel = max_k_kernel/KronMatCols[0];

      // printf("max_k_kernel %d\n", max_k_kernel);

      if (K > max_k_kernel) {
        max_k = 1;
        while (max_k <= max_k_kernel)
          max_k *= KronMatCols[kronMat];
        
        max_k = max_k/KronMatCols[kronMat];
        min_k = min(K, max_k);
      } else {
        min_k = K;
      }
    }
    
    int k_equals_var = (min_k == K) ? 1 : 0;
    // printf("min_k %d k_equals_var %d\n", min_k, k_equals_var);
    uint tileRowA = MaxTileRowsA[log2(KronMatRows[kronMat])-log2(MIN_KP_K)];
    row_mod_tile_zero = (M % tileRowA) == 0;

    //Check that kernel index is valid only in debug mode
    assert(typeKernelIdx < NUM_TYPE_KERNELS);
    assert(row_mod_tile_zero < NUM_ROWS_MOD_TILE_IS_ZERO);
    assert(log2(min_k)-log2(MIN_K) < NUM_MAX_K_KERNELS);
    assert(log2(KronMatRows[0])-log2(MIN_KP_K) < NUM_KP_N_K_KERNELS);

    KernelInfo kernelInfo = KronGemmKernels[typeKernelIdx][rowParallelism][k_equals_var][row_mod_tile_zero][log2(min_k)-log2(MIN_K)][log2(KronMatRows[0])-log2(MIN_KP_K)][0];
    cuda_gemm_func = (KronGemmKernel)kernelInfo.kernel;
    assert(cuda_gemm_func != NULL);
    const uint NumThreads = kernelInfo.NumThreads;
    {
      const uint CRegRows = kernelInfo.CRegRows;
      const uint CRegCols = kernelInfo.CRegCols;
      const uint MaxColsA = kernelInfo.MaxColsA;
      const uint KronRows = kernelInfo.KronRows;
      uint c1 = MAX(1, NumThreads/((kernelInfo.MaxColsA/kernelInfo.KronRows)/CRegRows));
      
      if (kernelInfo.KP_N_TILE_ != c1 * CRegCols) {
        printf("Invalid configuration: KP_N_TILE_ %d != c1*CRegCols %d; NumThreads %d CRegRows %d CRegCols %d MaxColsA %d\n", 
               kernelInfo.KP_N_TILE_, c1 * CRegCols, NumThreads, CRegRows, CRegCols, MaxColsA);
        abort();
      }
      if (MaxColsA/KronRows > kernelInfo.NumThreads*c1* kernelInfo.CRegRows) {
        printf("MaxColsA/KronRows %d kernelInfo.NumThreads*c1* kernelInfo.CRegRows %d\n", MaxColsA/KronRows, kernelInfo.NumThreads*c1* kernelInfo.CRegRows);
        printf("Invalid configuration: MaxColsA %d KronRows %d NumThreads %d CRegRows %d CRegCols %d\n",
               MaxColsA, KronRows, NumThreads, CRegRows, CRegCols);
        abort();
      }
    }
    uint tileKronCols = MaxTileKronCols[log2(KronMatRows[kronMat])-log2(MIN_KP_K)];
    //Create the grid and thread block
    grid = {
              DIVUP(M, tileRowA),
              (K/min_k) * DIVUP(KronMatCols[kronMat], tileKronCols),
              1// DIVUP(KronMatRows[kronMat], EXTERNAL_KP_K_TILE_)
           };
    block = {
              NumThreads, 
              1, 
              1
            };
    
    //Create kernel args;
    void *args[] = {
                    (void*)&M, (void*)&N, (void*)&K, 
                    (void*)&KronMatRows[kronMat],
                    (void*)&KronMatCols[kronMat],
                    &prevResult, 
                    (void*)&kronMats[kronMat], 
                    (void*)kronGemmResult, 
                    &i
                  };

    status = hipLaunchKernel((const void*)cuda_gemm_func, grid, block, &args[0], 0, stream);
    if (status != hipSuccess)
      return status;

    //Double/ring/circular buffer previous result and new result
    if (i < NumKronMats - 1) {
      prevResult = *kronGemmResult;
      if (prevResult == kronGemmResults[0]) {        
        *kronGemmResult = kronGemmResults[1];
      } else if (prevResult == kronGemmResults[1]) {
        *kronGemmResult = kronGemmResults[0];
      }
    }
    
    // CUDA_CHECK(hipDeviceSynchronize());
  }

  return status;
}

/**************************************************
          Library Functions
***************************************************/
hipError_t kronSGEMM(const uint NumKronMats, float* kronGemmResults[], float* x, float* kronMats[], float** result,
                      uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream) {
  if (result == NULL) return hipErrorInvalidValue;
  return generalKronGemm<float, float4>(NumKronMats, kronGemmResults, x, kronMats, result, M, N, K, KronMatCols, KronMatRows, stream);
}

hipError_t kronIGEMM(const uint NumKronMats, int* kronGemmResults[], int* x, int* kronMats[], int** result,
                      uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream) {
  if (result == NULL) return hipErrorInvalidValue;
  return generalKronGemm<int, int4>(NumKronMats, kronGemmResults, x, kronMats, result, M, N, K, KronMatCols, KronMatRows, stream);
}

hipError_t kronDGEMM(const uint NumKronMats, double* kronGemmResults[], double* x, double* kronMats[], double** result,
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream) {
  if (result == NULL) return hipErrorInvalidValue;
  return generalKronGemm<double, double4>(NumKronMats, kronGemmResults, x, kronMats, result, M, N, K, KronMatCols, KronMatRows, stream);
}
