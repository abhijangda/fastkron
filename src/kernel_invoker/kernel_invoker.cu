#include "kernel_invoker/kernel_invoker.h"
#include "utils/utils.h"

static bool isValidKernel(KernelInfo& kernelInfo) {
  const uint NumThreads = kernelInfo.NumThreads;
  const uint CRegRows = kernelInfo.CRegRows;
  const uint CRegCols = kernelInfo.CRegCols;
  const Factor tiledFactor = kernelInfo.tiledFactor;

  const uint ValidThreads = ((kernelInfo.tiledInput.N/tiledFactor.P)/CRegRows) * (tiledFactor.Q/CRegCols);
  if (NumThreads != ROUNDUP(ValidThreads, CUDA_WARP_SIZE)) {
    std::cout << "Invalid kernel config " << kernelInfo << std::endl; 
    return false;
  }

  return true;
}

//Launch cuda kernels
template<uint NumFusedKerns>
hipError_t generalSlicedMatmul(KernelInfo& kernelInfo, const uint kronIndex, 
                                KMMProblem problem,
                                EpilogueParams epilogueParams,
                                hipStream_t stream) {
  hipError_t status;

  //TODO: Do this when loading kernels
  if (!isValidKernel(kernelInfo)) abort();

  //Create the grid and thread block
  KernelParams<NumFusedKerns> params (problem, kronIndex);
  FusedParams<NumFusedKerns> fusedParams (problem, kernelInfo.tiledInput.N);
  // std::cout << "Invoking " << kernelInfo << std::endl;
  //Call kernel
  typedef void (*KronMatmulKernelTy)(KernelParams<NumFusedKerns>, FusedParams<NumFusedKerns>, 
                                     DistributedParams, EpilogueParams, dim3, dim3, hipStream_t);
  KronMatmulKernelTy(kernelInfo.kernel)(params, fusedParams, DistributedParams(), 
                                        epilogueParams, kernelInfo.grid(problem), 
                                        kernelInfo.block(), stream);
  status = hipGetLastError();
  CUDA_CHECK(status);
  return status;
}

hipError_t KernelInvoker::fusedSlicedMatmul(KernelInfo& kernelInfo, const uint kronIndex, 
                                             KMMProblem problem, EpilogueParams epilogueParams,
                                             hipStream_t stream) {
  switch(problem.n) {
    case 1:
      return generalSlicedMatmul<1>(kernelInfo, kronIndex, problem,
                                    epilogueParams, stream);
    case 2:
      return generalSlicedMatmul<2>(kernelInfo, kronIndex, problem,
                                    epilogueParams, stream);
    case 3:
      return generalSlicedMatmul<3>(kernelInfo, kronIndex, problem,
                                    epilogueParams, stream);
    case 4:
      return generalSlicedMatmul<4>(kernelInfo, kronIndex, problem,
                                    epilogueParams, stream);
    case 5:
      return generalSlicedMatmul<5>(kernelInfo, kronIndex, problem,
                                    epilogueParams, stream);
      break;
    default:
        std::cout << "Invalid number of fused kernels" << std::endl;
      return hipErrorInvalidValue;
  }
}

//Launch cuda kernels
template<uint NumFusedKerns>
static hipError_t generalDistributedSlicedMatmul(KernelInfo& kernelInfo, const uint kronIndex, 
                                                  KMMProblem problem,
                                                  DistributedParams distParams, EpilogueParams epilogueParams,
                                                  hipStream_t stream) {
  hipError_t status;
  
  //Do this when loading kernel
  if (!isValidKernel(kernelInfo)) abort();

  KernelParams<NumFusedKerns> params (problem, kronIndex);
  FusedParams<NumFusedKerns> fusedParams (problem, kernelInfo.tiledInput.N);

  //Call kernel
  //TODO: No need to have Type template (T) as part of Kernelparams and DistributedParams
  typedef void (*KronMatmulKernelTy)(KernelParams<NumFusedKerns>, FusedParams<NumFusedKerns>, 
                                     DistributedParams, EpilogueParams, dim3, dim3, hipStream_t);
  KronMatmulKernelTy(kernelInfo.kernel)(params, fusedParams, distParams, epilogueParams, 
                                        kernelInfo.grid(problem), 
                                        kernelInfo.block(), stream);
  status = hipGetLastError();
  CUDA_CHECK(status);
  return status;
}

hipError_t KernelInvoker::fusedDistributedSlicedMatmul(KernelInfo& kernel, const uint kronIndex, 
                                                        KMMProblem problem, DistributedParams distParams, 
                                                        EpilogueParams epilogueParams,
                                                        hipStream_t stream) {
  switch (problem.n) {
    case 1:
      return generalDistributedSlicedMatmul<1>(kernel, kronIndex, problem, 
                                               distParams, epilogueParams, stream);
    case 2:
      return generalDistributedSlicedMatmul<2>(kernel, kronIndex, problem, 
                                               distParams, epilogueParams, stream);
    case 3:
      return generalDistributedSlicedMatmul<3>(kernel, kronIndex, problem, 
                                               distParams, epilogueParams, stream);
    case 4:
      return generalDistributedSlicedMatmul<4>(kernel, kronIndex, problem, 
                                               distParams, epilogueParams, stream);
    case 5:
      return generalDistributedSlicedMatmul<5>(kernel, kronIndex, problem, 
                                               distParams, epilogueParams, stream);
  }

  return hipErrorInvalidValue;
}