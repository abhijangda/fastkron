//Example 1. Application Using C and cuBLAS: 1-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            a[IDX2F(i,j,M)] = (float)((i-1) * N + j);
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            printf ("%7.0f", a[IDX2F(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
