#include "hip/hip_runtime.h"
#include <cassert>

#include <iostream>
#include <unordered_map>

#include "handle.h"
#include "device/params.h"
#include "env.h"
#include "device/kernel_info.h"
#include "autotuner.h"
#include "utils.h"

/**************************************************
          Library Functions
***************************************************/
hipError_t fastKronInit(fastKronHandle* handle, int gpus, int gpusInM, int gpusInK, int gpuLocalKrons) {
  FastKronHandle* h = new FastKronHandle(gpus, gpusInM, gpusInK, gpuLocalKrons);
  *handle = h;
  return hipSuccess;
}

void fastKronDestroy(fastKronHandle handle) {
  handle->free();
  delete handle;
}

hipError_t sgekmm(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], float* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float* temp1, float* temp2,
                   float alpha, float beta, float *z, hipStream_t stream) {
  return handle->sgekmm(NumKronMats, x, kronMats, result,
                                            M, N, K, KronMatCols, KronMatRows, temp1, temp2, 
                                            EpilogueParams<float>(alpha, beta, z), stream);
}

hipError_t igekmm(fastKronHandle handle, const uint NumKronMats, int* x, int* kronMats[], int* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], int* temp1, int* temp2,
                   int alpha, int beta, int *z, hipStream_t stream) {
  return handle->igekmm(NumKronMats, x, kronMats, result, 
                                        M, N, K, KronMatCols, KronMatRows, temp1, temp2,
                                        EpilogueParams<int>(alpha, beta, z), stream);
}

hipError_t dgekmm(fastKronHandle handle, const uint NumKronMats, double* x, double* kronMats[], double* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], double* temp1, double* temp2,
                   double alpha, double beta, double *z, hipStream_t stream) {
  return hipSuccess;
                    // return handle->gekmm(FastKronType::Double, NumKronMats, x, kronMats, result, 
  //                                             M, N, K, KronMatCols, KronMatRows, temp1, temp2,
  //                                             EpilogueParams<double>(alpha, beta, z), stream);
}


hipError_t kronSGEMMOutofCore(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], float** result,
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream) {
  // return singleGPUOutOfCoreKronMatmul<float, float4>(handle, NumKronMats, x, kronMats, result, 
  //                                                    M, N, K, KronMatCols, KronMatRows, stream);
}

// hipError_t kronSGEMMOutofCoreX(FastKronHandle& handle, const uint NumKronMats, float* x, float* kronMats[], float** result,
//   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream[]) {
//   return singleGPUOutOfCoreKronMatmul<float, float4>(handle, NumKronMats, x, kronMats, result, 
//                                                      M, N, K, KronMatCols, KronMatRows, stream);
// }

// hipError_t kronIGEMMOutofCoreX(FastKronHandle& handle, const uint NumKronMats, int* x, int* kronMats[], int** result,
//   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream[]) {
//   return singleGPUOutOfCoreKronMatmul<int, int4>(handle, NumKronMats, x, kronMats, result, 
//                                                  M, N, K, KronMatCols, KronMatRows, stream);
// }

hipError_t kronDistributedSGEMM(fastKronHandle handle, const uint NumKronMats, float* x[], float* kronMats[], float* result[],
                                 uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float** temp1, float** temp2,
                                 hipStream_t streams[]) {
  return handle->distributedsgekmm(NumKronMats, x, kronMats, result, M, N, K, 
                                   KronMatCols, KronMatRows, temp1, temp2, streams);
}

hipError_t gekmmSizes(fastKronHandle handlePtr, const uint NumKronMats, uint M, uint N, uint K, 
                          uint KronMatCols[], uint KronMatRows[], size_t* resultSize, size_t* tempSize) {
  if (resultSize == nullptr) return hipErrorInvalidValue;
  if (tempSize   == nullptr) return hipErrorInvalidValue;
  uint gpuM, gpuK;
  FastKronHandle& handle = *handlePtr;
  if (handle.isDistributed_) {
    if (!checkDistributedKronSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows, 
                                   handle.perGPUKronBatch_, handle.gpusInK_))
      return hipErrorInvalidValue;
    gpuM = M/handle.gpusInM_;
    gpuK = K/handle.gpusInK_;
  } else {
    if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
      return hipErrorInvalidValue;
    gpuM = M;
    gpuK = K;
  }
  size_t tempN = gpuK;
  size_t maxTempN = tempN;
  for (int i = NumKronMats - 1; i >= 0; i--) {
    tempN = (tempN/KronMatRows[i])*KronMatCols[i];
    if (maxTempN < tempN)
      maxTempN = tempN;
  }

  *tempSize   = gpuM * maxTempN;
  if (handle.isDistributed_ and handle.distComm_ == DistComm::NCCL)
    //Include size of send and recv buffers 
    *tempSize = (*tempSize) * 2;
  *resultSize = gpuM * tempN;

  return hipSuccess;
}

hipError_t sgekmmTune(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], 
                                 uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                                 hipStream_t stream) {
  return Autotuner().tune(*handle, NumKronMats, x, kronMats,
                         M, N, K, KronMatCols, KronMatRows,
                         stream);
}

hipError_t dgekmmTune(fastKronHandle handle, const uint NumKronMats, double* x, double* kronMats[], 
                          uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                          hipStream_t stream) {
  return Autotuner().tune(*handle, NumKronMats, x, kronMats, 
                          M, N, K, KronMatCols, KronMatRows,
                          stream);
}

hipError_t idgemmTune(fastKronHandle handle, const uint NumKronMats, int* x, int* kronMats[],
                          uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                          hipStream_t stream) {
  return Autotuner().tune(*handle, NumKronMats, x, kronMats,
                       M, N, K, KronMatCols, KronMatRows,
                       stream);
}

void FastKronHandle::getDistributedSizes(uint M, uint K, uint& gpuM, uint& gpuK) {
  gpuM = M/gpusInM_;
  gpuK = K/gpusInK_;
}