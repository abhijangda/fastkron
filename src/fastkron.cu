#include "hip/hip_runtime.h"
#include <cassert>

#include <iostream>
#include <unordered_map>

#include "handle.h"
#include "device/params.h"
#include "env.h"
#include "device/kernel_info.h"
#include "autotuner.h"
#include "utils.h"
#include "kmmalgo.h"

/**************************************************
          Library Functions
***************************************************/
hipError_t fastKronInit(fastKronHandle* handle, int gpus, int gpusInM, int gpusInK, int gpuLocalKrons) {
  FastKronHandle* h = new FastKronHandle(gpus, gpusInM, gpusInK, gpuLocalKrons);
  *handle = h;
  return hipSuccess;
}

void fastKronDestroy(fastKronHandle handle) {
  handle->free();
  delete handle;
}

hipError_t sgekmm(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], float* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float* temp1, float* temp2,
                   float alpha, float beta, float *z, hipStream_t stream) {
  return handle->xgekmm(NumKronMats, (void*)x, (void**)kronMats, (void*)result,
                        M, N, K, KronMatCols, KronMatRows, (void*)temp1, (void*)temp2, 
                        EpilogueParams::create<float>(alpha, beta, z), stream);
}

hipError_t igekmm(fastKronHandle handle, const uint NumKronMats, int* x, int* kronMats[], int* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], int* temp1, int* temp2,
                   int alpha, int beta, int *z, hipStream_t stream) {
  return handle->xgekmm(NumKronMats, (void*)x, (void**)kronMats, (void*)result, 
                        M, N, K, KronMatCols, KronMatRows, (void*)temp1, (void*)temp2,
                        EpilogueParams::create<int>(alpha, beta, z), stream);
}

hipError_t dgekmm(fastKronHandle handle, const uint NumKronMats, double* x, double* kronMats[], double* result,
                   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], double* temp1, double* temp2,
                   double alpha, double beta, double *z, hipStream_t stream) {
  return hipSuccess;
                    // return handle->gekmm(FastKronType::Double, NumKronMats, x, kronMats, result, 
  //                                             M, N, K, KronMatCols, KronMatRows, temp1, temp2,
  //                                             EpilogueParams<double>(alpha, beta, z), stream);
}


hipError_t kronSGEMMOutofCore(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], float** result,
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream) {
  // return singleGPUOutOfCoreKronMatmul<float, float4>(handle, NumKronMats, x, kronMats, result, 
  //                                                    M, N, K, KronMatCols, KronMatRows, stream);
}

// hipError_t kronSGEMMOutofCoreX(FastKronHandle& handle, const uint NumKronMats, float* x, float* kronMats[], float** result,
//   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream[]) {
//   return singleGPUOutOfCoreKronMatmul<float, float4>(handle, NumKronMats, x, kronMats, result, 
//                                                      M, N, K, KronMatCols, KronMatRows, stream);
// }

// hipError_t kronIGEMMOutofCoreX(FastKronHandle& handle, const uint NumKronMats, int* x, int* kronMats[], int** result,
//   uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], hipStream_t stream[]) {
//   return singleGPUOutOfCoreKronMatmul<int, int4>(handle, NumKronMats, x, kronMats, result, 
//                                                  M, N, K, KronMatCols, KronMatRows, stream);
// }

hipError_t kronDistributedSGEMM(fastKronHandle handle, const uint NumKronMats, float* x[], float* kronMats[], float* result[],
                                 uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float** temp1, float** temp2,
                                 hipStream_t streams[]) {
  return handle->distributedsgekmm(NumKronMats, x, kronMats, result, M, N, K, 
                                   KronMatCols, KronMatRows, temp1, temp2, streams);
}

hipError_t sgekmmTune(fastKronHandle handle, const uint NumKronMats, float* x, float* kronMats[], 
                                 uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                                 hipStream_t stream) {
  return Autotuner(*handle).tune(NumKronMats, (void*)x, (void**)kronMats,
                                 M, N, K, KronMatCols, KronMatRows,
                                 stream);
}

hipError_t dgekmmTune(fastKronHandle handle, const uint NumKronMats, double* x, double* kronMats[], 
                          uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                          hipStream_t stream) {
  return Autotuner(*handle).tune(NumKronMats, (void*)x, (void**)kronMats,
                                 M, N, K, KronMatCols, KronMatRows,
                                 stream);
}

hipError_t idgemmTune(fastKronHandle handle, const uint NumKronMats, int* x, int* kronMats[],
                          uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                          hipStream_t stream) {
  return Autotuner(*handle).tune(NumKronMats, (void*)x, (void**)kronMats,
                       M, N, K, KronMatCols, KronMatRows,
                       stream);
}


hipError_t allocDistributedX(fastKronHandle handle, float* dX[], float* hX, uint M, uint K) {
  handle->allocDistributedX((void**)dX, (void*)hX, M, K);
}
hipError_t gatherDistributedY(fastKronHandle handle, float* dY[], float* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  handle->gatherDistributedY((void**)dY, (void*)hY, M, K, NumKronMats, KronMatCols, KronMatRows);
}

hipError_t gekmmSizes(fastKronHandle handlePtr, const uint NumKronMats, uint M, uint N, uint K, 
                       uint KronMatCols[], uint KronMatRows[], size_t* resultSize, size_t* tempSize) {
  if (resultSize == nullptr) return hipErrorInvalidValue;
  if (tempSize   == nullptr) return hipErrorInvalidValue;

  uint gpuM, gpuK;
  FastKronHandle& handle = *handlePtr;
  KMMProblem problem(KMMShape(M, NumKronMats, KronMatRows, KronMatCols),
                     GeKMMPtrs());
  if (handle.isDistributed_) {
    if (!checkDistributedKronSizes(problem, handle.perGPUKronBatch_, handle.gpusInK_))
      return hipErrorInvalidValue;
    gpuM = M/handle.gpusInM_;
    gpuK = K/handle.gpusInK_;
  } else {
    if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
      return hipErrorInvalidValue;
    gpuM = M;
    gpuK = K;
  }

  uint maxTempN = 0;
  uint resultCols = 0;
                     
  auto e = executeGeKMM(problem, nullptr, nullptr,
    [&maxTempN, &resultCols](const KMMProblem kmm, void* t1, void* t2, hipError_t& e) {
                            maxTempN = std::max(maxTempN, kmm.l);
                            resultCols = kmm.l;
                            e = hipSuccess;
                            return 1U;
                          });

  *tempSize   = gpuM * maxTempN;
  if (handle.isDistributed_ and handle.distComm_ == DistComm::NCCL)
    //Include size of send and recv buffers 
    *tempSize = (*tempSize) * 2;
  *resultSize = gpuM * resultCols;

  return e;
}

// hipError_t allocDistributedX(fastKronHandle handle, int* dX[], int* hX, uint M, uint K) {
//   assert(false); handle->allocDistributedX((void**)dX, (void*)hX, M, K);
// }
// hipError_t gatherDistributedY(fastKronHandle handle, int* dY[], int* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
//   assert(false); handle->gatherDistributedY((void**)dY, (void*)hY, M, K, NumKronMats, KronMatCols, KronMatRows);
// }

// hipError_t allocDistributedX(fastKronHandle handle, double* dX[], double* hX, uint M, uint K) {
//   assert(false);handle->allocDistributedX((void**)dX, (void*)hX, M, K);
// }
// hipError_t gatherDistributedY(fastKronHandle handle, double* dY[], double* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
//   assert(false);handle->gatherDistributedY((void**)dY, (void*)hY, M, K, NumKronMats, KronMatCols, KronMatRows);
// }
