#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <type_traits>
#include <thread>

#include <vector>
#include <iostream>
#include <sstream>
#include <limits>
#include <iomanip>
#include <cstring>

#include "utils.h"
#include "handle.h"
#include "thread_pool.h"
#include "device/otherkernels.cuh"
#include "env.h"
#include "autotuner.h"
#include "kernel_defs.cuh"

/*TODOs:
 1. Using fusion or not should be an environemnt flag
 2. Debug message environment flag*/

std::size_t std::hash<KronMatmulShape>::operator()(const KronMatmulShape& k) const {
  return hash<uint>()(k.KronCols) ^ hash<uint>()(k.KronRows) ^ hash<uint>()(k.ColsA);
}

/**Library entry points to launch cuda kernels**/

//Check N and K is a multiplication of KronMatCols and KronMatRows
bool checkKronMatrixSizes(const uint NumKronMats, 
                                 const uint M, const uint N, const uint K, 
                                 const uint KronMatCols[], const uint KronMatRows[]) {
  uint n=1,k=1;
  for (uint i = 0; i < NumKronMats; i++) {
    k *= KronMatRows[i];
    n *= KronMatCols[i];
  }
  if (n != N || k != K) {
    printf("Invalid Kron product sizes %d != %d, %d != %d\n", n, N, k, K);
    return false;
  }

  return true;
}

bool checkDistributedKronSizes(const uint NumKronMats, 
                                      const uint M, const uint N, const uint K, 
                                      const uint KronMatCols[], const uint KronMatRows[],
                                      const uint LocalKrons, const uint gpusInK) {
  uint prevTempN = K;
  
  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return false;
  
  if (prevTempN % gpusInK != 0) return false;
    
  for (uint i = 0; i < NumKronMats; i += LocalKrons) {
    const uint kronMat = NumKronMats - i - 1;
    const uint NumFusedKerns = min(LocalKrons, NumKronMats - i);
    uint currTempN = prevTempN;
    // printf("243: NumFusedKerns %d kronMat \n", NumFusedKerns);
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }
  
    if (currTempN % gpusInK != 0) return false;
    prevTempN = currTempN;
  }

  return true;
}

KronMatmulShape FastKronHandle::maxCompiledColsA(KronMatmulShape shape) {
  while (compiledKernels.find(shape) == compiledKernels.end()) {
    shape.ColsA /= 2;
    if (shape.ColsA == 1) {
     break;
    }
  }

  return shape;
}

uint FastKronHandle::maxFusedKernels(KronMatmulShape shape) {
  uint numFusedKernels = 0;
  //Go through fused kernels starting from 1 
  //find if the shape exists for the fused kernel
  //if it exist then go to next fused kernel
  while (true) {
    shape.NumFusedKerns = numFusedKernels + 1;
    auto shapeFound = maxCompiledColsA(shape);
    if (shapeFound.ColsA == 1) {
      break;
    }
    numFusedKernels++;
  }

  return numFusedKernels;
}

KernelInfo FastKronHandle::selectKernel(KronMatmulShape shape) {
  //Go through all MaxColsA starting from MAX_K and select the relevant
  KronMatmulShape maxColsAShape = maxCompiledColsA(shape);
  //TODO: Remove kEqVar. it provides only a little improvement in perf
  //but makes writing code hard
  int kEqVar = 0; //(maxColsAShape.ColsA == shape.ColsA) ? 1 : 0;
  auto iter = compiledKernels.find(maxColsAShape);
  if (iter == compiledKernels.end()) {
    std::cout << "No kernel found for " << shape << std::endl;
    abort();
    return KernelInfo{};
  }
  auto kernelInfos = iter->second;
  KernelInfo kernelInfo;
  for (auto info : kernelInfos) {
    //TODO: need to check for type
    //TODO: make use of KernelInfo.canCompute
    if (info.KEqVar == kEqVar) {
      uint tileRowA = info.TileRowsA;
      bool row_mod_tile_zero = (shape.RowsA % tileRowA) == 0;    
      if (info.RowModTileIsZero == row_mod_tile_zero) {
        return info;
      }
    }
  }

  std::cout<<"No kernel selected" << std::endl;
  abort();
  return KernelInfo();
}

bool isValidKernel(KernelInfo& kernelInfo) {
  const uint NumThreads = kernelInfo.NumThreads;
  const uint KronRows = kernelInfo.KronRows;
  const uint KronCols = kernelInfo.KronCols;
  const uint CRegRows = kernelInfo.CRegRows;
  const uint CRegCols = kernelInfo.CRegCols;
  const uint MaxColsA = kernelInfo.MaxColsA;
  const uint TileKronCols = kernelInfo.TileKronCols;

  const uint ValidThreads = ((MaxColsA/KronRows)/CRegRows) * (TileKronCols/CRegCols);
  if (NumThreads != ROUNDUP(ValidThreads, CUDA_WARP_SIZE)) {
    std::cout << "Invalid kernel config " << kernelInfo << std::endl; 
    return false;
  }

  return true;
}

//Launch cuda kernels
template<typename T, uint NumFusedKerns>
hipError_t generalSlicedMatmul(KernelInfo& kernelInfo, const uint kronIndex, 
                                T* x, T* kronMat[NumFusedKerns], T* kronGemmResult,
                                const uint M, const uint N, const uint K, 
                                const uint KronMatCols[NumFusedKerns], const uint KronMatRows[NumFusedKerns],
                                EpilogueParams<T> epilogueParams,
                                hipStream_t stream) {
  hipError_t status;
  
  if (!isValidKernel(kernelInfo)) abort();
  
  //Create the grid and thread block
  dim3 grid;
  dim3 block;
  grid = {
          (K/kernelInfo.MaxColsA) * DIVUP(KronMatCols[0], kernelInfo.TileKronCols),
          DIVUP(M, kernelInfo.TileRowsA),
          1
         };
  block = {
            kernelInfo.NumThreads, 
            1, 
            1
          };
  
  KernelParams<T, NumFusedKerns> params (M, N, K,
                                         KronMatRows, 
                                         KronMatCols, x, 
                                         kronMat, 
                                         kronGemmResult, 
                                         kronIndex);
  FusedParams<T, NumFusedKerns> fusedParams (M, N, K, kernelInfo.MaxColsA, KronMatRows, KronMatCols);
  // std::cout << "Invoking " << kernelInfo << std::endl;
  //Call kernel
  typedef void (*KronMatmulKernelTy)(KernelParams<T, NumFusedKerns>, FusedParams<T, NumFusedKerns>, 
                                     DistributedParams<T>, EpilogueParams<T>, dim3, dim3, hipStream_t);
  KronMatmulKernelTy(kernelInfo.kernel)(params, fusedParams, DistributedParams<T>(), 
                                        epilogueParams, grid, block, stream);
  status = hipGetLastError();
  CUDA_CHECK(status);
  return status;
}

template<typename T>
hipError_t fusedSlicedMatmul(uint NumFusedKerns, KernelInfo& kernelInfo, const uint kronIndex, 
                                T* x, T** krons, T* kronGemmResult,
                                const uint M, const uint N, const uint K, 
                                const uint* FusedKronMatCols, const uint* FusedKronMatRows,
                                EpilogueParams<T> epilogueParams,
                                hipStream_t stream) {
  switch(NumFusedKerns) {
    case 1:
      return generalSlicedMatmul<T, 1>(kernelInfo, kronIndex, x,
                                        krons, kronGemmResult, M, N, K,
                                        FusedKronMatCols, FusedKronMatRows,
                                        epilogueParams, stream);
    case 2:
      return generalSlicedMatmul<T, 2>(kernelInfo, kronIndex, x,
                                          krons, kronGemmResult, M, N, K,
                                          FusedKronMatCols, FusedKronMatRows,
                                          epilogueParams, stream);
    case 3:
      return generalSlicedMatmul<T, 3>(kernelInfo, kronIndex, x,
                                          krons, kronGemmResult, M, N, K,
                                          FusedKronMatCols, FusedKronMatRows,
                                          epilogueParams, stream);
    case 4:
      return generalSlicedMatmul<T, 4>(kernelInfo, kronIndex, x,
                                          krons, kronGemmResult, M, N, K,
                                          FusedKronMatCols, FusedKronMatRows,
                                          epilogueParams, stream);
    case 5:
      return generalSlicedMatmul<T, 5>(kernelInfo, kronIndex, x,
                                          krons, kronGemmResult, M, N, K,
                                          FusedKronMatCols, FusedKronMatRows,
                                          epilogueParams, stream);
      break;
    default:
        std::cout << "Invalid number of fused kernels" << std::endl;
      return hipErrorInvalidValue;
  }
}

//Launch cuda kernels
template<typename T, uint NumFusedKerns>
hipError_t generalDistributedSlicedMatmul(KernelInfo& kernelInfo, const uint kronIndex, 
                                           T* x, T* kronMat[NumFusedKerns], T* kronGemmResult,
                                           const uint M, const uint N, const uint K, 
                                           const uint KronMatCols[NumFusedKerns], const uint KronMatRows[NumFusedKerns],
                                           DistributedParams<T> distParams, hipStream_t stream) {
  hipError_t status;
  
  if (!isValidKernel(kernelInfo)) abort();

  //Create the grid and thread block
  dim3 grid;
  dim3 block;
  
  grid = {
          (K/kernelInfo.MaxColsA) * DIVUP(KronMatCols[0], kernelInfo.TileKronCols),
          DIVUP(M, kernelInfo.TileRowsA),
          1
         };
  block = {
            kernelInfo.NumThreads, 
            1, 
            1
          };

  KernelParams<T, NumFusedKerns> params(M, N, K,
                                        KronMatRows, 
                                        KronMatCols, x, 
                                        kronMat, 
                                        kronGemmResult, 
                                        kronIndex);
  FusedParams<T, NumFusedKerns> fusedParams(M, N, K, kernelInfo.MaxColsA, KronMatRows, KronMatCols);

  //Call kernel
  //TODO: No need to have Type template (T) as part of Kernelparams and DistributedParams
  typedef void (*KronMatmulKernelTy)(KernelParams<T, NumFusedKerns>, FusedParams<T, NumFusedKerns>, 
                                     DistributedParams<T>, EpilogueParams<T>, dim3, dim3, hipStream_t);
  KronMatmulKernelTy(kernelInfo.kernel)(params, fusedParams, distParams, EpilogueParams<T>(), 
                                        grid, block, stream);
  status = hipGetLastError();
  CUDA_CHECK(status);
  return status;
}

template<typename T>
hipError_t fusedDistributedSlicedMatmul(const uint NumFusedKerns, KernelInfo& kernel, const uint kronIndex, 
                                           T* x, T** kronMat, T* kronGemmResult,
                                           const uint M, const uint N, const uint K, 
                                           const uint* FusedKronMatCols, const uint* FusedKronMatRows,
                                           DistributedParams<T> distParams, hipStream_t stream) {
  switch (NumFusedKerns) {
    case 1:
      return generalDistributedSlicedMatmul<T, 1>(kernel, kronIndex, x, 
                                                    kronMat, kronGemmResult, M, N, K, 
                                                    FusedKronMatCols, FusedKronMatRows, 
                                                    distParams, stream);
    case 2:
      return generalDistributedSlicedMatmul<T, 2>(kernel, kronIndex, x, 
                                                    kronMat, kronGemmResult, M, N, K,
                                                    FusedKronMatCols, FusedKronMatRows, 
                                                    distParams, stream);
    case 3:
      return generalDistributedSlicedMatmul<T, 3>(kernel, kronIndex, x, 
                                                    kronMat, kronGemmResult, M, N, K,
                                                    FusedKronMatCols, FusedKronMatRows, 
                                                    distParams, stream);
    case 4:
      return generalDistributedSlicedMatmul<T, 4>(kernel, kronIndex, x, 
                                                    kronMat, kronGemmResult, M, N, K,
                                                    FusedKronMatCols, FusedKronMatRows, 
                                                    distParams, stream);
    case 5:
      return generalDistributedSlicedMatmul<T, 5>(kernel, kronIndex, x, 
                                                    kronMat, kronGemmResult, M, N, K, 
                                                    FusedKronMatCols, FusedKronMatRows, 
                                                    distParams, stream);
  }

  return hipErrorInvalidValue;
}

//TODO: These methods that take handle should be private methods of FastKronHandle
TunedKernelsSeries selectKernelSeries(FastKronHandle& handle, const uint NumKronMats,
                                      uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                                      bool distributedKernel) {
  uint MaxFusedKerns = handle.getUseFusion() ? handle.maxFusedKernels(KronMatmulShape{KronMatCols[0], KronMatRows[0], K, M, 0}) : 1;
  MaxFusedKerns = min(MaxFusedKerns, NumKronMats);
  TunedKernelsSeries tunedSeries;
  uint prevTempN = K;
  for (uint i = 0; i < NumKronMats; i += MaxFusedKerns) {
    const uint kronMat = NumKronMats - i - 1;
    const uint NumFusedKerns = min(MaxFusedKerns, NumKronMats - i);
    uint currTempN = prevTempN;
    // printf("243: NumFusedKerns %d kronMat \n", NumFusedKerns);
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }
  
    bool DistributeToGPUs = distributedKernel && handle.distComm_ == DistComm::P2P && handle.gpusInK_ > 1 && (i == NumKronMats - 1);
    auto selectedKernel = handle.selectKernel(KronMatmulShape{KronMatCols[kronMat], KronMatRows[kronMat], 
                                       prevTempN, M, NumFusedKerns, DistributeToGPUs});
    tunedSeries.push_back({selectedKernel, kronMat - NumFusedKerns, kronMat, prevTempN, 0.0f});
    prevTempN = currTempN;
  }

  return tunedSeries;
}

template<typename T>
hipError_t singleGPUKronMatmul(FastKronHandle& handle, const uint NumKronMats, T* x, T* kronMats[], 
                                T* result,
                                uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], 
                                T* temp1, T* temp2, 
                                EpilogueParams<T> epilogueParams,
                                hipStream_t stream) {
  //Only row major layout of all matrics is supported.
  if (result == nullptr) return hipErrorInvalidValue;
  if (temp1  == nullptr) return hipErrorInvalidValue;

  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return hipErrorInvalidValue;
  
  T* kronGemmResults[2] = {temp1, temp2};
  T* prevKronResult = x;
  T* currKronResult = kronGemmResults[0];

  //TODO: Assumes all factors are of same size and square shape
  TunedKernelsSeries kernelSeries;
  if (handle.tunedKernelSeries.size() > 0) {
    kernelSeries = handle.tunedKernelSeries;
  } else {
    kernelSeries = selectKernelSeries(handle, NumKronMats, M, N, K, 
                                      KronMatCols, KronMatRows, false);
  }

  if (temp2 == nullptr) {
    if (kernelSeries.size() % 2 == 1) {
      kronGemmResults[0] = result;
      kronGemmResults[1] = temp1;
    } else {
      kronGemmResults[0] = temp1;
      kronGemmResults[1] = result;
    }

    currKronResult = kronGemmResults[0];
    prevKronResult = x;
  }

  //Use double buffering for writing result and using output 
  //of previous iteration as input to current
  uint prevTempN = K;
  uint currTempN;
  for (auto kernel : kernelSeries) {
    const uint kronMat = kernel.end;
    const uint NumFusedKerns = kernel.kernel.NumFusedKerns;
    T* krons[NumFusedKerns];
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    currTempN = prevTempN;
    for (int k = 0; k < NumFusedKerns; k++) {
      krons[k] = kronMats[kronMat - k];
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }

    //In the last iteration, write result to the results.    
    if (kronMat - NumFusedKerns + 1 == 0)
      currKronResult = result;

    hipError_t status;

    KernelInfo selectedKernel = kernel.kernel;
    // std::cout << "Invoking " << selectedKernel << " for " << FusedKronMatCols[0] << "x" << FusedKronMatRows[0] << "  " << prevTempN << " " << currTempN << std::endl;
    status = fusedSlicedMatmul<T>(NumFusedKerns, selectedKernel, kronMat, prevKronResult,
                                           krons, currKronResult, M, currTempN, prevTempN,
                                           FusedKronMatCols, FusedKronMatRows,
                                           epilogueParams, stream);
    
    if (status != hipSuccess) return status;
    
    // if (kronMat >= 1)
    // printGPUArray<float>(M, currTempN, (kronMat == 3) ? 8.0f : (kronMat == 2 ? 64.0f : 512.0f),
    //                      (float*)currKronResult, stream);
    // if (kronMat == 3) return hipSuccess;
    prevTempN = currTempN;
    // if (kronMat == 1) return hipSuccess;
    // return hipSuccess;
    //Double/ring/circular buffer previous result and new result
    prevKronResult = currKronResult;
    if (prevKronResult == kronGemmResults[0]) {        
      currKronResult = kronGemmResults[1];
    } else if (prevKronResult == kronGemmResults[1]) {
      currKronResult = kronGemmResults[0];
    }
  }

  return hipSuccess;
}

float minExecTimeOfSeries(uint M, uint K, const uint NumKronMats, 
                          uint KronMatCols[], uint KronMatRows[],
                          uint startKron, bool isDistributed,
                          TunedKernelsSeries& tunedKernels,
                          std::unordered_map<KronMatmulShape, std::pair<KernelInfo, float>> bestKernels) {
  if (startKron >= NumKronMats) return 0;
  bool distP2PStore = isDistributed;
  float minTime = std::numeric_limits<float>::max();
  TunedKernelsSeries minEpilogueKernels;
  TunedKernelFromStart minPrologueKernel;
  for (uint endKron = startKron; endKron < NumKronMats; endKron++) {
    const uint kronMat = endKron;
    //Include KronMats [startKron, ..., endKron]
    const uint NumFusedKerns = endKron - startKron + 1;
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
    }

    //TODO: Change tempN to tempK everywhere else
    uint tempK = K;
    for (int reverseKron = NumKronMats - 1; reverseKron > endKron; reverseKron--) {
      tempK = (tempK/KronMatRows[reverseKron])*KronMatCols[reverseKron];
    }

    KronMatmulShape shape = KronMatmulShape{KronMatCols[kronMat], KronMatRows[kronMat], 
                                            tempK, M, NumFusedKerns, 
                                            distP2PStore && startKron == 0};
    if (bestKernels.find(shape) == bestKernels.end()) continue;
    auto iter = bestKernels.find(shape);
    TunedKernelsSeries epilogueKernels;
    float kernelTime = iter->second.second;
    float epilogueTime = minExecTimeOfSeries(M, K, NumKronMats, KronMatCols,
                                             KronMatRows, endKron + 1, isDistributed, 
                                             epilogueKernels, bestKernels);
    if (minTime > kernelTime + epilogueTime) {
      minTime = kernelTime + epilogueTime;
      minEpilogueKernels = epilogueKernels;
      minPrologueKernel = TunedKernelFromStart(iter->second.first, 
                                               startKron, endKron, tempK, kernelTime);
    }
  }
  tunedKernels = minEpilogueKernels;
  tunedKernels.push_back(minPrologueKernel);

  assert(minTime < std::numeric_limits<float>::max());
  return minTime;
}

//TODO: Create another autotuning object?
template<typename T>
hipError_t singleGPUAutotune(FastKronHandle& handle, const uint NumKronMats, T* x, T* kronMats[],
                              uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                              T* temp1, T* temp2,
                              bool isDistributed, DistributedParams<T> distParams,
                              std::unordered_map<KronMatmulShape, std::pair<KernelInfo, float>>& bestKernels,
                              hipStream_t stream) {
  //Only row major layout of all matrics is supported.
  T* kronGemmResults[2] = {(T*)temp1, (T*)temp2};
  //For performance eval we do not need these to contain any value
  T* prevKronResult = kronGemmResults[0];
  T* currKronResult = kronGemmResults[1];
  //TODO: Assumes all factors are of same size and square shape
  // const uint MaxFusedKerns = handle.getUseFusion() ? 
  //                            maxFusedKernels(KronMatmulShape{KronMatCols[0], KronMatRows[0], K}) : 1;
  //Use double buffering for writing result and using output 
  //of previous iteration as input to current
  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));
  
  //A KronMat is a series of SlicedMats
  //We need to get best kernel for all contiguous SlicedMats
  for (uint startKron = 0; startKron < NumKronMats; startKron++) {
  for (uint endKron = startKron; endKron < NumKronMats; endKron++)   {
    const uint kronMat = endKron;
    //KronMats[startKron, ..., endKron] including endKron
    const uint NumFusedKerns = endKron - startKron + 1;
    T* krons[NumFusedKerns];
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      krons[k] = kronMats[kronMat - k];
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
    }
    uint tempN = K;
    for (int reverseKron = NumKronMats - 1; reverseKron > endKron; reverseKron--) {
      tempN = (tempN/KronMatRows[reverseKron])*KronMatCols[reverseKron];
    }
    uint outTempN = (tempN/KronMatRows[endKron])*KronMatCols[endKron];
    // std::cout << "endKron " << endKron << " startKron " << startKron << " tempN " << tempN << std::endl;
    bool distP2PStore = isDistributed && startKron == 0;
    hipError_t status;
    KronMatmulShape shape = KronMatmulShape{KronMatCols[kronMat], KronMatRows[kronMat], 
                                            tempN, M, NumFusedKerns, distP2PStore};
    if (bestKernels.find(shape) != bestKernels.end()) {
      continue;
    }
    if (!handle.getUseFusion() and NumFusedKerns > 1) continue;
    KernelInfo bestKernel;
    float minTime = std::numeric_limits<float>::max();
    const uint runs = 5;
    const uint warmups = 2;
    std::cout << "Tuning for shape "  << shape << std::endl;
    for (auto shapeAndKernels : handle.compiledKernels) {
      if (!shapeAndKernels.first.sameKronSize(shape)) continue;
      for (auto kernel : shapeAndKernels.second) {
        if (!kernel.canCompute(shape)) continue;
        CUDA_CHECK(hipStreamSynchronize(stream));
        for (int r = 0; r < warmups + runs; r++) {
          if (r == warmups) CUDA_CHECK(hipEventRecord(start, stream));
          if (distP2PStore) {
            status = fusedDistributedSlicedMatmul<T>(NumFusedKerns, kernel, endKron, prevKronResult, 
                                                              krons, currKronResult, M, outTempN, tempN, 
                                                              FusedKronMatCols, FusedKronMatRows, 
                                                              distParams, stream);
          } else {
            status = fusedSlicedMatmul<T>(NumFusedKerns, kernel, endKron, prevKronResult,
                                                  krons, currKronResult, M, outTempN, tempN, 
                                                  FusedKronMatCols, FusedKronMatRows,
                                                  EpilogueParams<T>(), stream);
          }
          // if (status != hipSuccess) break;
        }
        CUDA_CHECK(hipEventRecord(end, stream));
        CUDA_CHECK(hipEventSynchronize(end));
        
        if (status != hipSuccess)
          std::cout << "Error: " << hipGetErrorString(status) << " for " << kernel << " tempN " << tempN << std::endl;
        float kernelTime;
        CUDA_CHECK(hipEventElapsedTime(&kernelTime, start, end));
        std::cout << std::fixed << std::setprecision(2) << 
                     kernel << " runs in " << (kernelTime/runs) << " ms " << std::endl;
        if (kernelTime < minTime) {
          bestKernel = kernel;
          minTime = kernelTime;
        }
        if (status != hipSuccess) return status;
      }
    }

    if (minTime < std::numeric_limits<float>::max()) {
      std::cout << std::fixed << std::setprecision(2) <<
                   "Best kernel for " << shape << ": " << bestKernel << " runs in " << (minTime/runs) << " ms" << std::endl;
      bestKernels.emplace(std::make_pair(shape, std::make_pair(bestKernel, minTime/runs)));
    }
  }}

  return hipSuccess;
}

template<typename T>
hipError_t autotune(FastKronHandle& handle, const uint NumKronMats, T* x, T* kronMats[], 
                     uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                     hipStream_t stream) {
  //Only row major layout of all matrics is supported.
  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return hipErrorInvalidValue;

  std::cout << "N " << N << " K " << K << " KronMatCols[0] " << KronMatCols[0] << " KronMatRows[0] " << KronMatRows[0] << std::endl;
  float minTime = 0;
  if (!handle.isDistributed_) {
    //TODO: temp1_ and temp2_ declaration/allocation is same for both cases
    T* temp1_, *temp2_;
    size_t resultSize = 0, tempSize = 0;
    gekmmSizes(&handle, NumKronMats, M, N, K, KronMatCols, KronMatRows, 
               &resultSize, &tempSize);  
    std::unordered_map<KronMatmulShape, std::pair<KernelInfo, float>> bestKernels;
    CUDA_CHECK(hipMalloc(&temp1_, tempSize * sizeof(T)));
    CUDA_CHECK(hipMalloc(&temp2_, tempSize * sizeof(T)));
    singleGPUAutotune(handle, NumKronMats, x, kronMats, M, N, K, KronMatCols, KronMatRows, 
                      (T*)temp1_, (T*)temp2_, false, DistributedParams<T>(), 
                      bestKernels, stream);
    std::cout << "Finding min execution time of the series" << std::endl;
    TunedKernelsSeries tunedKernels;
    minTime = minExecTimeOfSeries(M, K, NumKronMats,
                                  KronMatCols, KronMatRows, 0, false,
                                  tunedKernels, bestKernels);
    handle.tunedKernelSeries = tunedKernels;
    CUDA_CHECK(hipFree(temp1_));
    CUDA_CHECK(hipFree(temp2_));
  } else {
    if (!checkDistributedKronSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows, 
                                   handle.perGPUKronBatch_, handle.gpusInK_))
      return hipErrorInvalidValue;

    //In distributed case run every LocalKron series on a single GPU
    CUDA_CHECK(hipSetDevice(0));
    T* temp1_[handle.numGPUs_], *temp2_[handle.numGPUs_];
    size_t resultSize = 0, tempSize = 0;
    gekmmSizes(&handle, NumKronMats, M, N, K, KronMatCols, KronMatRows, 
                  &resultSize, &tempSize);
    for (int g = 0; g < handle.numGPUs_; g++) {
      CUDA_CHECK(hipSetDevice(g));
      CUDA_CHECK(hipMalloc(&temp1_[g], tempSize * sizeof(T)));
      CUDA_CHECK(hipMalloc(&temp2_[g], tempSize * sizeof(T)));
    }
    CUDA_CHECK(hipSetDevice(0));
    minTime = std::numeric_limits<float>::max();
    uint gpuM, gpuK;
    handle.getDistributedSizes(M, K, gpuM, gpuK);
    uint prevTempN = gpuK;
    //TODO: This loop is really common and should be a macro?
    std::unordered_map<KronMatmulShape, std::pair<KernelInfo, float>> bestKernels;

    uint bestMaxLocalKrons = 1;
    TunedKernelsSeries minKernelSeries;
    //For P2P go through all MaxLocalKrons and for NCCL set MaxLocalKrons to maximum value
    uint MaxLocalKrons;
    if (handle.distComm_ == DistComm::P2P) {
      MaxLocalKrons = 1;
    } else if (handle.distComm_ == DistComm::NCCL) {
      if (handle.perGPUKronBatch_ > 1)
        MaxLocalKrons = NumKronMats - 1;
      else
        MaxLocalKrons = 1;
    }
    uint UpperLocalKrons = NumKronMats;
    if (handle.distComm_ == DistComm::NCCL && handle.perGPUKronBatch_ == 1)
      UpperLocalKrons = 2;
    
    if (handle.gpusInK_ == 1)
      UpperLocalKrons = 2;

    //TODO: consider only valid krons 
    for (; MaxLocalKrons < UpperLocalKrons; MaxLocalKrons += 1) {
    uint seriesTime = 0;
    TunedKernelsSeries tunedKernelSeries;
    
    for (uint i = 0; i < NumKronMats; i += MaxLocalKrons) {
      const uint kronMat = NumKronMats - i - 1;
      const uint LocalKrons = min(MaxLocalKrons, NumKronMats - i);
      uint currTempN = prevTempN;
      uint LocalKronMatCols[LocalKrons];
      uint LocalKronMatRows[LocalKrons];
      for (int k = 0; k < LocalKrons; k++) {
        LocalKronMatCols[k] = KronMatCols[kronMat - k];
        LocalKronMatRows[k] = KronMatRows[kronMat - k];
        currTempN = (currTempN/LocalKronMatRows[k])*LocalKronMatCols[k];
      }

      T** gpuResults = (T**)temp2_;
      int prevFullK = prevTempN * handle.gpusInK_;
      int currFullN = currTempN * handle.gpusInK_;
      DistributedParams<T> distParams(0, 0, handle.gpusInK_, prevFullK, currFullN, 
                                      prevFullK, currFullN, LocalKronMatCols, LocalKronMatRows, LocalKrons);
      distParams.updateGPUResults(gpuResults);
      singleGPUAutotune(handle, LocalKrons, x, kronMats, gpuM, currTempN, prevTempN, 
                        LocalKronMatCols, LocalKronMatRows, temp1_[0], temp2_[0],
                        handle.gpusInK_ > 1 && handle.isDistributed_ && handle.distComm_ == DistComm::P2P, 
                        distParams, bestKernels, stream);
      TunedKernelsSeries tunedKernels;
      seriesTime += minExecTimeOfSeries(gpuM, prevTempN, LocalKrons,
                                     LocalKronMatCols, LocalKronMatRows, 0,
                                     handle.gpusInK_ > 1 &&handle.isDistributed_ && handle.distComm_ == DistComm::P2P,
                                     tunedKernels, bestKernels);

      for (auto tunedKernel : tunedKernels) {
        tunedKernel.start += kronMat + 1 - LocalKrons;
        tunedKernel.end   += kronMat + 1 - LocalKrons;
        tunedKernelSeries.insert(tunedKernelSeries.begin(), tunedKernel);
      }
    }
    
    if (seriesTime < minTime) {
      minTime = seriesTime;
      handle.tunedKernelSeries = tunedKernelSeries;
      handle.perGPUKronBatch_ = MaxLocalKrons;
    }
    }

    for (int g = 0; g < handle.numGPUs_; g++) {
      CUDA_CHECK(hipSetDevice(g));
      CUDA_CHECK(hipFree(temp1_[g]));
      CUDA_CHECK(hipFree(temp2_[g]));
    }
  }

  std::cout <<"Minimum Time " << minTime << " through kernels: " << std::endl;
  for (auto iter = handle.tunedKernelSeries.rbegin(); iter != handle.tunedKernelSeries.rend(); iter++) {
    std::cout << "  " << (*iter) << std::endl;
  }
  return hipSuccess;
}

void thread_barrier_wait(pthread_barrier_t* barrier) {
  int s = pthread_barrier_wait(barrier);
  assert (s == 0 || s == PTHREAD_BARRIER_SERIAL_THREAD);
}

template<typename T>
void perGPUKronMatmul(ThreadArgs* thArgs) {
  // ThreadArgs<T>& thArgs = *(ThreadArgs<T>*)arg;

  FastKronHandle& handle = *thArgs->handle;
  uint NumKronMats = thArgs->NumKronMats;
  T* x = (T*)thArgs->x;
  T** kronMats = (T**)thArgs->kronMats;
  T** results = (T**)thArgs->result;
  T** temp1 = (T**)thArgs->temp1;
  T** temp2 = (T**)thArgs->temp2;
  uint M = thArgs->M;
  uint N = thArgs->N;
  uint K = thArgs->K;
  uint *KronMatCols = thArgs->KronMatCols;
  uint *KronMatRows = thArgs->KronMatRows;
  hipStream_t* stream = thArgs->stream;
  uint gr = thArgs->gpuRow;
  uint gc = thArgs->gpuCol;
  uint gpusInM_ = thArgs->gpusInM_;
  uint gpusInK_ = thArgs->gpusInK_; 
  uint currTempN;
  uint g = gr * gpusInK_ + gc;
  CUDA_CHECK(hipSetDevice(g));

  hipError_t status;
  
  //Temporaries are swaped after every slicedMatmul
  //TODO: User supplied result should be used as a temp and the final results are written in it
  //TODO: What if Rows are not multiple of GPUs in Rows
  T* innerResults[2] = {(T*)temp1[g], (T*)temp2[g]};
  // std::cout << "handle.gpuM_ " << handle.gpuM_ << " handle.gpuK_ " <<handle.gpuK_ << " gpusInCols " << gpusInCols << " gpusInRows " << gpusInRows << " K " << K << std::endl;
  T* innerPrevResult;
  T* innerCurrResult;
  uint gpuM, gpuK;
  handle.getDistributedSizes(M, K, gpuM, gpuK);
  uint prevTempN = gpuK;

  uint startGpuM = gpuM * gr;
  // const uint gpuM = min(gpuM, M - startGpuM);
  //For first slicedMatmul, x is the input
  innerPrevResult = x;
  innerCurrResult = innerResults[0];
  CUDA_CHECK(hipSetDevice(g));

  //Calculate number of swaps
  if (temp2[g] == nullptr) {
    uint currTempN;
    uint prevTempN = gpuK;
    uint numSwaps = 0;

    for (uint io = 0; io < NumKronMats; io += handle.perGPUKronBatch_) {
      uint KronMulBatchSize = min(handle.perGPUKronBatch_, NumKronMats - io);
      uint MaxI = io + KronMulBatchSize;
      const uint endKron = NumKronMats - io - KronMulBatchSize;
      
      currTempN = prevTempN;

      TunedKernelsSeries kernelSeries;
      uint LocalKronCols[KronMulBatchSize];
      uint LocalKronRows[KronMulBatchSize];
      for (int i = KronMulBatchSize - 1; i >= 0 ; i--) {
        LocalKronCols[i] = KronMatCols[NumKronMats - MaxI + i];
        LocalKronRows[i] = KronMatRows[NumKronMats - MaxI + i];
        currTempN = (currTempN/LocalKronRows[i])*LocalKronCols[i];
      }

      if (handle.tunedKernelSeries.size() > 0) {
        for (auto tunedKernel : handle.tunedKernelSeries) {
          if (tunedKernel.start >= endKron  and tunedKernel.end < endKron + KronMulBatchSize) {
            kernelSeries.insert(kernelSeries.begin(), tunedKernel);
          }
        }
      } else {
        auto localSeries = selectKernelSeries(handle, KronMulBatchSize, gpuM, gpuK, gpuK, 
                                              LocalKronCols, LocalKronRows, true);
        for (auto& kernel : localSeries) {
          kernel.end += endKron;
        }
        kernelSeries = localSeries;
      }

      numSwaps += kernelSeries.size() + ((handle.distComm_ == DistComm::P2P) ? 0 : 1);
    }

    if (numSwaps%2 == 1) {
      innerResults[0] = results[g];
      innerResults[1] = temp1[g];
    } else {
      innerResults[0] = temp1[g];
      innerResults[1] = results[g];
    }

    innerCurrResult = innerResults[0]; 
  }

  for (uint io = 0; io < NumKronMats; io += handle.perGPUKronBatch_) {
    uint KronMulBatchSize = min(handle.perGPUKronBatch_, NumKronMats - io);
    uint MaxI = io + KronMulBatchSize;
    {
      const uint endKron = NumKronMats - io - KronMulBatchSize;
      
      currTempN = prevTempN;

      TunedKernelsSeries kernelSeries;
      uint LocalKronCols[KronMulBatchSize];
      uint LocalKronRows[KronMulBatchSize];
      for (int i = KronMulBatchSize - 1; i >= 0 ; i--) {
        LocalKronCols[i] = KronMatCols[NumKronMats - MaxI + i];
        LocalKronRows[i] = KronMatRows[NumKronMats - MaxI + i];
        currTempN = (currTempN/LocalKronRows[i])*LocalKronCols[i];
      }

      if (handle.tunedKernelSeries.size() > 0) {
        for (auto tunedKernel : handle.tunedKernelSeries) {
          if (tunedKernel.start >= endKron  and tunedKernel.end < endKron + KronMulBatchSize) {
            kernelSeries.insert(kernelSeries.begin(), tunedKernel);
          }
        }
      } else {
        auto localSeries = selectKernelSeries(handle, KronMulBatchSize, gpuM, gpuK, gpuK, 
                                              LocalKronCols, LocalKronRows, true);
        for (auto& kernel : localSeries) {
          kernel.end += endKron;
        }
        kernelSeries = localSeries;
      }

      int prevFullK = prevTempN * handle.gpusInK_;
      int currFullN = currTempN * handle.gpusInK_;
      DistributedParams<T> distParams(gr, gc, handle.gpusInK_, 
                                      prevFullK, currFullN,
                                      prevTempN, currTempN, LocalKronCols, LocalKronRows, KronMulBatchSize);
      uint slicedMuls = 0;
      bool ncclRecvInResult = false;
      for (auto kernel : kernelSeries) {
        //TODO: probably will need to change for fused kernels
        const uint NumFusedKerns = kernel.kernel.NumFusedKerns;
        
        T* krons[NumFusedKerns];
        uint kronCols[NumFusedKerns];
        uint kronRows[NumFusedKerns];
        
        currTempN = prevTempN;
        for (int kk = 0; kk < NumFusedKerns; kk++) {
          krons[kk] = kronMats[g * NumKronMats + kernel.end - kk];
          kronRows[kk] = KronMatRows[kernel.end - kk];
          kronCols[kk] = KronMatCols[kernel.end - kk];
          currTempN = (currTempN/kronRows[kk])*kronCols[kk];
        }

        if (slicedMuls == KronMulBatchSize - 1) {
          CUDA_CHECK(hipStreamSynchronize(stream[g]));
          thread_barrier_wait(thArgs->barrier);
        }
        
        if (kernel.end - NumFusedKerns + 1 == 0) {
          if (handle.distComm_ == DistComm::P2P or handle.gpusInK_ == 1)
            innerCurrResult = results[g];
          else
            ncclRecvInResult = true;
        } 
        
        T** gpuTempResults;
        if (innerCurrResult == temp1[g]) {
          gpuTempResults = (T**)temp1;
        } else if (innerCurrResult == temp2[g]) {
          gpuTempResults = (T**)temp2;
        } else if (innerCurrResult == results[g]) {
          gpuTempResults = (T**)results;
        }
        
        T* gpuResults[handle.gpusInK_];
        for (int _gc = 0; _gc < handle.gpusInK_; _gc++) {
          gpuResults[_gc] = gpuTempResults[gr * handle.gpusInK_ + _gc];
        }
        distParams.updateGPUResults(gpuResults);

        //TODO: a single switch case for FusedKernels?
        hipError_t status;
        status = fusedDistributedSlicedMatmul<T>(NumFusedKerns, kernel.kernel, kernel.end, innerPrevResult, 
                                                          krons, innerCurrResult, gpuM, currTempN, 
                                                          prevTempN, kronCols, kronRows, distParams, 
                                                          stream[g]);
        assert(status == hipSuccess);        
        CUDA_CHECK(hipStreamSynchronize(stream[g]));
        
        // if (gc == 0 and kernel.end == 1) {
        //   printGPUArray(handle.gpuM_, handle.gpuK_, 128.0f*128.0f, innerCurrResult, stream[g]);
        // }
        // if (gc == 0) printf("slicedMuls %d innerCurrResult %p innerPrevResult %p\n", slicedMuls, innerCurrResult, innerPrevResult);
        // if (status != hipSuccess) goto end;
        prevTempN = currTempN;
        //Double/ring/circular buffer previous result and new result
        innerPrevResult = innerCurrResult;
        if (innerPrevResult == innerResults[0]) {
          innerCurrResult = innerResults[1];
        } else if (innerPrevResult == innerResults[1]) {
          innerCurrResult = innerResults[0];
        }
        slicedMuls++;
      }

      CUDA_CHECK(hipStreamSynchronize(stream[g]));
      
      thread_barrier_wait(thArgs->barrier);

      if (handle.distComm_ == DistComm::NCCL && handle.gpusInK_ > 1) {
        size_t resultSize = 0, tempSize = 0;
        if (ncclRecvInResult)
          innerCurrResult = results[g];
        gekmmSizes(&handle, NumKronMats, M, N, K, KronMatCols, KronMatRows, 
                   &resultSize, &tempSize);
        T* sendTemp = temp1[g] + tempSize/2;
        T* recvTemp = temp2[g] + tempSize/2;
        //Call we want to use NCCL Send/Recv
        {
          const uint SliceRows = gpuM;
          const uint SliceCols = currTempN/handle.gpusInK_;
          const size_t sendRecvSize = SliceRows * SliceCols;
          const uint startRow = 0;
          const uint startCol = gc * SliceCols;
          matrixSlice(gpuM, currTempN, innerPrevResult, 
                      startRow, startCol, SliceRows, SliceCols,
                      recvTemp, stream[g], g, io, true);
          dim3 grid = {gpuM, 1,1};
          dim3 block = {256, 1, 1};
          storeGPUTile<T, 256><<<grid, block, 0, stream[g]>>>(M, currTempN*handle.gpusInK_, prevTempN*handle.gpusInK_,
                                                                    KronMatRows[0], KronMatCols[0], gc, handle.gpusInK_,
                                                                    recvTemp, gpuM, currTempN,
                                                                    innerCurrResult, gc, KronMulBatchSize, io, distParams, false);
          // if (g == 0) {
          //   std::cout << "io " << io << " SliceCols " << SliceCols << std::endl;
          //   float val;
          //   if (io == 0) val = 64.0f;
          //   else if (io == 1) val = 64.0f * 64.0f;
          //   else if (io == 2) val = 64.0f * 64.0f * 64.0f;
          //   else if (io == 3) val = 64.0f * 64.0f * 64.0f * 64.0f;
          //   if (io <= 0)
          //   printGPUArray<float>(handle.gpuM_, SliceCols, val,
          //     (float*)innerCurrResult, stream[g]);
          // }
          CUDA_CHECK(hipStreamSynchronize(stream[g]));
        }

        //All GPUs with the same gr share their intermediates
        for (int dst = 0; dst < handle.gpusInK_; dst++) {
          const uint SliceRows = gpuM;
          const uint SliceCols = currTempN/handle.gpusInK_;
          const size_t sendRecvSize = SliceRows * SliceCols;
          if (dst == gc) {
            for (int src = 0; src < handle.gpusInK_; src++) {
              // printf("g %d dst %d src %d\n", g, dst, src);
              if (src == dst) {
              } else {
                NCCLCHECK(ncclRecv(recvTemp, sendRecvSize, ncclFloat, gr * handle.gpusInK_ + src, handle.ncclComms[g], stream[g]));
                CUDA_CHECK(hipStreamSynchronize(stream[g]));
                dim3 grid = {gpuM, 1,1};
                dim3 block = {256, 1, 1};
                storeGPUTile<T, 256><<<grid, block, 0, stream[g]>>>(M, currTempN*handle.gpusInK_, prevTempN*handle.gpusInK_,
                                                                          KronMatRows[0], KronMatCols[0], gc, handle.gpusInK_,
                                                                          recvTemp, gpuM, currTempN,
                                                                          innerCurrResult, src, KronMulBatchSize, io, distParams, false);
                CUDA_CHECK(hipStreamSynchronize(stream[g]));
              }
            }
          } else {
            const uint startRow = 0;
            const uint startCol = dst * SliceCols;
            matrixSlice(gpuM, currTempN, innerPrevResult, 
                        startRow, startCol, SliceRows, SliceCols,
                        sendTemp, stream[g], g, io);
            CUDA_CHECK(hipStreamSynchronize(stream[g]));
            // if (g == 1 && dst == 0) {
            //    printGPUArray<float>(SliceRows, SliceCols, (float*)handle.sendTemps_[g], stream[g]);
            //    printf("699 dst %d g %d\n", dst, g);
            // }
            NCCLCHECK(ncclSend(sendTemp, sendRecvSize, ncclFloat, gr * handle.gpusInK_ + dst, handle.ncclComms[g], stream[g]));
            CUDA_CHECK(hipStreamSynchronize(stream[g]));
          }
        }

        innerPrevResult = innerCurrResult;
        if (innerPrevResult == innerResults[0]) {        
          innerCurrResult = innerResults[1];
        } else if (innerPrevResult == innerResults[1]) {
          innerCurrResult = innerResults[0];
        }
      }
    }

    CUDA_CHECK(hipStreamSynchronize(stream[g]));
    thread_barrier_wait(thArgs->barrier);
  }

  end:
  thArgs->threadResult = {status, (void*)innerPrevResult};
}

template<typename T>
hipError_t distributedKronMatmul(FastKronHandle& handle, const uint NumKronMats, T* x[], T* kronMats[], T* result[],
                                  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float** temp1, float** temp2,
                                  hipStream_t streams[]) {
  uint gpuM, gpuK;
  handle.getDistributedSizes(M, K, gpuM, gpuK);
  if (!checkDistributedKronSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows, handle.perGPUKronBatch_, handle.gpusInK_))
    return hipErrorInvalidValue;

  if (result == NULL)                        return hipErrorInvalidValue;
  if (M % gpuM != 0)                         return hipErrorInvalidValue;
  if (NumKronMats < handle.perGPUKronBatch_) return hipErrorInvalidValue;
  if (temp1 == nullptr)                      return hipErrorInvalidValue;
                      
  const uint batchedKronMuls = handle.perGPUKronBatch_;

  thread_pool<ThreadArgs*>::task tasks[handle.numGPUs_];
  ThreadArgs threadArgs[handle.numGPUs_];

  for (uint thread = 0; thread < handle.numGPUs_; thread++) {
    ThreadArgs args = ThreadArgs(
      &handle,
      NumKronMats,
      (void*)x[thread],
      (void**)kronMats,
      (void**)result,
      M, N, K,
      &KronMatCols[0],
      &KronMatRows[0],
      (void**)temp1, (void**)temp2,
      streams,
      thread/handle.gpusInK_,
      thread % handle.gpusInK_,
      handle.gpusInM_,
      handle.gpusInK_,
      &handle.barriers_[thread/handle.gpusInK_]
    );

    threadArgs[thread] = args;
    tasks[thread] = thread_pool<ThreadArgs*>::task(perGPUKronMatmul<T>, &threadArgs[thread]);
  }

  handle.threads_->execute_tasks(tasks);
  handle.threads_->join_tasks();

  hipError_t status;
  for (uint thread = 0; thread < handle.numGPUs_; thread++) {
    status = threadArgs[thread].threadResult.status;
    // result[thread] =(T*)threadArgs[thread].threadResult.result;
  }

  return status;
}

uint getYColumns(uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  size_t tempN = K;
  size_t maxTempN = tempN;
  for (int i = 0; i < NumKronMats; i++) {
    tempN = (tempN/KronMatRows[i])*KronMatCols[i];
    if (maxTempN < tempN)
      maxTempN = tempN;
  }

  return tempN;
}

template<typename T> hipError_t FastKronHandle_allocDistributedX(FastKronHandle& handle, T* dX[], T* hX, uint M, uint K) {
  //TODO: Make FastKronError type
  if (!handle.isDistributed_) return hipErrorInvalidValue;
  uint gpuM, gpuK;
  handle.getDistributedSizes(M, K, gpuM, gpuK);
  //TODO: Check that hX is on host memory
  T* gpuHostX = new T[((size_t)gpuM) * ((size_t)gpuK)];
  std::cout << "Distributing X to all GPUs "<<std::endl;
  // std::cout << handle.gpuM_ << "  " << handle.gpuK_ << "  " << sizeof(T) << std::endl;
  for (int g = 0; g < handle.numGPUs_; g++) {
    CUDA_CHECK(hipSetDevice(g));
    CUDA_CHECK(hipMalloc(&dX[g], sizeof(T) * gpuM * gpuK));
  }

  for(int gr = 0; gr < handle.gpusInM_; gr++) {
    for (uint gc = 0; gc < handle.gpusInK_; gc++) {
      const uint g = gr * handle.gpusInK_ + gc;
      // std::cout << "g " << g << " gr " <<gr << " gc " << gc << std::endl;
      CUDA_CHECK(hipSetDevice(g));
      uint startGpuM = gpuM * gr;
      uint startGpuK = gpuK * gc;
        
      for (uint m = 0; m < gpuM; m++) {
        std::memcpy(&gpuHostX[m * gpuK], &hX[(startGpuM+m)*K + startGpuK], sizeof(T)*gpuK);
      }
      CUDA_CHECK(hipMemcpy(dX[g], gpuHostX, sizeof(T) * gpuM * gpuK, hipMemcpyHostToDevice));
    }
  }
  delete gpuHostX;
  std::cout << "Distributed X " << std::endl;
  return hipSuccess;
}

template<typename T> hipError_t FastKronHandle_gatherDistributedY(FastKronHandle& handle, T* dY[], T* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  //TODO: Make FastKronError type
  if (!handle.isDistributed_) return hipErrorInvalidValue;
  //TODO: Check that hY is on host memory
  uint gpuM, gpuYCols, YCols;
  YCols = getYColumns(M, K, NumKronMats, KronMatCols, KronMatRows);
  handle.getDistributedSizes(M, YCols, gpuM, gpuYCols);
  T* gpuHostY = new T[gpuM * gpuYCols];
  std::cout << "Gather Y from all GPUs"<<std::endl;

  for(int gr = 0; gr < handle.gpusInM_; gr++) {
    for (uint gc = 0; gc < handle.gpusInK_; gc++) {
      uint g = gr * handle.gpusInK_ + gc;
      CUDA_CHECK(hipSetDevice(g));
      //TODO: check that dX[g] is on GPU g
      CUDA_CHECK(hipMemcpy(gpuHostY, dY[g], 
                            sizeof(T) * gpuM * gpuYCols,
                            hipMemcpyDeviceToHost));
      const uint startGpuM = gpuM * gr;
      const uint startGpuN = gpuYCols * gc;
      for (int m = 0; m < gpuM; m++) {
        std::memcpy(&hY[(startGpuM+m)*YCols + startGpuN],
                    &gpuHostY[m * gpuYCols], sizeof(T)*gpuYCols);
      }
    }
  }
  
  delete gpuHostY;

  std::cout << "Gathered Y" << std::endl;

  return hipSuccess;
}

template<> hipError_t FastKronHandle::allocDistributedX(float* dX[], float* hX, uint M, uint K) {
  return FastKronHandle_allocDistributedX<float>(*this, dX, hX, M, K);
}

template<> hipError_t FastKronHandle::allocDistributedX(double* dX[], double* hX, uint M, uint K) {
  return FastKronHandle_allocDistributedX<double>(*this, dX, hX, M, K);
}

template<> hipError_t FastKronHandle::allocDistributedX(int* dX[], int* hX, uint M, uint K) {
  return FastKronHandle_allocDistributedX<int>(*this, dX, hX, M, K);
}

template<> hipError_t FastKronHandle::gatherDistributedY(float* dY[], float* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  return FastKronHandle_gatherDistributedY<float>(*this, dY, hY, M, K, NumKronMats, KronMatCols, KronMatRows);
}

template<> hipError_t FastKronHandle::gatherDistributedY(double* dY[], double* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  return FastKronHandle_gatherDistributedY<double>(*this, dY, hY, M, K, NumKronMats, KronMatCols, KronMatRows);
}

template<> hipError_t FastKronHandle::gatherDistributedY(int* dY[], int* hY, uint M, uint K, uint NumKronMats, uint KronMatCols[], uint KronMatRows[]) {
  return FastKronHandle_gatherDistributedY<int>(*this, dY, hY, M, K, NumKronMats, KronMatCols, KronMatRows);
}

hipError_t FastKronHandle::distributedsgekmm(const uint NumKronMats, float* x[], float* kronMats[], float* result[],
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], float** temp1, float** temp2,
  hipStream_t streams[]) {
    return distributedKronMatmul<float>(*this, NumKronMats, x, kronMats, result, M, N, K, 
      KronMatCols, KronMatRows, temp1, temp2, streams);
}

hipError_t Autotuner::tune(FastKronHandle& handle, const uint NumKronMats, float* x, float* kronMats[], 
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
  hipStream_t stream) {
    return autotune<float>(handle, NumKronMats, x, kronMats,
      M, N, K, KronMatCols, KronMatRows,
      stream);
}

hipError_t Autotuner::tune(FastKronHandle& handle, const uint NumKronMats, int* x, int* kronMats[], 
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
  hipStream_t stream) {
    return autotune<int>(handle, NumKronMats, x, kronMats,
      M, N, K, KronMatCols, KronMatRows,
      stream);
}

hipError_t Autotuner::tune(FastKronHandle& handle, const uint NumKronMats, double* x, double* kronMats[], 
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
  hipStream_t stream) {
    return autotune<double>(handle, NumKronMats, x, kronMats, 
      M, N, K, KronMatCols, KronMatRows,
      stream);
}

FastKronHandle::FastKronHandle(int gpus, int gpusInM, int gpusInK, int gpuKrons) : tunedKernelSeries() {
  //TODO: Support both modes. Single Process multi gpu and multi process multi gpu
  useFusion_ = true;
  isDistributed_ = gpus > 1;
  if (isDistributed_) {
    //TODO: Setting DistComm in another function
    setUseFusion(false);
    numGPUs_ = gpus;
    bool allP2PAccess = true;
    for (int g1 = 0; g1 < gpus; g1++) {
      for (int g2 = 0; g2 < gpus; g2++) {
        if (g1 == g2) continue;
        int p2pAccess = -1;
        CUDA_CHECK(hipDeviceCanAccessPeer(&p2pAccess, g1, g2));
        if (p2pAccess == 0) {allP2PAccess = false; break;}
        CUDA_CHECK(hipSetDevice(g1));
        CUDA_CHECK(hipDeviceEnablePeerAccess(g2, 0));
      }
      if (!allP2PAccess) break;
    }

    distComm_ = env::getDistComm();

    if (distComm_ == DistComm::P2P) {
      if (!allP2PAccess) {
        std::cout << "P2P Access among GPUs not available using NCCL" << std::endl;
        distComm_ = DistComm::DistCommNone;
      }
    } else if (distComm_ == DistComm::NCCL) {
      int devs[gpus];
      distComm_ = DistComm::NCCL;
      ncclUniqueId ncclId;
      ncclGetUniqueId(&ncclId);
      std::cout << "Initializing NCCL"<<std::endl;
      for (int i = 0; i < gpus; i++) {
        CUDA_CHECK(hipSetDevice(i));
        ncclComms.push_back(nullptr);
        devs[i] = i;
      }
      NCCLCHECK(ncclCommInitAll(&ncclComms[0], gpus, devs));
    }

    if (distComm_ == DistComm::DistCommNone) {
      if (allP2PAccess) {
        distComm_ = DistComm::P2P;
      } else {
        int devs[gpus];
        distComm_ = DistComm::NCCL;
        ncclUniqueId ncclId;
        ncclGetUniqueId(&ncclId);
        std::cout << "Initializing NCCL"<<std::endl;
        for (int i = 0; i < gpus; i++) {
          CUDA_CHECK(hipSetDevice(i));
          ncclComms.push_back(nullptr);
          devs[i] = i;
        }
        NCCLCHECK(ncclCommInitAll(&ncclComms[0], gpus, devs));
      }
    }

    std::cout << "Using " << distComm_ << " for distributed comm" << std::endl;

    if (gpusInK >= 1)
      gpusInK_ = gpusInK;
    else
      gpusInK_ = 2;//ilog2(gpus);
    
    if (gpusInM >= 1)
      gpusInM_ = gpusInM;  
    else
      gpusInM_ = 1;//ilog2(gpus);
      
    //TODO: Check that gpuKrons batch is valid, i.e., P1*P2..PBatch <= gpusInK
    if (gpuKrons > 0)
      perGPUKronBatch_ = gpuKrons;
    else 
      perGPUKronBatch_ = 1;

    //TODO: Check if gpusInK_ == 1 then perGPUKronBatch = NumKrons

    std::cout << "gpusInRows " << gpusInM_ <<
                 " gpusInCols " << gpusInK_ << 
                 " gpuKronBatch " << perGPUKronBatch_ <<
                 std::endl;
    if (gpusInK_ * gpusInM_ != numGPUs_)  {
      std::cout << "gpusInCols * gpusInRows != total gpus (" << 
                   gpusInK_ * gpusInM_ << "!= " << 
                   numGPUs_<< ")" << std::endl;
      abort();
    }
    //TODO: Check that localKrons <= log (gpuK_)_P
    // gpuM_ = M_/gpusInM_;
    // gpuK_ = K_/gpusInK_;
    // gpuN_ = N_/gpusInK_;
    
    //All gpus with same row shares the same barrier
    //TODO: free
    barriers_ = new pthread_barrier_t[gpusInM_];
    threads_ = new thread_pool<ThreadArgs*>(numGPUs_);

    for (int i = 0; i < gpusInM_; i++) {
      int s = pthread_barrier_init(&barriers_[i], NULL, gpusInK_);
      //TODO: Create PTHREAD_CHECK?
      assert (s == 0);
    }
    
    // size_t tempN = gpuK_;
    // size_t maxTempN = tempN;
    // for (int i = 0; i < NumKronMats_; i++) {
    //   tempN = (tempN/KronMatRows_[i])*KronMatCols_[i];
    //   if (maxTempN < tempN)
    //     maxTempN = tempN;
    // }

    // size_t sz = gpuM_ * maxTempN * sizeof(T);
    // std::cout << "Allocating temporaries of size "<< sz << std::endl;
    // std::cout << "Allocated temporaries"<<std::endl;

  }

  //Load kernels into compiledKernels map
  for (uint i = 0; i < sizeof(KronGemmKernels)/sizeof(KernelInfo); i++) {
    KernelInfo& info = KronGemmKernels[i];
    KronMatmulShape shape {info.KronCols, info.KronRows, info.MaxColsA, 0, info.NumFusedKerns, info.DistributeToGPUs};
    auto iter = compiledKernels.find(shape);
    if (iter == compiledKernels.end()) {
      compiledKernels.emplace(std::make_pair(shape, std::vector<KernelInfo>()));
    }
    compiledKernels.at(shape).push_back(info);
  }
  
  //TODO: Check that if distP2PStore is needed then there is a kernel that can 
  //do it
  //TODO: Add if debug
  if (false) {
    uint numKernels = 0;
    std::cout << "Loading compiled kernels" << std::endl;
    for (auto iter : compiledKernels) {
      for (auto kernel : iter.second) {
        // std::cout << kernel << std::endl;
      }
      numKernels += iter.second.size();
    }
    std::cout << "Number of kernels loaded: " << numKernels << std::endl;
  }  
}

hipError_t FastKronHandle::sgekmm(const uint NumKronMats, float* x, float* kronMats[], 
  float* result,
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], 
  float* temp1, float* temp2, 
  EpilogueParams<float> epilogueParams,
  hipStream_t stream) {
    return singleGPUKronMatmul<float>(*this, NumKronMats, x, kronMats, result,
                              M, N, K, KronMatCols, KronMatRows, temp1, temp2,
                              epilogueParams, stream);
}

hipError_t FastKronHandle::igekmm(const uint NumKronMats, int* x, int* kronMats[],
  int* result,
  uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], 
  int* temp1, int* temp2, 
  EpilogueParams<int> epilogueParams,
  hipStream_t stream) {
    return singleGPUKronMatmul<int>(*this, NumKronMats, x, kronMats, result,
                              M, N, K, KronMatCols, KronMatRows, temp1, temp2,
                              epilogueParams, stream);
}