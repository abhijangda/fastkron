#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <type_traits>
#include <thread>

#include <vector>
#include <iostream>
#include <sstream>
#include <limits>
#include <iomanip>
#include <cstring>

#include "utils.h"
#include "handle.h"
#include "thread_pool.h"
#include "device/otherkernels.cuh"
#include "env.h"
#include "autotuner.h"
#include "kernel_defs.cuh"

/*TODOs:
 1. Using fusion or not should be an environemnt flag
 2. Debug message environment flag*/

std::size_t std::hash<KronMatmulShape>::operator()(const KronMatmulShape& k) const {
  return hash<uint>()(k.KronCols) ^ hash<uint>()(k.KronRows) ^ hash<uint>()(k.ColsA);
}

/**Library entry points to launch cuda kernels**/

//Check N and K is a multiplication of KronMatCols and KronMatRows
bool checkKronMatrixSizes(const uint NumKronMats, 
                                 const uint M, const uint N, const uint K, 
                                 const uint KronMatCols[], const uint KronMatRows[]) {
  uint n=1,k=1;
  for (uint i = 0; i < NumKronMats; i++) {
    k *= KronMatRows[i];
    n *= KronMatCols[i];
  }
  if (n != N || k != K) {
    printf("Invalid Kron product sizes %d != %d, %d != %d\n", n, N, k, K);
    return false;
  }

  return true;
}

bool checkDistributedKronSizes(const uint NumKronMats, 
                                      const uint M, const uint N, const uint K, 
                                      const uint KronMatCols[], const uint KronMatRows[],
                                      const uint LocalKrons, const uint gpusInK) {
  uint prevTempN = K;
  
  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return false;
  
  if (prevTempN % gpusInK != 0) return false;
    
  for (uint i = 0; i < NumKronMats; i += LocalKrons) {
    const uint kronMat = NumKronMats - i - 1;
    const uint NumFusedKerns = min(LocalKrons, NumKronMats - i);
    uint currTempN = prevTempN;
    // printf("243: NumFusedKerns %d kronMat \n", NumFusedKerns);
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }
  
    if (currTempN % gpusInK != 0) return false;
    prevTempN = currTempN;
  }

  return true;
}

KronMatmulShape FastKronHandle::maxCompiledColsA(KronMatmulShape shape) {
  while (compiledKernels.find(shape) == compiledKernels.end()) {
    shape.ColsA /= 2;
    if (shape.ColsA == 1) {
     break;
    }
  }

  return shape;
}

uint FastKronHandle::maxFusedKernels(KronMatmulShape shape) {
  uint numFusedKernels = 0;
  //Go through fused kernels starting from 1 
  //find if the shape exists for the fused kernel
  //if it exist then go to next fused kernel
  while (true) {
    shape.NumFusedKerns = numFusedKernels + 1;
    auto shapeFound = maxCompiledColsA(shape);
    if (shapeFound.ColsA == 1) {
      break;
    }
    numFusedKernels++;
  }

  return numFusedKernels;
}

KernelInfo FastKronHandle::selectKernel(KronMatmulShape shape) {
  //Go through all MaxColsA starting from MAX_K and select the relevant
  KronMatmulShape maxColsAShape = maxCompiledColsA(shape);
  //TODO: Remove kEqVar. it provides only a little improvement in perf
  //but makes writing code hard
  int kEqVar = 0; //(maxColsAShape.ColsA == shape.ColsA) ? 1 : 0;
  auto iter = compiledKernels.find(maxColsAShape);
  if (iter == compiledKernels.end()) {
    std::cout << "No kernel found for " << shape << std::endl;
    abort();
    return KernelInfo{};
  }
  auto kernelInfos = iter->second;
  KernelInfo kernelInfo;
  for (auto info : kernelInfos) {
    //TODO: need to check for type
    //TODO: make use of KernelInfo.canCompute
    if (info.KEqVar == kEqVar) {
      uint tileRowA = info.TileRowsA;
      bool row_mod_tile_zero = (shape.RowsA % tileRowA) == 0;    
      if (info.RowModTileIsZero == row_mod_tile_zero) {
        return info;
      }
    }
  }

  std::cout<<"No kernel selected" << std::endl;
  abort();
  return KernelInfo();
}

//TODO: These methods that take handle should be private methods of FastKronHandle
TunedKernelsSeries FastKronHandle::selectKernelSeries(const uint NumKronMats,
                                      uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[],
                                      bool distributedKernel) {
  uint MaxFusedKerns = getUseFusion() ? maxFusedKernels(KronMatmulShape{KronMatCols[0], KronMatRows[0], K, M, 0}) : 1;
  MaxFusedKerns = min(MaxFusedKerns, NumKronMats);
  TunedKernelsSeries tunedSeries;
  uint prevTempN = K;
  for (uint i = 0; i < NumKronMats; i += MaxFusedKerns) {
    const uint kronMat = NumKronMats - i - 1;
    const uint NumFusedKerns = min(MaxFusedKerns, NumKronMats - i);
    uint currTempN = prevTempN;
    // printf("243: NumFusedKerns %d kronMat \n", NumFusedKerns);
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    for (int k = 0; k < NumFusedKerns; k++) {
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }
  
    bool DistributeToGPUs = distributedKernel && distComm_ == DistComm::P2P && gpusInK_ > 1 && (i == NumKronMats - 1);
    auto selectedKernel = selectKernel(KronMatmulShape{KronMatCols[kronMat], KronMatRows[kronMat], 
                                       prevTempN, M, NumFusedKerns, DistributeToGPUs});
    tunedSeries.push_back({selectedKernel, kronMat - NumFusedKerns, kronMat, prevTempN, 0.0f});
    prevTempN = currTempN;
  }

  return tunedSeries;
}

hipError_t FastKronHandle::xgekmm(const uint NumKronMats, void* x, void** kronMats,
                                void* result,
                                uint M, uint N, uint K, uint KronMatCols[], uint KronMatRows[], 
                                void* temp1, void* temp2, 
                                EpilogueParams epilogueParams,
                                hipStream_t stream) {
  //Only row major layout of all matrics is supported.
  if (result == nullptr) return hipErrorInvalidValue;
  if (temp1  == nullptr) return hipErrorInvalidValue;

  if (!checkKronMatrixSizes(NumKronMats, M, N, K, KronMatCols, KronMatRows))
    return hipErrorInvalidValue;
  
  void* kronGemmResults[2] = {temp1, temp2};
  void* prevKronResult = x;
  void* currKronResult = kronGemmResults[0];

  //TODO: Assumes all factors are of same size and square shape
  TunedKernelsSeries kernelSeries;
  if (tunedKernelSeries.size() > 0) {
    kernelSeries = tunedKernelSeries;
  } else {
    kernelSeries = selectKernelSeries(NumKronMats, M, N, K, 
                                      KronMatCols, KronMatRows, false);
  }

  if (temp2 == nullptr) {
    if (kernelSeries.size() % 2 == 1) {
      kronGemmResults[0] = result;
      kronGemmResults[1] = temp1;
    } else {
      kronGemmResults[0] = temp1;
      kronGemmResults[1] = result;
    }

    currKronResult = kronGemmResults[0];
    prevKronResult = x;
  }

  //Use double buffering for writing result and using output 
  //of previous iteration as input to current
  uint prevTempN = K;
  uint currTempN;
  for (auto kernel : kernelSeries) {
    const uint kronMat = kernel.end;
    const uint NumFusedKerns = kernel.kernel.NumFusedKerns;
    void* krons[NumFusedKerns];
    uint FusedKronMatCols[NumFusedKerns];
    uint FusedKronMatRows[NumFusedKerns];
    currTempN = prevTempN;
    for (int k = 0; k < NumFusedKerns; k++) {
      krons[k] = kronMats[kronMat - k];
      FusedKronMatCols[k] = KronMatCols[kronMat - k];
      FusedKronMatRows[k] = KronMatRows[kronMat - k];
      currTempN = (currTempN/FusedKronMatRows[k])*FusedKronMatCols[k];
    }

    //In the last iteration, write result to the results.    
    if (kronMat - NumFusedKerns + 1 == 0)
      currKronResult = result;

    hipError_t status;

    KernelInfo selectedKernel = kernel.kernel;
    // std::cout << "Invoking " << selectedKernel << " for " << FusedKronMatCols[0] << "x" << FusedKronMatRows[0] << "  " << prevTempN << " " << currTempN << std::endl;
    status = kernelInvoker.fusedSlicedMatmul(NumFusedKerns, selectedKernel, kronMat, (void*)prevKronResult,
                               (void**)krons, (void*)currKronResult, M, currTempN, prevTempN,
                               FusedKronMatCols, FusedKronMatRows,
                               epilogueParams, stream);
    
    if (status != hipSuccess) return status;
    
    // if (kronMat >= 1)
    // printGPUArray<float>(M, currTempN, (kronMat == 3) ? 8.0f : (kronMat == 2 ? 64.0f : 512.0f),
    //                      (float*)currKronResult, stream);
    // if (kronMat == 3) return hipSuccess;
    prevTempN = currTempN;
    // if (kronMat == 1) return hipSuccess;
    // return hipSuccess;
    //Double/ring/circular buffer previous result and new result
    prevKronResult = currKronResult;
    if (prevKronResult == kronGemmResults[0]) {        
      currKronResult = kronGemmResults[1];
    } else if (prevKronResult == kronGemmResults[1]) {
      currKronResult = kronGemmResults[0];
    }
  }

  return hipSuccess;
}

FastKronHandle::FastKronHandle(int gpus, int gpusInM, int gpusInK, int gpuKrons) : tunedKernelSeries() {
  //TODO: Support both modes. Single Process multi gpu and multi process multi gpu
  useFusion_ = true;
  isDistributed_ = gpus > 1;
  numGPUs_ = gpus;

  if (isDistributed_) {
    bool allP2PAccess = true;
    for (int g1 = 0; g1 < gpus; g1++) {
      for (int g2 = 0; g2 < gpus; g2++) {
        if (g1 == g2) continue;
        int p2pAccess = -1;
        CUDA_CHECK(hipDeviceCanAccessPeer(&p2pAccess, g1, g2));
        if (p2pAccess == 0) {allP2PAccess = false; break;}
        CUDA_CHECK(hipSetDevice(g1));
        CUDA_CHECK(hipDeviceEnablePeerAccess(g2, 0));
      }
      if (!allP2PAccess) break;
    }

    distComm_ = env::getDistComm();

    if (distComm_ == DistComm::P2P) {
      if (!allP2PAccess) {
        std::cout << "P2P Access among GPUs not available using NCCL" << std::endl;
        distComm_ = DistComm::DistCommNone;
      }
    } else if (distComm_ == DistComm::NCCL) {
      int devs[gpus];
      distComm_ = DistComm::NCCL;
      ncclUniqueId ncclId;
      ncclGetUniqueId(&ncclId);
      std::cout << "Initializing NCCL"<<std::endl;
      for (int i = 0; i < gpus; i++) {
        CUDA_CHECK(hipSetDevice(i));
        ncclComms.push_back(nullptr);
        devs[i] = i;
      }
      NCCLCHECK(ncclCommInitAll(&ncclComms[0], gpus, devs));
    }

    if (distComm_ == DistComm::DistCommNone) {
      if (allP2PAccess) {
        distComm_ = DistComm::P2P;
      } else {
        int devs[gpus];
        distComm_ = DistComm::NCCL;
        ncclUniqueId ncclId;
        ncclGetUniqueId(&ncclId);
        std::cout << "Initializing NCCL"<<std::endl;
        for (int i = 0; i < gpus; i++) {
          CUDA_CHECK(hipSetDevice(i));
          ncclComms.push_back(nullptr);
          devs[i] = i;
        }
        NCCLCHECK(ncclCommInitAll(&ncclComms[0], gpus, devs));
      }
    }

    std::cout << "Using " << distComm_ << " for distributed comm" << std::endl;

    if (gpusInK >= 1)
      gpusInK_ = gpusInK;
    else
      gpusInK_ = 2;//ilog2(gpus);
    
    if (gpusInM >= 1)
      gpusInM_ = gpusInM;  
    else
      gpusInM_ = 1;//ilog2(gpus);
      
    //TODO: Check that gpuKrons batch is valid, i.e., P1*P2..PBatch <= gpusInK
    if (gpuKrons > 0)
      perGPUKronBatch_ = gpuKrons;
    else 
      perGPUKronBatch_ = 1;

    //TODO: Check if gpusInK_ == 1 then perGPUKronBatch = NumKrons

    std::cout << "gpusInRows " << gpusInM_ <<
                 " gpusInCols " << gpusInK_ << 
                 " gpuKronBatch " << perGPUKronBatch_ <<
                 std::endl;
    if (gpusInK_ * gpusInM_ != numGPUs_)  {
      std::cout << "gpusInCols * gpusInRows != total gpus (" << 
                   gpusInK_ * gpusInM_ << "!= " << 
                   numGPUs_<< ")" << std::endl;
      abort();
    }
    //TODO: Check that localKrons <= log (gpuK_)_P
    // gpuM_ = M_/gpusInM_;
    // gpuK_ = K_/gpusInK_;
    // gpuN_ = N_/gpusInK_;
    
    //All gpus with same row shares the same barrier
    //TODO: free
    barriers_ = new pthread_barrier_t[gpusInM_];
    threads_ = new thread_pool<ThreadArgs*>(numGPUs_);

    for (int i = 0; i < gpusInM_; i++) {
      int s = pthread_barrier_init(&barriers_[i], NULL, gpusInK_);
      PTHREAD_BARRIER_CHECK(s);
    }
    
    // size_t tempN = gpuK_;
    // size_t maxTempN = tempN;
    // for (int i = 0; i < NumKronMats_; i++) {
    //   tempN = (tempN/KronMatRows_[i])*KronMatCols_[i];
    //   if (maxTempN < tempN)
    //     maxTempN = tempN;
    // }

    // size_t sz = gpuM_ * maxTempN * sizeof(T);
    // std::cout << "Allocating temporaries of size "<< sz << std::endl;
    // std::cout << "Allocated temporaries"<<std::endl;

  }

  //Load kernels into compiledKernels map
  for (uint i = 0; i < sizeof(KronGemmKernels)/sizeof(KernelInfo); i++) {
    KernelInfo& info = KronGemmKernels[i];
    KronMatmulShape shape {info.KronCols, info.KronRows, info.MaxColsA, 0, info.NumFusedKerns, info.DistributeToGPUs};
    auto iter = compiledKernels.find(shape);
    if (iter == compiledKernels.end()) {
      compiledKernels.emplace(std::make_pair(shape, std::vector<KernelInfo>()));
    }
    compiledKernels.at(shape).push_back(info);
  }
  
  //TODO: Check that if distP2PStore is needed then there is a kernel that can 
  //do it
  //TODO: Add if debug
  if (false) {
    uint numKernels = 0;
    std::cout << "Loading compiled kernels" << std::endl;
    for (auto iter : compiledKernels) {
      for (auto kernel : iter.second) {
        // std::cout << kernel << std::endl;
      }
      numKernels += iter.second.size();
    }
    std::cout << "Number of kernels loaded: " << numKernels << std::endl;
  }  
}

void FastKronHandle::free() {
  if (isDistributed_) {
    for (uint g = 0; g < gpusInM_; g++) {
      int s = pthread_barrier_destroy(&barriers_[g]);
      PTHREAD_BARRIER_CHECK(s);
    }

    delete threads_;
    delete barriers_;

    if (distComm_ == DistComm::NCCL) {
      for (int i=0; i<ncclComms.size(); i++)
        ncclCommDestroy(ncclComms[i]);
    }
  }
  compiledKernels.clear();
}

void FastKronHandle::getDistributedSizes(uint M, uint K, uint& gpuM, uint& gpuK) {
  gpuM = M/gpusInM_;
  gpuK = K/gpusInK_;
}