#include "hip/hip_runtime.h"
// #define C_IN_SHMEM
template<uint MAX_KP_N, uint KP_N_TILE> __device__ uint get_tile_k() {return blockIdx.y/DIVUP(MAX_KP_N, KP_N_TILE);}
template<uint MAX_KP_N, uint KP_N_TILE> __device__ uint get_external_tile_kp_n() {return blockIdx.y%DIVUP(MAX_KP_N, KP_N_TILE);}

__device__ bool isfirstIdx(dim3 idx) {return idx.x == 0 && idx.y == 0 & idx.z == 0;}

__device__ constexpr uint sqrt(uint x)
{
  switch (x) {
    case 1:
      return 1;

    case 2:
      return 2;
    
    case 4:
      return 2;
    
    case 8:
      return 4;
    
    case 16:
      return 4;
    
    case 32:
      return 8;
    
    case 64:
      return 8;
    
    default:
      return 1;
  }
}

template<typename VecT, typename ElemT>
__device__ void loadVecToRegs(VecT& vec, ElemT* regs) {
  //Not implemented
}

template<>
__device__ void loadVecToRegs(float4& vec, float* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}

template<>
__device__ void loadVecToRegs(int4& vec, int* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}


template<>
__device__ void loadVecToRegs(double4& vec, double* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}

template<>
__device__ void loadVecToRegs(double2& vec, double* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
}

template<>
__device__ void loadVecToRegs(float& vec, float* regs) {
  regs[0] = vec;
}

// __launch_bounds__(N_THREADS)
template<typename ElemT, typename VecT, uint N_THREADS, uint N_COARSE_TB, uint TILE_X, uint MAX_K, uint MAX_KP_N, uint MAX_KP_K, uint KP_N_TILE_, uint K_EQUALS_VAR, uint KPK_EQUALS_VAR>
__global__ void kronGemmKernel(const uint RowsC,    const uint ColsC,   const uint ColsA,
                               const uint KronRows, const uint KronCols,
                               const ElemT * __restrict__ glA, 
                               const ElemT * __restrict__ glKronMats, 
                               ElemT       * __restrict__ glC,
                               const uint kp_idx) {
  const uint KP_N_TILE = MIN(KP_N_TILE_, MAX_KP_N);
  const uint NUM_KP_N_TILES = MAX_KP_N/KP_N_TILE;
  const uint INTERNAL_KP_N_TILE = MIN(128, KP_N_TILE);
  const uint EXTERNAL_KP_K_TILE = MIN(EXTERNAL_KP_K_TILE_, MAX_KP_K);
  const uint INTERNAL_KP_K_TILE = MIN(32, EXTERNAL_KP_K_TILE);

  // printf("MAX_K %d MAX_KP_N %d MAX_KP_K %d KP_N_TILE_ %d\n", MAX_K, MAX_KP_N, MAX_KP_K, KP_N_TILE_);
  const uint tileColA = MAX_K/(MAX_KP_K/INTERNAL_KP_K_TILE);
  __shared__ ElemT shKronMats[INTERNAL_KP_K_TILE][INTERNAL_KP_N_TILE];
  __shared__ ElemT shA[TILE_X][tileColA];

  const uint WarpSize   = 32;
  const uint tid        = threadIdx.x;
  const uint wid        = tid/WarpSize;
  const uint lane       = tid%WarpSize;
  const uint blockWarps = blockDim.x/WarpSize;
  uint kronCols;
  uint kronRows;
  uint colsA;
  uint colsC;
 
  if (KPK_EQUALS_VAR) {
    kronCols = MAX_KP_K;
    kronRows = MAX_KP_N;
  } else {
    kronCols = KronCols;
    kronRows = KronRows;
  }

  if (K_EQUALS_VAR) {
    colsA = MAX_K;
    colsC = colsA;
  } else {
    colsA = ColsA;
    colsC = RowsC;
  }

  const uint KPK_SPLIT_SIZE = MIN(16, INTERNAL_KP_K_TILE);
  const uint NUM_KPK_SPLITS = MAX(1, INTERNAL_KP_K_TILE/KPK_SPLIT_SIZE);
  const uint VecTNumElems = (sizeof(VecT)/sizeof(ElemT));

  uint external_tile_kp_k = blockIdx.z;
  
  if (KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K) {
    const uint ldSize = MIN(kronRows*kronCols, VecTNumElems);

    for (uint i = tid*ldSize; i < kronRows * kronCols; i += blockDim.x*ldSize) {
      // shKronMats[i%kronRows][i/kronCols] = glKronMats[i];
      VecT a = *(VecT*)&glKronMats[i];
      ElemT a1[VecTNumElems];
      loadVecToRegs(a, a1);
      #pragma unroll
      for (uint j = 0; j < ldSize; j++) {
        uint idx = i + j;
        shKronMats[idx/MAX_KP_K][idx%MAX_KP_K] = a1[j];
      }
    }
  } else {
  }

  const uint CRegSize = MAX((MAX_K/(MAX_KP_N/KP_N_TILE))/N_THREADS, 1);
  const uint CRegRows = MIN(8, MAX(sqrt(CRegSize), 1)); //MAX(MIN(Creg_SIZE, MIN(MAX_K/MAX_KP_K, 8*N_THREADS)/N_THREADS), 1); //Prefer rows > 4 than cols, to use 128-bit stores
  const uint CRegCols = MIN(MAX_KP_K, MIN(8, CRegSize/CRegRows)); //MIN(MAX_KP_K, Creg_SIZE/CRegRows);
  
#ifndef EVAL
  __syncthreads();
  if (kp_idx == 0 && isfirstIdx(threadIdx) && isfirstIdx(blockIdx)) {
    printf("CRegRows %d CRegCols %d\n", CRegRows, CRegCols);
    // for (int i = 0; i < kronRows; i++) 
    //   for (int j = 0; j < kronCols; j++)
    //     printf("%lf \n", (double)shKronMats[i][j]);
  }
#endif

  const uint NUM_INTERNAL_KP_N_TILES = KP_N_TILE/INTERNAL_KP_N_TILE; //2
  // assert(Creg_SIZE == CRegCols * CRegRows * NUM_INTERNAL_KP_N_TILES);

  register ElemT Creg[TILE_X][CRegRows][CRegCols];

  const uint kp_col_start_ = (tid / ((MAX_K/MAX_KP_K)/CRegRows)) * CRegCols;
  const uint a_col_start_  = (tid % ((MAX_K/MAX_KP_K)/CRegRows)) * CRegRows; 

  for (uint start_row = blockIdx.x * TILE_X; start_row < gridDim.x * TILE_X * N_COARSE_TB; 
       start_row += gridDim.x * TILE_X) {
  // if (start_row == 0 && tid == 0) {
  //   printf("CRegRows %d CRegCols %d\n", CRegRows, CRegCols);
  // }
  
  for (uint kp_col_start = kp_col_start_; kp_col_start < KP_N_TILE     ; 
       kp_col_start +=             MAX(1, N_THREADS/((MAX_K/MAX_KP_K)/CRegRows)) * CRegCols) { //TODO: Something missing in the increment
  for (uint a_col_start  = a_col_start_ ; a_col_start  < MAX_K/MAX_KP_K;
       a_col_start  += N_THREADS * MAX(1, N_THREADS/((MAX_K/MAX_KP_K)/CRegRows)) * CRegRows) {
    #pragma unroll
    for(uint tile_row = 0; tile_row < TILE_X; tile_row++) {
      #pragma unroll
      for (uint reg_i = 0; reg_i < CRegRows; reg_i++) {
        #pragma unroll
        for (uint reg_j = 0; reg_j < CRegCols; reg_j++) {
          Creg[tile_row][reg_i][reg_j] = 0;
        }
      }
    }
  
    for (uint internal_tile_kp_k = 0; internal_tile_kp_k < EXTERNAL_KP_K_TILE; internal_tile_kp_k += INTERNAL_KP_K_TILE) {
      for (uint aRow = 0; aRow < TILE_X; aRow += 1) {
        for (uint a_col = tid*VecTNumElems; a_col < tileColA; a_col += blockDim.x*VecTNumElems) {
          uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
          VecT a;
          if (INTERNAL_KP_K_TILE == MAX_KP_K) {
            a = *(VecT*)&glA[(aRow + start_row) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + a_col];
            // *(VecT*)&shA[aRow][a_col] = a;
            // ElemT a1[4] = {a.x, a.y, a.z, a.w};
            // for (int j = 0; j < VecTNumElems; j++) {
            //   shA[aRow][a_col + j] = a1[j];
            // }
          } else {
            a = *(VecT*)&glA[(aRow + start_row) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + \
                           (a_col/INTERNAL_KP_K_TILE)*kronCols + external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + a_col % INTERNAL_KP_K_TILE];
            // *(VecT*)&shA[aRow][a_col] = a;
          }
          
          ElemT a1[VecTNumElems];
          loadVecToRegs(a, a1);

          #pragma unroll
          for (uint i = 0; i < VecTNumElems; i++) {
            uint ash_col = a_col + i;
            uint a_col_start = (ash_col/INTERNAL_KP_K_TILE)/CRegRows;
           
            uint final_col = (ash_col/INTERNAL_KP_K_TILE)*INTERNAL_KP_K_TILE + (a_col_start + ash_col%INTERNAL_KP_K_TILE)%INTERNAL_KP_K_TILE;
            shA[aRow][final_col] = a1[i];
          }
        }
      }
    
      //TODO: nvcc unrolls this loop, which leads to high register usage
      for (uint internal_tile_kp_n = 0; internal_tile_kp_n < KP_N_TILE; internal_tile_kp_n += INTERNAL_KP_N_TILE) {
        if (!(KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K)) {
          //Create kronCols subwarps and each subwarp loads 0 to INTERNAL_KP_N_TILE elements
          const uint VecTNumElems = sizeof(VecT)/sizeof(ElemT);
          const uint ldSize = MIN(INTERNAL_KP_N_TILE, VecTNumElems);

          for (uint swid = tid/(INTERNAL_KP_N_TILE/ldSize); swid < INTERNAL_KP_K_TILE; swid += blockDim.x/(INTERNAL_KP_N_TILE/ldSize)) {
            uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
            uint col = external_tile_kp_n*KP_N_TILE + internal_tile_kp_n + (tid%(INTERNAL_KP_N_TILE/ldSize))*ldSize;
            uint row = swid;
            // shKronMats[tid%INTERNAL_KP_N_TILE][row] = glKronMats[(external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + row) * kronRows + col];
            VecT a = *(VecT*)&glKronMats[(external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + row) * kronRows + col];
            ElemT a1[VecTNumElems];
            loadVecToRegs(a, a1);
            #pragma unroll
            for (uint i = 0; i < ldSize; i++) {
              uint idx = (tid%(INTERNAL_KP_N_TILE/ldSize))*ldSize + i%ldSize;
              shKronMats[row][idx] = a1[i];
            }
          }
        }

        __syncthreads();
        
        const uint MAX_AR_SZ = MIN(8, KPK_SPLIT_SIZE);

        //Load MAX_AR_SZ elements at a time to limit the register usage
        for (uint ar_start_id = 0; ar_start_id < INTERNAL_KP_K_TILE; ar_start_id += MAX_AR_SZ) {
          register ElemT Ar[TILE_X][CRegRows][MAX_AR_SZ];
          register ElemT KPr[MAX_AR_SZ][CRegCols];

          uint round_start = (a_col_start / CRegRows)%INTERNAL_KP_K_TILE;

          #pragma unroll
          for (uint aRow = 0; aRow < TILE_X; aRow++) {
            #pragma unroll
            for (uint _a_col = 0; _a_col < CRegRows; _a_col++) {
              uint a_col = a_col_start + _a_col;
              for (uint a_elem = 0; a_elem < MAX_AR_SZ; a_elem++)    
                Ar[aRow][_a_col][a_elem] = shA[aRow][a_col * INTERNAL_KP_K_TILE + (ar_start_id + a_elem + round_start)%INTERNAL_KP_K_TILE]; 
            }
          }
          
          #pragma unroll
          for (uint _kp_col = 0; _kp_col < CRegCols; _kp_col++) {
            uint kp_col = kp_col_start + _kp_col;
            for (uint elem = 0; elem < MAX_AR_SZ; elem++)    
              KPr[elem][_kp_col] = shKronMats[ar_start_id + elem][kp_col];
          }

          #pragma unroll
          for (uint aRow = 0; aRow < TILE_X; aRow++)
            #pragma unroll
            for (int i = 0; i < CRegRows; i++)
              #pragma unroll
              for (int j = 0; j < CRegCols; j++)
                #pragma unroll
                for (int k = 0; k < MAX_AR_SZ; k++)
                  Creg[aRow][i][j] += Ar[aRow][i][k] * KPr[k][j];
        }
      }

      __syncthreads();
    }
    
    #pragma unroll 
    for (int aRow = 0; aRow < TILE_X; aRow++) {
      #pragma unroll 
      for (uint reg_j = 0; reg_j < CRegCols; reg_j++) {
        if (CRegRows % 4 == 0) {
          for (uint reg_i = 0; reg_i < CRegRows; reg_i += 4) {          
            const uint cRow = (aRow + start_row);
            uint cCol = kp_col_start*(MAX_K/MAX_KP_K) + reg_j*(MAX_K/MAX_KP_K) + a_col_start + reg_i;
            if (!K_EQUALS_VAR) {
              uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
              cCol = tile_k * (MAX_K/kronCols) + 
                  (cCol/(MAX_K/kronCols)) * (colsA/kronCols) +
                  cCol%(MAX_K/kronCols);
            }
            if (KP_N_TILE != MAX_KP_N) {
              uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
              cCol += external_tile_kp_n*(colsA/(MAX_KP_N/KP_N_TILE)); 
            }
            const uint cIdx = cRow * colsC + cCol;
            // assert(tid == cCol);
            // if (kp_idx == 0&& cRow == 0 && cCol < 64)
            //   printf("tid %d cCol %d kp_col_start %d a_col_start %d reg_i %d reg_j %d\n", tid, cCol, kp_col_start, a_col_start, reg_i, reg_j);
            if (cCol < colsA) {
              VecT c = {Creg[aRow][reg_i][reg_j], Creg[aRow][reg_i+1][reg_j], Creg[aRow][reg_i+2][reg_j], Creg[aRow][reg_i+3][reg_j]};
              *(VecT*)&glC[cIdx] = c;
            }
          }
        } else {
          for (uint reg_i = 0; reg_i < CRegRows; reg_i++) {            
            const uint cRow = (aRow + start_row);
            uint cCol = kp_col_start*(MAX_K/MAX_KP_K) + reg_j*(MAX_K/MAX_KP_K) + a_col_start + reg_i;
            
            if (!K_EQUALS_VAR) {
              uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
              cCol = tile_k * (MAX_K/kronCols) + 
                  (cCol/(MAX_K/kronCols)) * (colsA/kronCols) +
                  cCol%(MAX_K/kronCols);
            }
            if (KP_N_TILE != MAX_KP_N) {
              uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
              cCol += external_tile_kp_n*(colsA/(MAX_KP_N/KP_N_TILE)); 
            }
            const uint cIdx = cRow * colsC + cCol;
            // assert(tid == cCol);
            // if (kp_idx == 0&& cRow == 0 && cCol < 64)
            //   printf("tid %d cCol %d kp_col_start %d a_col_start %d reg_i %d reg_j %d\n", tid, cCol, kp_col_start, a_col_start, reg_i, reg_j);
            if (cCol < colsA) {
              glC[cIdx] = Creg[aRow][reg_i][reg_j];
            }
          }
        }
      }
    }

    __syncthreads();
  }}}
}