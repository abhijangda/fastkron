#include "hip/hip_runtime.h"
// #define C_IN_SHMEM
template<uint MaxKronCols, uint MaxTileSizeKronCols> __device__ uint get_tile_k() {return blockIdx.y/DIVUP(MaxKronCols, MaxTileSizeKronCols);}
template<uint MaxKronCols, uint MaxTileSizeKronCols> __device__ uint get_external_tile_kp_n() {return blockIdx.y%DIVUP(MaxKronCols, MaxTileSizeKronCols);}

__device__ bool isfirstIdx(dim3 idx) {return idx.x == 0 && idx.y == 0 & idx.z == 0;}

__device__ constexpr uint sqrt(uint x) {
  switch (x) {
    case 1:
      return 1;

    case 2:
      return 2;
    
    case 4:
      return 2;
    
    case 8:
      return 4;
    
    case 16:
      return 4;
    
    case 32:
      return 8;
    
    case 64:
      return 8;
    
    default:
      return 1;
  }
}


template<typename VecT, typename ElemT>
__device__ void globalLoadVec(const ElemT* addr, VecT& vec) {
  //Not implemented
}

template<>
__device__ void globalLoadVec(const float* addr, float4& vec) {
  asm ("ld.ca.global.v4.f32 {%0, %1, %2, %3}, [%4];" : "=f"(vec.x), "=f"(vec.y), "=f"(vec.z), "=f"(vec.w) : "l"(addr));
}

template<>
__device__ void globalLoadVec(const int* addr, int4& vec) {
  vec = *(int4*)addr;
}

template<>
__device__ void globalLoadVec(const double* addr, double4& vec) {
  vec = *(double4*)addr;
}

template<>
__device__ void globalLoadVec(const float* addr, float& vec) {
  vec = *addr;
}

template<typename VecT, typename ElemT>
__device__ void loadVecToRegs(VecT& vec, ElemT* regs) {
  //Not implemented
}

//Four Element Vectors
template<typename VecT, typename ElemT>
__device__ void load4ElemVecToRegs(VecT& vec, ElemT* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}

template<>
__device__ void loadVecToRegs(float4& vec, float* regs) {
  load4ElemVecToRegs(vec, regs);
}

template<>
__device__ void loadVecToRegs(int4& vec, int* regs) {
  load4ElemVecToRegs(vec, regs);
}


template<>
__device__ void loadVecToRegs(double4& vec, double* regs) {
  load4ElemVecToRegs(vec, regs);
}

//Two element vectors
template<>
__device__ void loadVecToRegs(double2& vec, double* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
}


//Single element
template<>
__device__ void loadVecToRegs(float& vec, float* regs) {
  regs[0] = vec;
}

//Store PTX instructions for each vector type
template<typename ElemT>
__device__ void globalStore4Elems(ElemT* addr, ElemT elem1, ElemT elem2, ElemT elem3, ElemT elem4) {
}

template<>
__device__ void globalStore4Elems(float* addr, float elem1, float elem2, float elem3, float elem4) {
  // asm ("st.global.v4.f32 [%0], {%1, %2, %3, %4};" :: "l"(addr), "=f"(elem1), "=f"(elem2), "=f"(elem3), "=f"(elem4));
  float4 vec = {elem1, elem2, elem3, elem4};
  *(float4*)addr = vec;
}

template<>
__device__ void globalStore4Elems(int* addr, int elem1, int elem2, int elem3, int elem4) {
  // asm ("st.global.v4.f32 [%0], {%1, %2, %3, %4};" :: "l"(addr), "=f"(elem1), "=f"(elem2), "=f"(elem3), "=f"(elem4));
  int4 vec = {elem1, elem2, elem3, elem4};
  *(int4*)addr = vec;
}

template<>
__device__ void globalStore4Elems(double* addr, double elem1, double elem2, double elem3, double elem4) {
  // asm ("st.global.v4.f32 [%0], {%1, %2, %3, %4};" :: "l"(addr), "=f"(elem1), "=f"(elem2), "=f"(elem3), "=f"(elem4));
  double4 vec = {elem1, elem2, elem3, elem4};
  *(double4*)addr = vec;
}

template<typename ElemT>
__device__ void globalStore2Elems(ElemT* addr, ElemT elem1, ElemT elem2) {
}

template<>
__device__ void globalStore2Elems(float* addr, float elem1, float elem2) {
  float2 vec = {elem1, elem2};
  *(float2*)addr = vec;
}

template<>
__device__ void globalStore2Elems(int* addr, int elem1, int elem2) {
  int2 vec = {elem1, elem2};
  *(int2*)addr = vec;
}

template<>
__device__ void globalStore2Elems(double* addr, double elem1, double elem2) {
  double2 vec = {elem1, elem2};
  *(double2*)addr = vec;
}

template<typename ElemT>
__device__ void globalStore1Elems(ElemT* addr, ElemT elem1) {
  *addr = elem1;
}

template<typename ElemT, typename VecT, uint NumThreads>
__global__ void copyXtoUVAX(const uint RowsC,    const uint ColsC,   const uint ColsA,
                            const uint KronRows, const uint KronCols,
                            ElemT * __restrict__ uvaTemp,
                            const uint uvaRows, const uint uvaCols,
                            const ElemT * __restrict__ glA,
                            const uint uvaPart) {
  const uint WarpSize     = 32;
  const uint tid          = threadIdx.x;
  const uint wid          = tid/WarpSize;
  const uint lane         = tid%WarpSize;
  const uint blockWarps   = blockDim.x/WarpSize;
  
  const uint rowA = blockIdx.x;

  for (uint uvaElem = tid; uvaElem < uvaCols; uvaElem += NumThreads) {
    uvaTemp[rowA * uvaCols + uvaElem] = glA[rowA * ColsA + uvaPart * uvaCols + uvaElem];  
  }
}

template<typename ElemT, typename VecT, uint NumThreads>
__global__ void copyUVATempToY(const uint RowsC,    const uint ColsC,   const uint ColsA,
                            const uint KronRows, const uint KronCols,
                            ElemT * __restrict__ uvaTemp,
                            const uint uvaRows, const uint uvaCols,
                            ElemT * __restrict__ glC,
                            const uint uvaPart, const uint batchedKronMuls, const uint startKronIdx) {
  const uint WarpSize     = 32;
  const uint tid          = threadIdx.x;
  const uint wid          = tid/WarpSize;
  const uint lane         = tid%WarpSize;
  const uint blockWarps   = blockDim.x/WarpSize;
  const uint rowA = blockIdx.x;

  for (uint uvaElem = tid; uvaElem < uvaCols; uvaElem += NumThreads) {
    // uint cCol = outerTileKronCol*(MaxColsA/MaxKronRows) + reg_j*(MaxColsA/MaxKronRows) + shVecI;
    // //(0,0,0,0,0,16,16,16)*128 + (0,1,2,3,..16)*128
    // if (!K_EQUALS_VAR) {
    //   uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronCols>();
    //   cCol = tile_k * (MaxColsA/kronCols) + 
    //       (cCol/(MaxColsA/kronCols)) * (colsA/kronCols) +
    //       cCol%(MaxColsA/kronCols);
    // }
    
    if (batchedKronMuls == 1) {
      uint cCol = uvaPart * (uvaCols/KronRows) + (uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
      glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
    } else if (batchedKronMuls == 2) {
      uint UVAColsRatioKronRowsSquare = (uvaCols/(KronRows * KronRows));
      uint withinP5 = uvaPart * UVAColsRatioKronRowsSquare + 
                      ((uvaElem%(uvaCols/KronRows))/UVAColsRatioKronRowsSquare)*(ColsC/(uvaCols/UVAColsRatioKronRowsSquare)) + 
                      uvaElem % UVAColsRatioKronRowsSquare;
      uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
      uint cCol = p5Index + withinP5; //(uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
      glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
        
      continue;

      if (uvaCols == KronRows * KronRows) {
        uint withinP5 = uvaPart + ((uvaElem%(uvaCols/KronRows))/1)*(ColsC/(uvaCols/1)) + uvaElem % 1; 
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol = p5Index + withinP5; //(uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
        glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
        // if (rowA * ColsA + cCol == 0) printf("208: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);      
      } else if (uvaCols == KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + uvaElem%KronRows;
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      } else if (uvaCols == KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows*KronRows + ((uvaElem%(uvaCols/KronRows))/(KronRows*KronRows))*(ColsA/(uvaCols/(KronRows*KronRows))) + uvaElem%(KronRows*KronRows);
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      } else if (uvaCols == KronRows * KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows*KronRows*KronRows + ((uvaElem%(uvaCols/KronRows))/(KronRows*KronRows*KronRows))*(ColsA/(uvaCols/(KronRows*KronRows*KronRows))) + uvaElem%(KronRows*KronRows*KronRows);
        //uvaPart = 0 to P^3 - 1; uvaPart*KronRows = 0 to P^4 - P; (uvaPart/KronRows) = 0 to P^2 - 1; (uvaPart/KronRows)*KronRows*KronRows = 0 to P^4 - P  
        //uvaElem = 0 to P^3-1; uvaElem%(uvaCols/KronRows) = 0 to P^2 - 1; ((uvaElem%(uvaCols/KronRows))/KronRows) = 0 to P - 1; ((uvaElem%(uvaCols/KronRows))/KronRows) * (ColsA/(uvaCols/KronRows)) = 0 to P^5 - P^4 
        //uvaElem%KronRows = 0 to P - 1
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5; //(uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
        // if (startKronIdx == 0 && rowA == 0 && uvaTemp[rowA * uvaCols + uvaElem] != 256) printf("209: %f to %p (prev %f) at %d\n", uvaTemp[rowA * uvaCols + uvaElem], glC, glC[rowA * ColsA + cCol], uvaElem);
        // uvaTemp[rowA * uvaCols + uvaElem] = (ElemT)1.0f;
        if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
        // else
        //   printf("cCol %d uvaElem %d\n", cCol, uvaElem);
      }
    } else if (batchedKronMuls == 3) {
      uint UVAColsRatioKronRowsSquare = (uvaCols/(KronRows * KronRows * KronRows));
      uint withinP5 = uvaPart * UVAColsRatioKronRowsSquare + 
                      ((uvaElem%(uvaCols/KronRows))/UVAColsRatioKronRowsSquare)*(ColsC/(uvaCols/UVAColsRatioKronRowsSquare)) + 
                      uvaElem % UVAColsRatioKronRowsSquare;
      uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
      uint cCol = p5Index + withinP5; //(uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
      glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
        
      continue;

      if (uvaCols == KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + (uvaElem%KronRows)*(ColsC/uvaCols);
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      } else if (uvaCols == KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + uvaElem%KronRows;
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        // if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      } else if (uvaCols == KronRows * KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows*KronRows + ((uvaElem%(uvaCols/KronRows))/(KronRows*KronRows))*(ColsA/(uvaCols/(KronRows*KronRows))) + uvaElem%(KronRows*KronRows);
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      }
    } else if (batchedKronMuls == 4) {
      if (uvaCols == KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + (uvaElem%KronRows)*(ColsC/uvaCols);;
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        // if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      } else if (uvaCols == KronRows * KronRows * KronRows * KronRows * KronRows) {
        //Assuming ColsA = KronRows ** 6
        uint withinP5 = uvaPart*KronRows + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + uvaElem%KronRows;
        uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
        uint cCol =  p5Index + withinP5;
        if (rowA * ColsA + cCol == 0) printf("209: from %p %f to %p\n", uvaTemp, uvaTemp[rowA * uvaCols + uvaElem], glC);
        if (cCol < ColsA)
          glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
      }
    }
    
    
    // else {
    //   uint withinP5 = uvaPart*KronRows + ((uvaElem%(uvaCols/KronRows))/KronRows)*(ColsA/(uvaCols/KronRows)) + uvaElem%KronRows;
    //   //uvaPart = 0 to P^3 - 1; uvaPart*KronRows = 0 to P^4 - P; (uvaPart/KronRows) = 0 to P^2 - 1; (uvaPart/KronRows)*KronRows*KronRows = 0 to P^4 - P  
    //   //uvaElem = 0 to P^3-1; uvaElem%(uvaCols/KronRows) = 0 to P^2 - 1; ((uvaElem%(uvaCols/KronRows))/KronRows) = 0 to P - 1; ((uvaElem%(uvaCols/KronRows))/KronRows) * (ColsA/(uvaCols/KronRows)) = 0 to P^5 - P^4 
    //   //uvaElem%KronRows = 0 to P - 1
    //   uint p5Index = (uvaElem/(uvaCols/KronRows))*(ColsA/KronRows);
    //   uint cCol =  p5Index + withinP5; //(uvaElem/(uvaCols/KronRows))*(ColsC/KronRows) + uvaElem%(uvaCols/KronRows);
    //   // if (startKronIdx == 0 && rowA == 0 && uvaTemp[rowA * uvaCols + uvaElem] != 256) printf("209: %f to %p (prev %f) at %d\n", uvaTemp[rowA * uvaCols + uvaElem], glC, glC[rowA * ColsA + cCol], uvaElem);
    //   // uvaTemp[rowA * uvaCols + uvaElem] = (ElemT)1.0f;
    //   // if (rowA * ColsA + cCol < 16) printf("209: %f to %p\n", uvaTemp[rowA * uvaCols + uvaElem], glC);
    //   if (cCol < ColsA)
    //     glC[rowA * ColsA + cCol] = uvaTemp[rowA * uvaCols + uvaElem];
    //   // else
    //   //   printf("cCol %d uvaElem %d\n", cCol, uvaElem);
    // }
  }
}

//KP_N is KronCols
//KP_K is KronRows
// __launch_bounds__(NumThreads)
template<typename ElemT, typename VecT, uint NumThreads, RowParallelismTy RowParallelism, uint TileSizeRowsA, 
         bool RowsCModTileIsZero, uint MaxColsA, uint MaxKronCols, uint MaxKronRows, uint KP_N_TILE_, uint K_EQUALS_VAR,
         uint KPK_EQUALS_VAR, uint CRegRows, uint CRegCols, uint SharedTileKronRows>
__global__ void kronGemmKernel(const uint RowsC,    const uint ColsC,   const uint ColsA,
                               const uint KronRows, const uint KronCols,
                               const ElemT * __restrict__ glA, 
                               const ElemT * __restrict__ glKronMats, 
                               ElemT       * __restrict__ glC,
                               const uint kp_idx) {
  const uint WarpSize     = 32;
  const uint tid          = threadIdx.x;
  const uint wid          = tid/WarpSize;
  const uint lane         = tid%WarpSize;
  const uint blockWarps   = blockDim.x/WarpSize;
  const uint VecTNumElems = (sizeof(VecT)/sizeof(ElemT));

  const uint MaxTileSizeKronCols = MIN(KP_N_TILE_,          MaxKronCols);
  const uint MaxTileSizeKronRows = MIN(EXTERNAL_KP_K_TILE_, MaxKronRows);
  const uint TileSizeKronRows    = MIN(SharedTileKronRows,  MaxTileSizeKronRows);
  const uint TileSizeKronCols    = MIN(256,                 MaxTileSizeKronCols);
  const uint TileSizeColsA       = MaxColsA/(MaxKronRows/TileSizeKronRows);
  
  // const uint CRegSize = MAX((MaxColsA/(MaxKronCols/MaxTileSizeKronCols))/NumThreads, 1);
  // const uint CRegRows = MIN(8, MAX(sqrt(CRegSize), 1));
  // const uint CRegCols = MIN(MaxKronRows, MIN(8, CRegSize/CRegRows));
  
  register   ElemT regC[TileSizeRowsA][CRegRows][CRegCols];
  __shared__ ElemT shA[TileSizeRowsA][TileSizeColsA];
  __shared__ ElemT shKronMats[TileSizeKronRows][TileSizeKronCols];

#ifndef EVAL
  __syncthreads();
  if (kp_idx == 0 && isfirstIdx(threadIdx) && isfirstIdx(blockIdx)) {
    printf("CRegRows %d CRegCols %d\n", CRegRows, CRegCols);
    // for (int i = 0; i < kronRows; i++) 
    //   for (int j = 0; j < kronCols; j++)
    //     printf("%lf \n", (double)shKronMats[i][j]);
  }
#endif

  // const uint NUM_INTERNAL_KP_N_TILES = MaxTileSizeKronRows/TileSizeKronRows;
  // assert(Creg_SIZE == CRegCols * CRegRows * NUM_INTERNAL_KP_N_TILES);
  uint kronCols;
  uint kronRows;
  uint colsA;
  uint colsC;
 
  if (KPK_EQUALS_VAR) {
    kronCols = MaxKronCols;
    kronRows = MaxKronRows;
  } else {
    kronCols = KronCols;
    kronRows = KronRows;
  }

  if (K_EQUALS_VAR) {
    colsA = MaxColsA;
    colsC = colsA;
  } else {
    colsA = ColsA;
    colsC = ColsC;
  }

  const uint RegTileSizeACols = MIN(8, TileSizeKronCols);
  
  const uint external_tile_kp_k = blockIdx.z;
  constexpr uint wSz = ((MaxColsA/MaxKronRows)/CRegRows);

  const uint kp_col_start_ = (tid / wSz) * CRegCols; 
  const uint a_col_start_  = (tid % wSz) * CRegRows; 

  if (MaxTileSizeKronCols == MaxKronCols && TileSizeKronCols == MaxKronCols && TileSizeKronRows == MaxKronRows) {
    const uint loadInstr = MIN(kronRows*kronCols, VecTNumElems);

    for (uint eIdx = tid*loadInstr; eIdx < kronRows*kronCols; eIdx += blockDim.x*loadInstr) {
      ElemT regElems[VecTNumElems];
      VecT vec;

      vec = *(VecT*)&glKronMats[eIdx];
      loadVecToRegs(vec, regElems);

      #pragma unroll
      for (uint vecElem = 0; vecElem < loadInstr; vecElem++) {
        uint idx = eIdx + vecElem;
        shKronMats[idx/MaxKronRows][idx%MaxKronRows] = regElems[vecElem];
      }
    }
  }

  const uint tileRowA  = blockIdx.x * TileSizeRowsA;
  // for (uint tileRowA  = blockIdx.x * TileSizeRowsA;
  //           tileRowA  < gridDim.x  * TileSizeRowsA;
  //           tileRowA += gridDim.x  * TileSizeRowsA) 
  {
  // if (tid == 0) {
  //   printf("tileRowA %d blockIdx.x %d gridDim.x %d TileSizeRowsA %d\n", tileRowA, blockIdx.x, gridDim.x, TileSizeRowsA);
  // }
  const uint outerTileKronCol =  kp_col_start_;
  // for (uint outerTileKronCol =  kp_col_start_;
  //           outerTileKronCol <  MaxTileSizeKronCols;
  //           outerTileKronCol += MAX(1, NumThreads/((MaxColsA/MaxKronRows)/CRegRows)) * CRegCols) 
  {
  const uint tileColA    =  a_col_start_ ;
  // for (uint tileColA    =  a_col_start_ ;
  //           tileColA    <  MaxColsA/MaxKronRows;
  //           tileColA    += NumThreads * MAX(1, NumThreads/((MaxColsA/MaxKronRows)/CRegRows)) * CRegRows) 
  
  {

    #pragma unroll
    for (uint r = 0; r < TileSizeRowsA; r++) {
    #pragma unroll
    for (uint i = 0; i < CRegRows;      i++) {
    #pragma unroll
    for (uint j = 0; j < CRegCols;      j++) {
      regC[r][i][j] = 0;
    }}}

    for (uint tileKronRow = 0; tileKronRow < MaxTileSizeKronRows; tileKronRow += TileSizeKronRows) {
      for (uint rowA = 0; rowA < (RowsCModTileIsZero ? TileSizeRowsA : MIN(TileSizeRowsA, RowsC - tileRowA)); rowA += 1) {
        for (uint a_col = tid*VecTNumElems; a_col < TileSizeColsA; a_col += NumThreads*VecTNumElems) {
          uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronCols>();
          const ElemT* addrA;
          VecT  vec;
          ElemT elems[VecTNumElems];

          if (TileSizeKronRows == MaxKronRows) {
            addrA = &glA[(rowA + tileRowA) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MaxColsA) + a_col];
            // *(VecT*)&shA[rowA][a_col] = a;
            // ElemT a1[4] = {a.x, a.y, a.z, a.w};
            // for (int j = 0; j < VecTNumElems; j++) {
            //   shA[rowA][a_col + j] = a1[j];
            // }
          } else {
            addrA = &glA[(rowA + tileRowA) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MaxColsA) + \
                         (a_col/TileSizeKronRows)*kronCols + external_tile_kp_k * MaxTileSizeKronRows + tileKronRow + a_col % TileSizeKronRows];
            // *(VecT*)&shA[rowA][a_col] = a;
          }

          globalLoadVec(addrA, vec);
          loadVecToRegs(vec, elems);

          #pragma unroll
          for (uint i = 0; i < VecTNumElems; i++) {
            uint ash_col = a_col + i;
            uint tileColA = (ash_col/TileSizeKronRows)/CRegRows;
           
            uint final_col = (ash_col/TileSizeKronRows)*TileSizeKronRows + (tileColA + ash_col%TileSizeKronRows)%TileSizeKronRows;
            shA[rowA][final_col] = elems[i];
          }
        }
      }
    
      //TODO: nvcc unrolls this loop, which leads to high register usage
      for (uint tileKronCol = 0; tileKronCol < MaxTileSizeKronCols; tileKronCol += TileSizeKronCols) {
        if (!(MaxTileSizeKronCols == MaxKronCols && TileSizeKronCols == MaxKronCols && TileSizeKronRows == MaxKronRows)) {
          //Create kronCols subwarps and each subwarp loads 0 to TileSizeKronRows elements
          const uint loadInstr = MIN(TileSizeKronCols, VecTNumElems);

          for (uint swid = tid/(TileSizeKronCols/loadInstr); swid < TileSizeKronRows; swid += NumThreads/(TileSizeKronCols/loadInstr)) {
            VecT  vec;
            ElemT elems[VecTNumElems];

            const uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronCols>();
            const uint col = external_tile_kp_n*MaxTileSizeKronCols + tileKronCol + (tid%(TileSizeKronCols/loadInstr))*loadInstr;
            const uint row = swid;
            // shKronMats[tid%TileSizeKronRows][row] = glKronMats[(external_tile_kp_k * MaxTileSizeKronCols + tileKronRow + row) * kronRows + col];

            globalLoadVec(&glKronMats[(external_tile_kp_k * MaxTileSizeKronRows + tileKronRow + row) * kronRows + col], vec);
            loadVecToRegs(vec, elems);

            #pragma unroll
            for (uint e = 0; e < loadInstr; e++) {
              uint linearIdx = (tid%(TileSizeKronCols/loadInstr))*loadInstr + e;
              shKronMats[row][linearIdx] = elems[e];
            }
          }
        }

        __syncthreads();

        //Load RegTileSizeACols elements at a time to limit the register usage
        for (uint regTileACol = 0; regTileACol < TileSizeKronRows; regTileACol += RegTileSizeACols) {
          register ElemT Ar[TileSizeRowsA][CRegRows][RegTileSizeACols];
          register ElemT KPr[RegTileSizeACols][CRegCols];

          uint round_start = (tileColA / CRegRows)%TileSizeKronRows;

          #pragma unroll
          for (uint rowA = 0; rowA < TileSizeRowsA; rowA++) {
          if (RowsCModTileIsZero || (TileSizeRowsA > 1 && rowA < RowsC - tileRowA)) {
            #pragma unroll
            for (uint rowC = 0; rowC < CRegRows; rowC++) {
              uint shACol = tileColA + rowC;
              #pragma unroll
              for (uint colC = 0; colC < RegTileSizeACols; colC++)
                Ar[rowA][rowC][colC] = shA[rowA][shACol * TileSizeKronRows + (regTileACol + colC + round_start)%TileSizeKronRows];
          }}}
          
          #pragma unroll
          for (uint colC = 0; colC < CRegCols; colC++) {
            uint shKronCol = outerTileKronCol + colC;//TODO: Should outerTileKronCol be here?
            #pragma unroll
            for (uint elem = 0; elem < RegTileSizeACols; elem++)    
              KPr[elem][colC] = shKronMats[regTileACol + elem][shKronCol];
          }

          #pragma unroll
          for (uint rowA = 0; rowA < TileSizeRowsA; rowA++)
          if (RowsCModTileIsZero || (TileSizeRowsA > 1 && rowA < RowsC - tileRowA)) 
          {
            #pragma unroll
            for (uint i = 0;    i < CRegRows;         i++)
            #pragma unroll
            for (uint j = 0;    j < CRegCols;         j++)
            #pragma unroll
            for (uint k = 0;    k < RegTileSizeACols; k++)
              regC[rowA][i][j] += Ar[rowA][i][k] * KPr[k][j];
          }
        }
      }

      __syncthreads();
    }

    #pragma unroll
    for (int rowA = 0; rowA < TileSizeRowsA; rowA++) {
      if (RowsCModTileIsZero || (TileSizeRowsA > 1 && rowA < RowsC - tileRowA)) {
        #pragma unroll
        for (uint reg_j = 0; reg_j < CRegCols; reg_j++) {
          //Three least significant bits of CRegRows can be either 4, 2, or 1
          constexpr uint vecTyNumElems = CRegRows & (8 - 1);
  #ifndef EVAL
          if (vecTyNumElems != 4 && vecTyNumElems != 2 && vecTyNumElems != 1)
            printf("Invalid vecTyNumElems %d\n", vecTyNumElems);
  #endif
          for (uint reg_i = 0; reg_i < CRegRows; reg_i += vecTyNumElems) {
            if (vecTyNumElems > 1) {
              shA[0][tid * vecTyNumElems] = regC[rowA][reg_i][reg_j];
              shA[0][tid * vecTyNumElems+1] = regC[rowA][reg_i+1][reg_j];
              if (vecTyNumElems > 2) {
                shA[0][tid * vecTyNumElems+2] = regC[rowA][reg_i+2][reg_j];
                shA[0][tid * vecTyNumElems+3] = regC[rowA][reg_i+3][reg_j];
              }
              
              __syncwarp();
              for (uint shVecI = tid%wSz; shVecI < vecTyNumElems*wSz; shVecI += wSz) {
                const uint cRow = rowA + tileRowA;
                uint cCol = outerTileKronCol*(MaxColsA/MaxKronRows) + reg_j*(MaxColsA/MaxKronRows) + shVecI;
                //(0,0,0,0,0,16,16,16)*128 + (0,1,2,3,..16)*128
                if (!K_EQUALS_VAR) {
                  uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronCols>();
                  cCol = tile_k * (MaxColsA/kronCols) + 
                      (cCol/(MaxColsA/kronCols)) * (colsA/kronCols) +
                      cCol%(MaxColsA/kronCols);
                }
                if (MaxTileSizeKronCols != MaxKronCols) {
                  uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronCols>();
                  cCol += external_tile_kp_n*(colsA/(MaxKronCols/MaxTileSizeKronCols)); 
                }
                const uint cIdx = cRow * colsC + cCol;
                // assert(tid == cCol);
                // if (kp_idx == 0&& cRow == 0)
                //   printf("tid %d cCol %d outerTileKronCol %d tileColA %d reg_i %d reg_j %d vecTyNumElems %d\n", tid, cCol, outerTileKronCol, tileColA, reg_i, reg_j, vecTyNumElems);
                if (cCol < colsA) {
                  // printf("tid %d cCol %d outerTileKronCol %d tileColA %d reg_i %d reg_j %d vecTyNumElems %d shVecI %d\n", 
                  // tid, cCol, outerTileKronCol, tileColA, reg_i, reg_j, vecTyNumElems, shVecI);
                  glC[cIdx] = shA[0][(tid/wSz)*wSz*vecTyNumElems + shVecI];
                }
              }
              __syncwarp();
            } else {
              const uint cRow = (rowA + tileRowA);
              uint cCol = outerTileKronCol*(MaxColsA/MaxKronRows) + reg_j*(MaxColsA/MaxKronRows) + tileColA + reg_i;
              if (!K_EQUALS_VAR) {
                uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronCols>();
                cCol = tile_k * (MaxColsA/kronCols) + 
                    (cCol/(MaxColsA/kronCols)) * (colsA/kronCols) +
                    cCol%(MaxColsA/kronCols);
              }
              if (MaxTileSizeKronCols != MaxKronCols) {
                uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronCols>();
                cCol += external_tile_kp_n*(colsA/(MaxKronCols/MaxTileSizeKronCols)); 
              }
              const uint cIdx = cRow * colsC + cCol;
              // assert(tid == cCol);
              // if (kp_idx == 0&& cRow == 0 && cCol < 64)
              //   printf("tid %d cCol %d outerTileKronCol %d tileColA %d reg_i %d reg_j %d\n", tid, cCol, outerTileKronCol, tileColA, reg_i, reg_j);
              if (cCol < colsA) {
                switch (vecTyNumElems) {
                  case 4:
                    globalStore4Elems(&glC[cIdx], regC[rowA][reg_i][reg_j], regC[rowA][reg_i+1][reg_j], regC[rowA][reg_i+2][reg_j], regC[rowA][reg_i+3][reg_j]);
                  case 2:
                    globalStore2Elems(&glC[cIdx], regC[rowA][reg_i][reg_j], regC[rowA][reg_i+1][reg_j]);
                  case 1:
                    globalStore1Elems(&glC[cIdx], regC[rowA][reg_i][reg_j]);
                }
              }
            }
          }
        }
      }
    }

    __syncthreads();
  }}}
}