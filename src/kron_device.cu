#include "hip/hip_runtime.h"
// #define C_IN_SHMEM
template<uint MaxKronCols, uint MaxTileSizeKronRows> __device__ uint get_tile_k() {return blockIdx.y/DIVUP(MaxKronCols, MaxTileSizeKronRows);}
template<uint MaxKronCols, uint MaxTileSizeKronRows> __device__ uint get_external_tile_kp_n() {return blockIdx.y%DIVUP(MaxKronCols, MaxTileSizeKronRows);}

__device__ bool isfirstIdx(dim3 idx) {return idx.x == 0 && idx.y == 0 & idx.z == 0;}

__device__ constexpr uint sqrt(uint x)
{
  switch (x) {
    case 1:
      return 1;

    case 2:
      return 2;
    
    case 4:
      return 2;
    
    case 8:
      return 4;
    
    case 16:
      return 4;
    
    case 32:
      return 8;
    
    case 64:
      return 8;
    
    default:
      return 1;
  }
}

template<typename VecT, typename ElemT>
__device__ void loadVecToRegs(VecT& vec, ElemT* regs) {
  //Not implemented
}

template<>
__device__ void loadVecToRegs(float4& vec, float* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}

template<>
__device__ void loadVecToRegs(int4& vec, int* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}


template<>
__device__ void loadVecToRegs(double4& vec, double* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
  regs[2] = vec.z;
  regs[3] = vec.w;
}

template<>
__device__ void loadVecToRegs(double2& vec, double* regs) {
  regs[0] = vec.x;
  regs[1] = vec.y;
}

template<>
__device__ void loadVecToRegs(float& vec, float* regs) {
  regs[0] = vec;
}

// __launch_bounds__(NumThreads)
template<typename ElemT, typename VecT, uint NumThreads, uint N_COARSE_TB, uint TileSizeRowsA, uint MaxColsA, uint MaxKronCols, uint MaxKronRows, uint KP_N_TILE_, uint K_EQUALS_VAR, uint KPK_EQUALS_VAR>
__global__ void kronGemmKernel(const uint RowsC,    const uint ColsC,   const uint ColsA,
                               const uint KronRows, const uint KronCols,
                               const ElemT * __restrict__ glA, 
                               const ElemT * __restrict__ glKronMats, 
                               ElemT       * __restrict__ glC,
                               const uint kp_idx) {
  
  const uint WarpSize     = 32;
  const uint tid          = threadIdx.x;
  const uint wid          = tid/WarpSize;
  const uint lane         = tid%WarpSize;
  const uint blockWarps   = blockDim.x/WarpSize;
  const uint VecTNumElems = (sizeof(VecT)/sizeof(ElemT));

  const uint MaxTileSizeKronRows = MIN(KP_N_TILE_,          MaxKronCols);
  const uint MaxTileSizeKronCols = MIN(EXTERNAL_KP_K_TILE_, MaxKronRows);
  const uint TileSizeKronRows    = MIN(128,                 MaxTileSizeKronRows);
  const uint TileSizeKronCols    = MIN(32,                  MaxTileSizeKronCols);
  const uint TileSizeColsA       = MaxColsA/(MaxKronRows/TileSizeKronCols);
  
  const uint CRegSize = MAX((MaxColsA/(MaxKronCols/MaxTileSizeKronRows))/NumThreads, 1);
  const uint CRegRows = MIN(8, MAX(sqrt(CRegSize), 1));
  const uint CRegCols = MIN(MaxKronRows, MIN(8, CRegSize/CRegRows));
  
  register   ElemT regC[TileSizeRowsA][CRegRows][CRegCols];
  __shared__ ElemT shA[TileSizeRowsA][TileSizeColsA];
  __shared__ ElemT shKronMats[TileSizeKronCols][TileSizeKronRows];

#ifndef EVAL
  __syncthreads();
  if (kp_idx == 0 && isfirstIdx(threadIdx) && isfirstIdx(blockIdx)) {
    printf("CRegRows %d CRegCols %d\n", CRegRows, CRegCols);
    // for (int i = 0; i < kronRows; i++) 
    //   for (int j = 0; j < kronCols; j++)
    //     printf("%lf \n", (double)shKronMats[i][j]);
  }
#endif

  // const uint NUM_INTERNAL_KP_N_TILES = MaxTileSizeKronRows/TileSizeKronRows;
  // assert(Creg_SIZE == CRegCols * CRegRows * NUM_INTERNAL_KP_N_TILES);
  uint kronCols;
  uint kronRows;
  uint colsA;
  uint colsC;
 
  if (KPK_EQUALS_VAR) {
    kronCols = MaxKronRows;
    kronRows = MaxKronCols;
  } else {
    kronCols = KronCols;
    kronRows = KronRows;
  }

  if (K_EQUALS_VAR) {
    colsA = MaxColsA;
    colsC = colsA;
  } else {
    colsA = ColsA;
    colsC = ColsC;
  }

  const uint KPK_SPLIT_SIZE = MIN(16, TileSizeKronCols);
  const uint NUM_KPK_SPLITS = MAX(1, TileSizeKronCols/KPK_SPLIT_SIZE);

  const uint external_tile_kp_k = blockIdx.z;

  const uint kp_col_start_ = (tid / ((MaxColsA/MaxKronRows)/CRegRows)) * CRegCols;
  const uint a_col_start_  = (tid % ((MaxColsA/MaxKronRows)/CRegRows)) * CRegRows; 

  if (MaxTileSizeKronRows == MaxKronCols && TileSizeKronRows == MaxKronCols && TileSizeKronCols == MaxKronRows) {
    const uint loadInstr = MIN(kronRows*kronCols, VecTNumElems);

    for (uint eIdx = tid*loadInstr; eIdx < kronRows*kronCols; eIdx += blockDim.x*loadInstr) {
      ElemT regElems[VecTNumElems];
      VecT vec;

      vec = *(VecT*)&glKronMats[eIdx];
      loadVecToRegs(vec, regElems);

      #pragma unroll
      for (uint vecElem = 0; vecElem < loadInstr; vecElem++) {
        uint idx = eIdx + vecElem;
        shKronMats[idx/MaxKronRows][idx%MaxKronRows] = regElems[vecElem];
      }
    }
  }

  for (uint tileRowA  = blockIdx.x * TileSizeRowsA;
            tileRowA  < gridDim.x  * TileSizeRowsA * N_COARSE_TB;
            tileRowA += gridDim.x  * TileSizeRowsA) {
  // if (tileRowA == 0 && tid == 0) {
  //   printf("CRegRows %d CRegCols %d\n", CRegRows, CRegCols);
  // }

  for (uint tileKronCol =  kp_col_start_;
            tileKronCol <  MaxTileSizeKronRows;
            tileKronCol += MAX(1, NumThreads/((MaxColsA/MaxKronRows)/CRegRows)) * CRegCols) {

  for (uint tileColA    =  a_col_start_ ;
            tileColA    <  MaxColsA/MaxKronRows;
            tileColA    += NumThreads * MAX(1, NumThreads/((MaxColsA/MaxKronRows)/CRegRows)) * CRegRows) {

    #pragma unroll
    for (uint r = 0; r < TileSizeRowsA; r++) {
    #pragma unroll
    for (uint i = 0; i < CRegRows;      i++) {
    #pragma unroll
    for (uint j = 0; j < CRegCols;      j++) {
      regC[r][i][j] = 0;
    }}}

    for (uint internal_tile_kp_k = 0; internal_tile_kp_k < MaxTileSizeKronCols; internal_tile_kp_k += TileSizeKronCols) {
      for (uint rowA = 0; rowA < TileSizeRowsA; rowA += 1) {
        for (uint a_col = tid*VecTNumElems; a_col < TileSizeColsA; a_col += blockDim.x*VecTNumElems) {
          uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronRows>();
          VecT a;
          if (TileSizeKronCols == MaxKronRows) {
            a = *(VecT*)&glA[(rowA + tileRowA) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MaxColsA) + a_col];
            // *(VecT*)&shA[rowA][a_col] = a;
            // ElemT a1[4] = {a.x, a.y, a.z, a.w};
            // for (int j = 0; j < VecTNumElems; j++) {
            //   shA[rowA][a_col + j] = a1[j];
            // }
          } else {
            a = *(VecT*)&glA[(rowA + tileRowA) * colsA + (K_EQUALS_VAR ? 0 : tile_k*MaxColsA) + \
                           (a_col/TileSizeKronCols)*kronCols + external_tile_kp_k * MaxTileSizeKronCols + internal_tile_kp_k + a_col % TileSizeKronCols];
            // *(VecT*)&shA[rowA][a_col] = a;
          }
          
          ElemT a1[VecTNumElems];
          loadVecToRegs(a, a1);

          #pragma unroll
          for (uint i = 0; i < VecTNumElems; i++) {
            uint ash_col = a_col + i;
            uint tileColA = (ash_col/TileSizeKronCols)/CRegRows;
           
            uint final_col = (ash_col/TileSizeKronCols)*TileSizeKronCols + (tileColA + ash_col%TileSizeKronCols)%TileSizeKronCols;
            shA[rowA][final_col] = a1[i];
          }
        }
      }
    
      //TODO: nvcc unrolls this loop, which leads to high register usage
      for (uint internal_tile_kp_n = 0; internal_tile_kp_n < MaxTileSizeKronRows; internal_tile_kp_n += TileSizeKronRows) {
        if (!(MaxTileSizeKronRows == MaxKronCols && TileSizeKronRows == MaxKronCols && TileSizeKronCols == MaxKronRows)) {
          //Create kronCols subwarps and each subwarp loads 0 to TileSizeKronRows elements
          const uint VecTNumElems = sizeof(VecT)/sizeof(ElemT);
          const uint ldSize = MIN(TileSizeKronRows, VecTNumElems);

          for (uint swid = tid/(TileSizeKronRows/ldSize); swid < TileSizeKronCols; swid += blockDim.x/(TileSizeKronRows/ldSize)) {
            uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronRows>();
            uint col = external_tile_kp_n*MaxTileSizeKronRows + internal_tile_kp_n + (tid%(TileSizeKronRows/ldSize))*ldSize;
            uint row = swid;
            // shKronMats[tid%TileSizeKronRows][row] = glKronMats[(external_tile_kp_k * MaxTileSizeKronCols + internal_tile_kp_k + row) * kronRows + col];
            VecT a = *(VecT*)&glKronMats[(external_tile_kp_k * MaxTileSizeKronCols + internal_tile_kp_k + row) * kronRows + col];
            ElemT a1[VecTNumElems];
            loadVecToRegs(a, a1);
            #pragma unroll
            for (uint i = 0; i < ldSize; i++) {
              uint idx = (tid%(TileSizeKronRows/ldSize))*ldSize + i%ldSize;
              shKronMats[row][idx] = a1[i];
            }
          }
        }

        __syncthreads();
        
        const uint MAX_AR_SZ = MIN(8, KPK_SPLIT_SIZE);

        //Load MAX_AR_SZ elements at a time to limit the register usage
        for (uint ar_start_id = 0; ar_start_id < TileSizeKronCols; ar_start_id += MAX_AR_SZ) {
          register ElemT Ar[TileSizeRowsA][CRegRows][MAX_AR_SZ];
          register ElemT KPr[MAX_AR_SZ][CRegCols];

          uint round_start = (tileColA / CRegRows)%TileSizeKronCols;

          #pragma unroll
          for (uint rowA = 0; rowA < TileSizeRowsA; rowA++) {
            #pragma unroll
            for (uint _a_col = 0; _a_col < CRegRows; _a_col++) {
              uint a_col = tileColA + _a_col;
              for (uint a_elem = 0; a_elem < MAX_AR_SZ; a_elem++)    
                Ar[rowA][_a_col][a_elem] = shA[rowA][a_col * TileSizeKronCols + (ar_start_id + a_elem + round_start)%TileSizeKronCols]; 
            }
          }
          
          #pragma unroll
          for (uint _kp_col = 0; _kp_col < CRegCols; _kp_col++) {
            uint kp_col = tileKronCol + _kp_col;
            for (uint elem = 0; elem < MAX_AR_SZ; elem++)    
              KPr[elem][_kp_col] = shKronMats[ar_start_id + elem][kp_col];
          }

          #pragma unroll
          for (uint rowA = 0; rowA < TileSizeRowsA; rowA++)
            #pragma unroll
            for (int i = 0; i < CRegRows; i++)
              #pragma unroll
              for (int j = 0; j < CRegCols; j++)
                #pragma unroll
                for (int k = 0; k < MAX_AR_SZ; k++)
                  regC[rowA][i][j] += Ar[rowA][i][k] * KPr[k][j];
        }
      }

      __syncthreads();
    }
    
    #pragma unroll 
    for (int rowA = 0; rowA < TileSizeRowsA; rowA++) {
      #pragma unroll 
      for (uint reg_j = 0; reg_j < CRegCols; reg_j++) {
        if (CRegRows % 4 == 0) {
          for (uint reg_i = 0; reg_i < CRegRows; reg_i += 4) {          
            const uint cRow = (rowA + tileRowA);
            uint cCol = tileKronCol*(MaxColsA/MaxKronRows) + reg_j*(MaxColsA/MaxKronRows) + tileColA + reg_i;
            if (!K_EQUALS_VAR) {
              uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronRows>();
              cCol = tile_k * (MaxColsA/kronCols) + 
                  (cCol/(MaxColsA/kronCols)) * (colsA/kronCols) +
                  cCol%(MaxColsA/kronCols);
            }
            if (MaxTileSizeKronRows != MaxKronCols) {
              uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronRows>();
              cCol += external_tile_kp_n*(colsA/(MaxKronCols/MaxTileSizeKronRows)); 
            }
            const uint cIdx = cRow * colsC + cCol;
            // assert(tid == cCol);
            // if (kp_idx == 0&& cRow == 0 && cCol < 64)
            //   printf("tid %d cCol %d tileKronCol %d tileColA %d reg_i %d reg_j %d\n", tid, cCol, tileKronCol, tileColA, reg_i, reg_j);
            if (cCol < colsA) {
              VecT c = {regC[rowA][reg_i][reg_j], regC[rowA][reg_i+1][reg_j], regC[rowA][reg_i+2][reg_j], regC[rowA][reg_i+3][reg_j]};
              *(VecT*)&glC[cIdx] = c;
            }
          }
        } else {
          for (uint reg_i = 0; reg_i < CRegRows; reg_i++) {            
            const uint cRow = (rowA + tileRowA);
            uint cCol = tileKronCol*(MaxColsA/MaxKronRows) + reg_j*(MaxColsA/MaxKronRows) + tileColA + reg_i;
            
            if (!K_EQUALS_VAR) {
              uint tile_k = get_tile_k<MaxKronCols, MaxTileSizeKronRows>();
              cCol = tile_k * (MaxColsA/kronCols) + 
                  (cCol/(MaxColsA/kronCols)) * (colsA/kronCols) +
                  cCol%(MaxColsA/kronCols);
            }
            if (MaxTileSizeKronRows != MaxKronCols) {
              uint external_tile_kp_n = get_external_tile_kp_n<MaxKronCols, MaxTileSizeKronRows>();
              cCol += external_tile_kp_n*(colsA/(MaxKronCols/MaxTileSizeKronRows)); 
            }
            const uint cIdx = cRow * colsC + cCol;
            // assert(tid == cCol);
            // if (kp_idx == 0&& cRow == 0 && cCol < 64)
            //   printf("tid %d cCol %d tileKronCol %d tileColA %d reg_i %d reg_j %d\n", tid, cCol, tileKronCol, tileColA, reg_i, reg_j);
            if (cCol < colsA) {
              glC[cIdx] = regC[rowA][reg_i][reg_j];
            }
          }
        }
      }
    }

    __syncthreads();
  }}}
}