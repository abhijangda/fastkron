#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

  This example has delibrately been kept similar to the basic_gemm example from cutass-1.3 to 
  highlight the minimum amount of differences needed to transition to cutlass-2.0.

  Cutlass-1.3 sgemm: https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/krongemm.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////


#include <iostream>
#include <string>
#include <cstdlib>
#include <cassert>
#include <vector>

void setMatrix(int* mat, int M, int N, int (*fnvalue)(int i, int j)) 
{
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      mat[i*N + j] = fnvalue(i,j);
    }
  }
}

void printMatrix(int* mat, int M, int N) 
{
  printf("[");
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      // if (mat[i*N + j] == 18496)
        // printf("%d,%d\n",i,j);
      printf("%d, ", mat[i*N + j]);
    }
    if (i < M-1)
      printf("\n");
  }
  printf("]");
}

void baselineKPThenMatmul(int NUM_KP_MATS, int* result, int* x, int* kpout[], int* kpMats[],
                          int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int cols;
  int rows;

  for (int kp = 0; kp < NUM_KP_MATS - 1; kp++) {
    int* kpFirst = (kp == 0) ? kpMats[0] : kpout[kp - 1];
    int kpFirstRows = (kp == 0) ? KP_MAT_K[0] : rows;
    int kpFirstCols = (kp == 0) ? KP_MAT_N[0] : cols;

    cols = kpFirstCols * KP_MAT_N[kp+1];
    rows = kpFirstRows * KP_MAT_K[kp+1];
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        int v2 = kpMats[kp+1][(i%KP_MAT_K[kp+1]) * KP_MAT_N[kp+1] + j%KP_MAT_N[kp+1]];
        int v1 = kpFirst[(i/KP_MAT_K[kp+1]) * kpFirstCols + j/KP_MAT_N[kp+1]];
        kpout[kp][i*cols + j] = v1 * v2;
      }
    }
  }

  for(int i = 0; i < M; i++) {    
    for(int j = 0; j < N; j++) {    
      result[i* N + j] = 0;    
      for(int k = 0; k < K; k++) {   
        result[i * N + j] += x[i*K + k]*kpout[NUM_KP_MATS-2][k*N + j];
      }    
    }    
  }
}

/**
 * 
*/
void slicedMatmul(int NUM_KP_MATS, int* kpMatmulResult[], int* x, int* kpMats[],
                  int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int secFacRowMulSize = 1;
  int rowsTillNow = 1;
  int colsTillNow = 1;
  int resultCols;
  for (int kp = 0; kp < NUM_KP_MATS; kp++) {
    int* prevKPMatmul = (kp == 0) ? x : kpMatmulResult[kp - 1];
    int kpSecondK = KP_MAT_K[NUM_KP_MATS - 1 - kp];
    int kpSecondN = KP_MAT_N[NUM_KP_MATS - 1 - kp];
    int prevKPMatmulCols = (kp == 0) ? K : resultCols;

    resultCols = (prevKPMatmulCols/kpSecondK) * kpSecondN;
    secFacRowMulSize = (kp == 0) ? K/kpSecondK : rowsTillNow * K/(colsTillNow * KP_MAT_K[NUM_KP_MATS - 1 - (kp)]);

    //Number of times a column is multiplied with input matrix is equal to 
    //N/(number of column elements of this matrix * cols so far) * number of rows so far.

    rowsTillNow *= KP_MAT_N[NUM_KP_MATS - 1 - (kp)];
    colsTillNow *= KP_MAT_K[NUM_KP_MATS - 1 - (kp)];

    for (int i = 0; i < M; i++) {
      for (int j = 0; j < resultCols; j++) {
        int r = 0;

        for (int kp_k = 0; kp_k < kpSecondK; kp_k++) {
          int slice = (j / secFacRowMulSize) % kpSecondN;

          int v2 = kpMats[NUM_KP_MATS - 1 - kp][kp_k*kpSecondN + slice];
          
          r += prevKPMatmul[i* prevKPMatmulCols + (j*kpSecondK)%prevKPMatmulCols + kp_k] * v2;
        }

        kpMatmulResult[kp][i*resultCols + j] = r;
      }
    }
  }
}

void cutlassKronGEMM(int NUM_KP_MATS, int* kpMatmulResult[], int* x, int* kpMats[],
                     int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  using RowMajor = cutlass::layout::RowMajor;

using MMAOp = cutlass::arch::OpClassSimt;



// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm61;
  using CutlassKronGemm = cutlass::gemm::device::KronGemm<int,        // Data-type of A matrix
                                                  RowMajor,  // Layout of A matrix
                                                  int,        // Data-type of Kron Factors matrix
                                                  RowMajor,
                                                  int,
                                                  RowMajor,
                                                  int,
                                                  MMAOp,
                                                  SmArch,
                                                  cutlass::gemm::GemmShape<32, 32, 8>,
    cutlass::gemm::GemmShape<32, 32, 8>,
    cutlass::gemm::GemmShape<1, 1, 1>>; // Layout of Kron Factors matrix

  CutlassKronGemm gemm_operator;
  
  CutlassKronGemm::TensorRefB b_krons[1];
  
  // printf("tensor_ref_x.stride() %d\n", tensor_ref_x.stride(0));
  for (int i = 0; i < NUM_KP_MATS; i++) {
    b_krons[0] = {kpMats[NUM_KP_MATS-i-1], KP_MAT_N[NUM_KP_MATS-i-1]};
    printf("kpMats[NUM_KP_MATS-i-1] %p\n", kpMats[NUM_KP_MATS-i-1]);
    CutlassKronGemm::TensorRefA tensor_ref_x = {(i==0) ? x : kpMatmulResult[i-1], M};

    CutlassKronGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    tensor_ref_x,    // Tensor-ref for source matrix A
                                    b_krons,
                                    1,
                                    KP_MAT_N, KP_MAT_K,
                                    {kpMatmulResult[i], M},
                                    {kpMatmulResult[i], M}); // Kron Factors

    //
    // Launch the CUTLASS GEMM kernel.
    //
    
    cutlass::Status status = gemm_operator(args);

    //
    // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
    //

    if (status != cutlass::Status::kSuccess) {
      printf("cutlass error\n")  ;
    }

    printf("cutlass succesfull\n");
    // Return success, if no errors were encountered.
  }
}


template<typename T, int TILE_Y, int TILE_X, int KP_N, int KP_K>
__global__ 
void cuda_gemm(int M, int N, int K, T * A, T * kron_fac, T * C) {
  /*Each threadblock compute TILE_X x KP_N of C*/

  //Each threadblock loads the KP_K x TILE_Y kron_fac into shared memory, loads every TILE_X x KP_K sub-matrix of A into shared memory,
  //multiplies each sub-matrix with every column of kron_fac, and stores the results.

  //TODO: For now TILE_Y = 1;

  __shared__ int kron_fac_sh[KP_K][TILE_Y];
  __shared__ int As[TILE_X][KP_K];
  __shared__ int Csh[TILE_X][KP_K];

  for (auto i = threadIdx.x; i < KP_K * TILE_Y; i += blockDim.x) {
    kron_fac_sh[i/TILE_Y][i%TILE_Y] = kron_fac[(i/TILE_Y) * KP_N + blockIdx.y *TILE_Y+ (i%TILE_Y)];
  }

  __syncthreads();

  int start_row = blockIdx.x * TILE_X;
  for (int a_col_batch = 0; a_col_batch < K; a_col_batch += KP_K)  {
    for (int a_row = threadIdx.x; a_row < TILE_X; a_row += blockDim.x) {
      for (int a_col = 0; a_col < KP_K; a_col++) {
        int a = A[(a_row + start_row) * K + (a_col_batch + a_col)];
        As[a_row][a_col] = a;
      }
    }
    __syncthreads();

    for (int tile_y = 0; tile_y < TILE_Y; tile_y++) {
      for (int a_row = threadIdx.x; a_row < TILE_X; a_row += blockDim.x) {
        int c = 0;

        for (int a_col = 0; a_col < KP_K; a_col++) {
          int a = As[a_row][a_col];
          int kp = kron_fac_sh[a_col][tile_y];// kron_fac[a_col * KP_K + blockIdx.y];
          // printf("%d: (%d x %d)\n", threadIdx.x, (a_row + start_row) * K + (a_col_batch + a_col), a_col * KP_K + blockIdx.y);
          c += a * kp;
        }


        Csh[a_row][a_col_batch/KP_K] = c;
      }

      __syncthreads();

      for (int a_row = threadIdx.x; a_row < TILE_X; a_row += blockDim.x) {
        int c_row = (a_row + start_row);
        int c_col = ((blockIdx.y * TILE_Y + tile_y) * KP_K + a_col_batch/KP_K);
        int c_idx = c_row * N + c_col;

        C[c_idx] = Csh[a_row][a_col_batch/KP_K];
      }
    }
  }
}

void customKronGEMM(int NUM_KP_MATS, int* kpMatmulResult[], int* x, int* kpMats[],
                     int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  //Row Major Layout of all matrics
  for (int i = 0; i < NUM_KP_MATS; i++) {
    int* prev_kp = (i==0) ? x : kpMatmulResult[i-1];
    
    const int TILE_Y = 32; //Y direction corresponds to tile of column of the KP factor
    const int TILE_X = 128; //X direction correspond to tile of row 

    dim3 grid = {M/TILE_X, (N/KP_MAT_N[NUM_KP_MATS-i-1])/TILE_Y}; 
    dim3 block = {128,1,1};
    cuda_gemm<int,TILE_Y,TILE_X,32,32><<<grid, block>>>(M, N, K, prev_kp, kpMats[NUM_KP_MATS-i-1], kpMatmulResult[i]);

    // CUDACHECK(hipDeviceSynchronize());
  }
}

bool check(int* ref, int* computed, int M, int N) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      if (ref[i*N + j] != computed[i* N + j]) {
        printf("Mismatch for %d x %d at (%d, %d): ref = %d, computed = %d\n", M, N, i, j, ref[i*N+j], computed[i*N+j]);
        return false;
      }
    }
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int one(int i, int j) {return 1;}
int zeroOne(int i, int j) {return i % 2;}
int setToI(int i, int j) {return i;}
int randMod(int i, int j) {return rand()%10;}

void setValues(int NUM_KP_MATS, int* kpMats[], int *x, int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[], int (*fnvalue)(int i, int j))
{
  for (int i = 0; i < NUM_KP_MATS; i++) {
    setMatrix(kpMats[i], KP_MAT_K[i], KP_MAT_N[i], fnvalue);
  }

  setMatrix(x, M, K, fnvalue);
}

struct MatrixSizes {
  const int M, N, K;
  const int NUM_KP_MATS;
  const std::vector<int> KP_MAT_N; 
  const std::vector<int> KP_MAT_K;
};

int main(int argc, char* argv[]) 
{
  std::vector<MatrixSizes> matrixSizes = {
                                          // {4,4,4, 2, {2,2},{2,2}},
                                          // {4,4,6, 2, {1,4},{2,3}},
                                          // {4,4,8, 2, {2,2},{2,4}},
                                          // {4,4,8, 2, {2,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{2,4}},
                                          // {8,8,8, 3, {2,2,2},{2,2,2}},
                                          // {8,8,32, 3, {2,2,2},{2,4,4}},
                                          // {8,16,32, 3, {4,2,2},{2,4,4}},
                                          // {8,8,16, 3, {2,2,2},{2,4,2}},
                                          // {16,8,8, 3, {2,2,2},{2,2,2}},
                                          // {16,16,16, 2, {4,4},{4,4}},
                                          // {16,16,16, 3, {4,2,2},{4,2,2}},
                                          // {16,16,16, 3, {4,2,2},{2,4,2}},
                                          // {16,16,16, 3, {8,2,1},{2,4,2}},
                                          // {16,16,16, 4, {2,2,2,2},{2,2,2,2}},
                                          // {16,16,64, 4, {2,2,2,2},{2,4,2,4}},
                                          // {256,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {256,256,256, 2, {16,16},{16,16}},
  #ifdef EVAL
                                          {65536,1024,1024, 2, {32,32},{32,32}},
  #else
                                          {512,1024,1024, 2, {32,32},{32,32}},
  #endif

                                          // {1024, 1024, 1024, 2, {32,32},{32,32}}
                                          };

  // int (*fnvalues[4])(int, int) = {&one, &zeroOne, &setToI, &randMod};
  int (*fnvalues[1])(int, int) = {&randMod};

  for (MatrixSizes matrixSize : matrixSizes) {
    int M = matrixSize.M;
    int N = matrixSize.N;
    int K = matrixSize.K;
    
    int NUM_KP_MATS = matrixSize.NUM_KP_MATS;
    int KP_MAT_N[NUM_KP_MATS];
    int KP_MAT_K[NUM_KP_MATS];

    printf("Matmul: %d x %d x %d, Num KP Factors: %d\n", M, N, K, NUM_KP_MATS);
    int n=1,k=1;
    for (int i = 0; i < NUM_KP_MATS; i++) {
      k *= matrixSize.KP_MAT_K[i];
      n *= matrixSize.KP_MAT_N[i];
    }
    if (n != N || k != K) {
      printf("Invalid KP Factors Sizes %d != %d, %d != %d\n", n, N, k, K);
    }

    int *kpout[NUM_KP_MATS];
    int *kpMats[NUM_KP_MATS];
    int* kpMatmulResult[NUM_KP_MATS];

    int *x = new int[M*K];

    int* dX;
    int** dKpOut;
    int** dKpMats;
    int** dKpMatmulResult;
    
    CUDACHECK(hipMalloc(&dX, M*K * sizeof(int)));
    CUDACHECK(hipMalloc(&dKpMats, NUM_KP_MATS * sizeof(int*)));
    CUDACHECK(hipMalloc(&dKpMatmulResult, NUM_KP_MATS * sizeof(int*)));
    CUDACHECK(hipMalloc(&dKpOut, NUM_KP_MATS * sizeof(int*)));

    int* __dKpOut[NUM_KP_MATS];
    int* __dKpMats[NUM_KP_MATS];
    int* __dKpMatmulResult[NUM_KP_MATS];

    for (int i = 0; i < NUM_KP_MATS; i++) {
      KP_MAT_K[i] = matrixSize.KP_MAT_K[i];
      KP_MAT_N[i] = matrixSize.KP_MAT_N[i];
      kpMats[i] = new int[KP_MAT_K[i] * KP_MAT_N[i]];
      kpout[i] = new int[K*N]; //TODO: larger than needed
      kpMatmulResult[i] = new int[M*std::max(N,K)];

      CUDACHECK(hipMalloc(&__dKpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(int)));
      // CUDACHECK(hipMalloc(&__dKpOut[i], K * N * sizeof(int)));
      CUDACHECK(hipMalloc(&__dKpMatmulResult[i], M*std::max(N,K) * sizeof(int)));

      CUDACHECK(hipMemset(__dKpMatmulResult[i], 0, M*std::max(N,K) * sizeof(int)));
      // CUDACHECK(hipMemset(__dKpOut[i], 0, K * N * sizeof(int)));
    }

    // CUDACHECK(hipMemcpy(&dKpOut[0], &__dKpOut[0], NUM_KP_MATS * sizeof(int*), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(&dKpMats[0], &__dKpMats[0], NUM_KP_MATS * sizeof(int*), hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(&dKpMatmulResult[0], &__dKpMatmulResult[0], NUM_KP_MATS * sizeof(int*), hipMemcpyHostToDevice));

    int* result = new int[M*N];

    int* dResult;

    CUDACHECK(hipMalloc(&dResult, M * N * sizeof(int)));

    for (int fnvalue = 0; fnvalue < sizeof(fnvalues)/sizeof(fnvalues[0]); fnvalue++) {
      setValues(NUM_KP_MATS, kpMats, x, M, N, K, KP_MAT_N, KP_MAT_K, fnvalues[fnvalue]);

      for (int i = 0; i < NUM_KP_MATS; i++) {
        CUDACHECK(hipMemcpy(__dKpMats[i], kpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(int), hipMemcpyHostToDevice));
      }
    
      CUDACHECK(hipMemcpy(dX, x, M * K * sizeof(int), hipMemcpyHostToDevice));
  #ifndef EVAL
      baselineKPThenMatmul(NUM_KP_MATS, result, x, kpout, kpMats, 
                           M, N, K, KP_MAT_N, KP_MAT_K);
  #endif
      // slicedMatmul(NUM_KP_MATS, kpMatmulResult, x, kpMats,
      //              M, N, K, KP_MAT_N, KP_MAT_K);

      for (int i = 0; i < NUM_KP_MATS; i++)
        CUDACHECK(hipMemset(__dKpMatmulResult[i], 0, M*std::max(N,K) * sizeof(int)));
  #ifdef EVAL
      for (int i = 0; i < 100; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K);
      CUDACHECK(hipDeviceSynchronize());
      return;
  #else
      for (int i = 0; i < 1; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K);
  #endif
      CUDACHECK(hipDeviceSynchronize());
      // return;
      int* hKpMatMulResult = new int[M*N];
      // return;
      for (int i = 0; i < NUM_KP_MATS; i++)
        CUDACHECK(hipMemcpy(kpMatmulResult[i], __dKpMatmulResult[i], M*N*sizeof(int), hipMemcpyDeviceToHost));
      // if (check(result, kpMatmulResult[NUM_KP_MATS-1], M, N))
      if (check(result, kpMatmulResult[NUM_KP_MATS-1], M,N))
        printf("Results Correct for test %d\n", fnvalue);
      else {
        // printf("\nMatmul:");
        // printMatrix(result, K, N);

        // printf("\nx:");
        // printMatrix(x, M, K);    
        // for (int kpMatId = 0; kpMatId < NUM_KP_MATS; kpMatId++) {
        //   printf("\nKP Mat %d:", kpMatId);
        //   printMatrix(kpMats[kpMatId], KP_MAT_K[kpMatId], KP_MAT_N[kpMatId]);
        // }
        // // printf("\nKP Out:");
        // // printMatrix(kpout[0], 8, 8);
        // for (int id = 0; id < NUM_KP_MATS; id++) {
        //   printf("\nKP result %d:", id);
        //   printMatrix(kpMatmulResult[id], M, N);
        // }
        // printf("\nKP result 2:");
        // printMatrix(kpMatmulResult[2], 16, 16);
        // printf("\nKP result 3:");
        // printMatrix(kpMatmulResult[3], 16, 16);
        // printf("\nKP result 1:");
        // printMatrix(kpMatmulResult[1], M, N);
        // printf("\n");
        return 0;
      }
    }

    //Is there really a need to free anything when you have tons of RAM, am I right?
  }

  return 0;
}