#include "hip/hip_runtime.h"

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#include <iostream>
#include <string>
#include <cstdlib>
#include <cassert>
#include <vector>

#define MIN(x,y) (((x) < (y)) ? (x) : (y))
#define MAX(x,y) (((x) > (y)) ? (x) : (y))
#define DIVUP(x, y) (((x) + (y) - 1)/((y)))

template<typename T>
void setMatrix(T* mat, int M, int N, int (*fnvalue)(int i, int j)) 
{
  // #pragma omp parallel for collapse(2)
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      mat[i*N + j] = (T)fnvalue(i,j);
    }
  }
}

void printMatrix(int* mat, int M, int N) 
{
  printf("[");
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      // if (mat[i*N + j] == 18496)
        // printf("%d,%d\n",i,j);
      printf("%d, ", mat[i*N + j]);
    }
    if (i < M-1)
      printf("\n");
  }
  printf("]");
}

void baselineKPThenMatmul(int NUM_KP_MATS, int* result, int* x, int* kpout[], int* kpMats[],
                          int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int cols;
  int rows;

  for (int kp = 0; kp < NUM_KP_MATS - 1; kp++) {
    int* kpFirst = (kp == 0) ? kpMats[0] : kpout[kp - 1];
    int kpFirstRows = (kp == 0) ? KP_MAT_K[0] : rows;
    int kpFirstCols = (kp == 0) ? KP_MAT_N[0] : cols;

    cols = kpFirstCols * KP_MAT_N[kp+1];
    rows = kpFirstRows * KP_MAT_K[kp+1];
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        int v2 = kpMats[kp+1][(i%KP_MAT_K[kp+1]) * KP_MAT_N[kp+1] + j%KP_MAT_N[kp+1]];
        int v1 = kpFirst[(i/KP_MAT_K[kp+1]) * kpFirstCols + j/KP_MAT_N[kp+1]];
        kpout[kp][i*cols + j] = v1 * v2;
      }
    }
  }

  for(int i = 0; i < M; i++) {    
    for(int j = 0; j < N; j++) {    
      result[i* N + j] = 0;    
      for(int k = 0; k < K; k++) {   
        result[i * N + j] += x[i*K + k]*kpout[NUM_KP_MATS-2][k*N + j];
      }    
    }    
  }
}

/**
 * 
*/
void slicedMatmul(int NUM_KP_MATS, int* kpMatmulResult[], int* x, int* kpMats[],
                  int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int secFacRowMulSize = 1;
  int rowsTillNow = 1;
  int colsTillNow = 1;
  int resultCols;
  for (int kp = 0; kp < NUM_KP_MATS; kp++) {
    int* prevKPMatmul = (kp == 0) ? x : kpMatmulResult[kp - 1];
    int kpSecondK = KP_MAT_K[NUM_KP_MATS - 1 - kp];
    int kpSecondN = KP_MAT_N[NUM_KP_MATS - 1 - kp];
    int prevKPMatmulCols = (kp == 0) ? K : resultCols;

    resultCols = (prevKPMatmulCols/kpSecondK) * kpSecondN;
    secFacRowMulSize = (kp == 0) ? K/kpSecondK : rowsTillNow * K/(colsTillNow * KP_MAT_K[NUM_KP_MATS - 1 - (kp)]);

    //Number of times a column is multiplied with input matrix is equal to 
    //N/(number of column elements of this matrix * cols so far) * number of rows so far.

    rowsTillNow *= KP_MAT_N[NUM_KP_MATS - 1 - (kp)];
    colsTillNow *= KP_MAT_K[NUM_KP_MATS - 1 - (kp)];

    for (int i = 0; i < M; i++) {
      for (int j = 0; j < resultCols; j++) {
        int r = 0;

        for (int kp_k = 0; kp_k < kpSecondK; kp_k++) {
          int slice = (j / secFacRowMulSize) % kpSecondN;

          int v2 = kpMats[NUM_KP_MATS - 1 - kp][kp_k*kpSecondN + slice];
          
          r += prevKPMatmul[i* prevKPMatmulCols + (j*kpSecondK)%prevKPMatmulCols + kp_k] * v2;
        }

        kpMatmulResult[kp][i*resultCols + j] = r;
      }
    }
  }
}

#define EXTERNAL_KP_K_TILE_ 64

__device__ bool isfirstIdx(dim3 idx) {return idx.x == 0 && idx.y == 0 & idx.z == 0;}

template<typename T,int N_THREADS, int N_COARSE_TB, int TILE_X, int MAX_K, int MAX_KP_N, int MAX_KP_K, int KP_N_TILE_, int K_EQUALS_VAR, int KPK_EQUALS_VAR>
__global__ void __launch_bounds__(N_THREADS) cuda_gemm(int M, int NVar, int KVar, T * A, T * kron_fac, T * C, int kpNVar, int kpKVar, int kp_idx) {
  const int KP_N_TILE = MIN(KP_N_TILE_, MAX_KP_N);
  const int NUM_KP_N_TILES = MAX_KP_N/KP_N_TILE;
  const int INTERNAL_KP_N_TILE = MIN(16, KP_N_TILE);
  const int EXTERNAL_KP_K_TILE = MIN(EXTERNAL_KP_K_TILE_, MAX_KP_K);
  const int INTERNAL_KP_K_TILE = MIN(32, EXTERNAL_KP_K_TILE);

  #ifdef EVAL
    typedef float4 LD_TYPE; 
  #else 
    typedef int4 LD_TYPE; 
  #endif 

  __shared__ __align__(128) T kron_fac_sh[INTERNAL_KP_N_TILE][INTERNAL_KP_K_TILE+1];//TODO: Change padding based on value o1, KP_K and TILE_Y
  const int Ash_COLS = MAX_K/(MAX_KP_K/INTERNAL_KP_K_TILE);
  __shared__ __align__(128) T Ash[TILE_X][Ash_COLS];
  const int C_ELEMS_STORE = N_THREADS * (sizeof(LD_TYPE)/sizeof(T));
  const int Csh_COLS = MAX_K/(MAX_KP_N/KP_N_TILE);
  const int Csh_COLS_SIZE = MIN(Csh_COLS, C_ELEMS_STORE);
  __shared__ __align__(128) T Csh[TILE_X][Csh_COLS];//Allocate Csh for only as many values that are produced

  int wid = threadIdx.x/32;
  int lane = threadIdx.x%32;
  int blockWarps = blockDim.x/32;
  int kpK;
  int kpN;
  int K;
  int N;
 
  if (KPK_EQUALS_VAR) {
    kpK = MAX_KP_K;
    kpN = MAX_KP_N;
  } else {
    kpK = kpKVar;
    kpN = kpNVar;
  }

  if (K_EQUALS_VAR) {
    K = MAX_K;
    N = K;
  } else {
    K = KVar;
    N = NVar;
  }

  const int KPK_SPLIT_SIZE = MIN(16, INTERNAL_KP_K_TILE);
  const int NUM_KPK_SPLITS = MAX(1, INTERNAL_KP_K_TILE/KPK_SPLIT_SIZE);

  int external_tile_kp_n;
  int tile_k;
  int external_tile_kp_k = blockIdx.z;
  
  if (KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K) {
    for (auto i = threadIdx.x; i < kpN * kpK; i += blockDim.x) {
      kron_fac_sh[i%kpN][i/kpK] = kron_fac[i];
    }

    external_tile_kp_n = 0;
    tile_k = blockIdx.x;
  } else {
    external_tile_kp_n = blockIdx.x%DIVUP(MAX_KP_N, KP_N_TILE);
    tile_k = blockIdx.x/DIVUP(MAX_KP_N, KP_N_TILE);
    // if (K/MAX_K == 32)  {
    //   external_tile_kp_n = blockIdx.x / 32;
    //   tile_k = blockIdx.x % 32;
    // } else if (K/MAX_K == 64) {
    //   external_tile_kp_n = blockIdx.x / 64;
    //   tile_k = blockIdx.x % 64;
    // } else {
    //   //TODO: Optimize by making more special cases
    //   external_tile_kp_n = blockIdx.x/(K/MAX_K);
    //   tile_k = blockIdx.x % (K/MAX_K);
    // }
  }

  const int ldNumElems = (sizeof(LD_TYPE)/sizeof(T));
  
  const int numKpColMult = MIN(MAX_K/kpK*NUM_KPK_SPLITS, N_THREADS); //Threads executing in parallel to multiply one column of KP with MAX_K row elements of A //128, 16, 128
  const int kpMulblockWarps = N_THREADS/numKpColMult; //1 , 8, 1
  const int Creg_SIZE = MAX(1, Csh_COLS/N_THREADS); // 8 , 2, 16
  const int Creg_Rows = (MAX_K/kpK)/numKpColMult; //1024/4/128 = 2, 1, 4
  const int Creg_Cols = INTERNAL_KP_N_TILE/kpMulblockWarps; //4/1 = 4, 2, 4

  register T Creg[Creg_SIZE];

  int kpSplitLane = threadIdx.x%NUM_KPK_SPLITS;
  int kpMullane = (threadIdx.x%numKpColMult)/NUM_KPK_SPLITS;
  int kpMulwid = threadIdx.x/numKpColMult; //0
   //TODO: Names should be different

  for (int start_row = blockIdx.y * TILE_X; start_row < gridDim.y * TILE_X * N_COARSE_TB; start_row += gridDim.y * TILE_X) {
    if (NUM_KPK_SPLITS != 1) {
      for (int a_row = 0; a_row < TILE_X; a_row += 1) {
        for (int i = threadIdx.x; i < Csh_COLS; i += blockDim.x)
          Csh[a_row][i] = 0;
      }
    } else {
      #pragma unroll
      for (int reg = 0; reg < Creg_SIZE; reg++) {
        Creg[reg] = 0;
      }
    }

    for (int internal_tile_kp_k = 0; internal_tile_kp_k < EXTERNAL_KP_K_TILE; internal_tile_kp_k += INTERNAL_KP_K_TILE) {
      for (int a_row = 0; a_row < TILE_X; a_row += 1) {
        for (int a_col = threadIdx.x*ldNumElems; a_col < Ash_COLS; a_col += blockDim.x*ldNumElems) {
          if (INTERNAL_KP_K_TILE == MAX_KP_K) {
            LD_TYPE a = *(LD_TYPE*)&A[(a_row + start_row) * K + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + a_col];

            *(LD_TYPE*)&Ash[a_row][a_col] = a;
          } else {
            LD_TYPE a = *(LD_TYPE*)&A[(a_row + start_row) * K + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + \
                                      (a_col/INTERNAL_KP_K_TILE)*kpK + external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + a_col % INTERNAL_KP_K_TILE];
            
            *(LD_TYPE*)&Ash[a_row][a_col] = a;
          }
        }      
      }
    
      //TODO: nvcc unrolls this loop, which leads to high register usage
      for (int internal_tile_kp_n = 0; internal_tile_kp_n < KP_N_TILE; internal_tile_kp_n += INTERNAL_KP_N_TILE) {
        if (!(KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K)) {
          //Create kpK subwarps and each subwarp loads 0 to INTERNAL_KP_N_TILE elements
    
          for (int swid = threadIdx.x/INTERNAL_KP_N_TILE; swid < INTERNAL_KP_K_TILE; swid += blockDim.x/INTERNAL_KP_N_TILE) {
            int col = external_tile_kp_n*KP_N_TILE + internal_tile_kp_n + threadIdx.x%INTERNAL_KP_N_TILE;
            int row = swid;
            kron_fac_sh[threadIdx.x%INTERNAL_KP_N_TILE][row] = kron_fac[(external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + row) * kpN + col];
          }
        }

        __syncthreads();

        for (int a_row = 0; a_row < TILE_X; a_row++) {
          for (int a_col_start = 0, c_reg_col_start = 0; a_col_start < MAX_K/kpK; a_col_start += numKpColMult, c_reg_col_start++) {
            const int MAX_AR_SZ = MIN(KPK_SPLIT_SIZE, MAX_KP_K);

            //Load MAX_AR_SZ elements at a time to limit the register usage
            // for (int ar_start = 0; ar_start < MAX_KP_K; ar_start += MAX_AR_SZ) 
            {
              register T Ar[MAX_AR_SZ];
              int kpKlane = kpMullane % MAX_AR_SZ;
              int ar_start = kpSplitLane * MAX_AR_SZ;

              for (int a_col = kpKlane, i = 0; i < MAX_AR_SZ; i++) { //
                if (INTERNAL_KP_K_TILE == MAX_KP_K) {
                  if (i < kpK) {
                    Ar[i] = Ash[a_row][(a_col_start+kpMullane)*kpK + ar_start + (a_col + i) % MAX_AR_SZ];//TODO: Shared memory bank conflicts here with KP_K = 4
                  }
                } else {
                  Ar[i] = Ash[a_row][(a_col_start+kpMullane)*INTERNAL_KP_K_TILE + ar_start + (a_col + i) % MAX_AR_SZ];//TODO: Shared memory bank conflicts here with KP_K = 4
                }
              }
              
              for (int kp_col = kpMulwid, creg_idx = 0; kp_col < min(kpN, INTERNAL_KP_N_TILE); kp_col += kpMulblockWarps, creg_idx++) {
                T c = 0;

                register T kron_fac_r;

                kron_fac_r = kron_fac_sh[kp_col][lane % INTERNAL_KP_K_TILE];
                
                #pragma unroll
                for (int a_col = 0; a_col < MIN(MAX_KP_K, MAX_AR_SZ); a_col++) {
                  if (a_col < kpK) {
                    T a = Ar[a_col]; //Ash[a_row][a_col_start/KP_K][a_col]; //Ar[a_col];
                    int kp_row;
                    if (KPK_EQUALS_VAR) {
                      kp_row = ar_start + (a_col + kpKlane)%min(kpK, KPK_SPLIT_SIZE); //kpMullane/(warpSize/kpK)
                    } else {kp_row = (a_col+kpKlane) < kpK ? (a_col+kpKlane) : (a_col+kpKlane) - kpK;} //TODO:
                    T kp;
                    if (INTERNAL_KP_K_TILE <= 32 && kpK <= 64) {
                      // kp = kron_fac_sh[kp_col][ar_start+(a_col+kpKlane)%min(kpK, KPK_SPLIT_SIZE)];
                      kp = __shfl_sync(0xffffffff, kron_fac_r, kp_row, INTERNAL_KP_K_TILE);
                      // if (kp_col == 0 && ar_start == 16 && kpK == 128 && kp != kp1 && isfirstIdx(blockIdx))
                      //   printf("kp_col %d kp_row %d %d, %d %d, %d %d %d\n", kp_col, kp_row, ar_start + (a_col+kpKlane) % min(MAX_AR_SZ, kpK), kp, kp1, ar_start, a_col, kpKlane);
                    } else {
                      //FIXME: For 1x16384 with 128x128 Kronecker factors, the results are incorrect for __shfl_sync because numkpcolmult != 32
                      // kp_row = ar_start + kpKlane + (a_col+kpKlane < min(MAX_AR_SZ, kpK) ? a_col : a_col - min(MAX_AR_SZ, kpK));
                      kp_row = ar_start + (a_col+kpKlane) % min(MAX_AR_SZ, kpK);
                      kp = kron_fac_sh[kp_col][kp_row];
                      // if (a_row == 0 && kp_col == 0 && kpMullane == 0 && isfirstIdx(blockIdx))
                      //   printf("kpSplitLane %d kp_row %d kp %d internal_tile_kp_k %d\n", kpSplitLane, kp_row, kp, internal_tile_kp_k);
                    } 

                    c += a * kp;
                  }
                }

                // if (a_row == 0 && kp_col == 0 && kpMullane == 0 && isfirstIdx(blockIdx))
                //   printf("kpSplitLane %d c %d internal_tile_kp_k %d\n", kpSplitLane, c, internal_tile_kp_k);
                int csh_col = (internal_tile_kp_n + kp_col)*(MAX_K/kpK) + a_col_start +kpMullane;
                if (NUM_KPK_SPLITS == 1) {
                  Creg[c_reg_col_start*Creg_Cols + creg_idx] += c;
                  int c_idx = start_row * N + c_reg_col_start*N_THREADS + creg_idx * (kpMulblockWarps * numKpColMult * (MAX_K/kpK)/numKpColMult) + threadIdx.x;
                  // C[c_idx] = c;
                // Csh[a_row][csh_col] += c;
                }
                else {
                  for (int offset = NUM_KPK_SPLITS/2; offset > 0; offset /= 2) 
                    c += __shfl_down_sync(0xffffffff, c, offset, NUM_KPK_SPLITS);
                  if (kpSplitLane == 0) {
                    Csh[a_row][csh_col] += c;
                  }
                }
              }
            }
          }
        }
      }
    }

    __syncthreads();
    
    if (NUM_KPK_SPLITS == 1) {
      for (int reg = 0; reg < Creg_SIZE; reg++) {
        int a_row = 0;
        int c_row = (a_row + start_row);
        int c_idx;
        if (K_EQUALS_VAR)
          c_idx = start_row * N + (reg/Creg_Cols)*N_THREADS + (reg%Creg_Cols) * (kpMulblockWarps * numKpColMult * (MAX_K/kpK)/numKpColMult) + threadIdx.x;
        else
          c_idx = start_row * N + tile_k * (MAX_K/kpK) + (reg/Creg_Cols)*N_THREADS + (reg%Creg_Cols) * (N_THREADS * (MAX_K/kpK)/numKpColMult) * (K/MAX_K) + threadIdx.x;
         
        C[c_idx] = Creg[reg];
      }
    } else {
      for (int a_row = 0; a_row < TILE_X; a_row++) {
        if (EXTERNAL_KP_K_TILE != MAX_KP_K) {
          //Atomic Store when there is an external KP_K tile
          for (int c_col = threadIdx.x; c_col < Csh_COLS; c_col += blockDim.x) {
            int c_row = (a_row + start_row);
            int c_idx;
            if (K_EQUALS_VAR)
              c_idx = c_row * N + external_tile_kp_n*Csh_COLS + c_col;
            else
              c_idx = c_row * N + external_tile_kp_n*(K/(MAX_KP_N/KP_N_TILE)) + tile_k * (MAX_K/kpK) + (c_col/(MAX_K/kpK)) * (K/kpK) + c_col%(MAX_K/kpK);
            
            atomicAdd(&C[c_idx], Csh[a_row][c_col]);
            // C[c_idx] = Csh[a_row][c_col];

            // if (kp_idx == 0 && c_idx >= 2048 && c_idx < 2048+64) {
            //   printf("Csh[a_row][%d] %d tile_k %d C[c_idx] %d\n", c_col, Csh[a_row][c_col], tile_k, C[c_idx]);
            // }
          }
        } else {
          //Normal Store
          for (int c_col = threadIdx.x*ldNumElems; c_col < Csh_COLS; c_col += blockDim.x*ldNumElems) {
            int c_row = (a_row + start_row);
            int c_idx;
            if (K_EQUALS_VAR)
              c_idx = c_row * N + external_tile_kp_n*Csh_COLS + c_col;
            else
              c_idx = c_row * N + external_tile_kp_n*(K/(MAX_KP_N/KP_N_TILE)) + tile_k * (MAX_K/kpK) + (c_col/(MAX_K/kpK)) * (K/kpK) + c_col%(MAX_K/kpK);
            
            *(LD_TYPE*)&C[c_idx] = *(LD_TYPE*)&Csh[a_row][c_col];
          }
        }
      }
    }
  }
}

// #define KP_EQUALS_VAR_KERNELS (N_COARSE_TB, MAX_K, KP_N_K, K_EQUALS_VAR) \
//   (void*)cuda_gemm<int,128,N_COARSE_TB,1,MAX_K,KP_N_K,KP_N_K,0>,\
//   (void*)cuda_gemm<int,128,N_COARSE_TB,1,MAX_K,KP_N_K,KP_N_K,1>,

#define N_THREADS 128
#define KP_N_TILE 64

#ifdef EVAL
    typedef float DATA_TYPE;
  #else
    typedef int DATA_TYPE;
  #endif

#define TILE_X 1

#define K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, K_EQUALS_VAR) \
  (void*)cuda_gemm<DATA_TYPE,N_THREADS,N_COARSE_TB,TILE_X,MAX_K,KP_N_K,KP_N_K,KP_N_TILE,K_EQUALS_VAR,0>,\
  (void*)cuda_gemm<DATA_TYPE,N_THREADS,N_COARSE_TB,TILE_X,MAX_K,KP_N_K,KP_N_K,KP_N_TILE,K_EQUALS_VAR,1>,

#define KP_N_K_KERNELS(N_COARSE_TB, MAX_K, KP_N_K) \
  K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, 0) \
  K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, 1)

#define MAX_K_KERNELS(N_COARSE_TB, MAX_K) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 2) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 4) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 8) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 16) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 32) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 64) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 128) 

#define COARSE_TB_KERNELS(N_COARSE_TB) \
  MAX_K_KERNELS(N_COARSE_TB, 16) \
  MAX_K_KERNELS(N_COARSE_TB, 32) \
  MAX_K_KERNELS(N_COARSE_TB, 64) \
  MAX_K_KERNELS(N_COARSE_TB, 128) \
  MAX_K_KERNELS(N_COARSE_TB, 256) \
  MAX_K_KERNELS(N_COARSE_TB, 512) \
  MAX_K_KERNELS(N_COARSE_TB, 1024) \
  MAX_K_KERNELS(N_COARSE_TB, 2048) \
  MAX_K_KERNELS(N_COARSE_TB, 4096) \

#define MAX_K 2048
#define NUM_MAX_K_KERNELS 9
#define NUM_KP_N_K_KERNELS 7
#define NUM_COARSE_TB_KERNELS 3
#define NUM_K_EQUALS_VAR 2
#define NUM_KPK_EQUALS_VAR 2

static void* cudaGemmSpecialized[NUM_COARSE_TB_KERNELS][NUM_MAX_K_KERNELS][NUM_KP_N_K_KERNELS][NUM_K_EQUALS_VAR][NUM_KPK_EQUALS_VAR] = {
  // KP_N_K_KERNELS(8, 1024, 32)
    COARSE_TB_KERNELS(1)
    COARSE_TB_KERNELS(2)
    COARSE_TB_KERNELS(4)
  };

static_assert(sizeof(cudaGemmSpecialized)/sizeof(void*) == NUM_COARSE_TB_KERNELS * NUM_KP_N_K_KERNELS * NUM_MAX_K_KERNELS*NUM_K_EQUALS_VAR*NUM_KPK_EQUALS_VAR);

int log2(int n){return 31 - __builtin_clz(n);}

template<typename T>
T* customKronGEMM(const int NUM_KP_MATS, T* kpMatmulResult[], T* x, T* kpMats[],
                    int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[], hipStream_t stream)
{
  typedef int (*cuda_gemm_ty)(int, int, int, T*, T*, T*, int kpNVar, int kpKVar);

  //Row Major Layout of all matrics
  T* resultMat = kpMatmulResult[0];
  T* prevResult = x;
  for (int i = 0; i < NUM_KP_MATS; i++) {

    const int KP_K_BATCH = 1;
    int N_COARSE_TB = (M > 100) ? 2 : 1;

    // int idx = (N_COARSE_TB/8)*NUM_MAX_K_KERNELS + (log2(K)-log2(16))*NUM_KP_N_K_KERNELS + (log2(KP_MAT_K[0])-log2(2));
    // printf("idx %d log2(K) %d log2(16) %d\n", idx, log2(K), log2(16));
    // assert(idx < sizeof(cudaGemmSpecialized)/sizeof(void*));
    
    int min_k = min(K, MAX_K);
    int k_equals_var = (min_k == K) ? 1 : 0;
    // if (min_k/KP_MAT_K[0] >= 256) {
    //   //K dimension is very high. Divide it in different threadblocks to have better parallelism
    //   min_k = min_k/KP_MAT_K[0];
    //   k_equals_var = 0;
    // }
    cuda_gemm_ty cuda_gemm_func = (cuda_gemm_ty)cudaGemmSpecialized[N_COARSE_TB/2][log2(min_k)-log2(16)][log2(KP_MAT_K[0])-log2(2)][k_equals_var][1];
    dim3 grid = {(K/min_k) * DIVUP(KP_MAT_N[0], KP_N_TILE), DIVUP((M/TILE_X), N_COARSE_TB), DIVUP(KP_MAT_K[0], EXTERNAL_KP_K_TILE_)}; 
    dim3 block = {N_THREADS,1,1};

    void *args[] = {&M, &N, &K, &prevResult, (void*)&kpMats[NUM_KP_MATS-i-1], (void*)&resultMat, (void*)&KP_MAT_N[NUM_KP_MATS-i-1], (void*)&KP_MAT_K[NUM_KP_MATS-i-1], &i};

    CUDACHECK(hipLaunchKernel((const void*)cuda_gemm_func, grid, block, &args[0], 0, stream));

    if (i < NUM_KP_MATS - 1) {
      prevResult = resultMat;
      if (resultMat == kpMatmulResult[0]) {        
        resultMat = kpMatmulResult[1];
      } else if (resultMat == kpMatmulResult[1]) {
        resultMat = kpMatmulResult[0];
      }
    }
    
    CUDACHECK(hipDeviceSynchronize());
  }

  return resultMat;
}

bool check(int* ref, int* computed, int M, int N) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      if (ref[i*N + j] != computed[i* N + j]) {
        printf("Mismatch for %d x %d at (%d, %d): ref = %d, computed = %d\n", M, N, i, j, ref[i*N+j], computed[i*N+j]);
        return false;
      }
    }
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int one(int i, int j) {return 1;}
int zeroOne(int i, int j) {return i % 2;}
int setToI(int i, int j) {return i;}
int randMod(int i, int j) {return rand()%5 + 1;}

template<typename T>
void setValues(int NUM_KP_MATS, T* kpMats[], T *x, int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[], int (*fnvalue)(int i, int j))
{
  for (int i = 0; i < NUM_KP_MATS; i++) {
    setMatrix(kpMats[i], KP_MAT_K[i], KP_MAT_N[i], fnvalue);
  }

  setMatrix(x, M, K, fnvalue);
}

struct MatrixSizes {
  const int M, N, K;
  const int NUM_KP_MATS;
  const std::vector<int> KP_MAT_N; 
  const std::vector<int> KP_MAT_K;
};

int main(int argc, char* argv[]) 
{

  #ifdef EVAL
  if (argc < 4) {printf("invalid command args\n"); return 0;}
  int npoints = atoi(argv[1]);
  int d = atoi(argv[2]);
  int twoPowerL = atoi(argv[3]);
  #endif

  std::vector<MatrixSizes> matrixSizes = {
                                          // {4,4,4, 2, {2,2},{2,2}},
                                          // {4,4,6, 2, {1,4},{2,3}},
                                          // {4,4,8, 2, {2,2},{2,4}},
                                          // {4,4,8, 2, {2,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{2,4}},
                                          // {8,8,8, 3, {2,2,2},{2,2,2}},
                                          // {8,8,32, 3, {2,2,2},{2,4,4}},
                                          // {8,16,32, 3, {4,2,2},{2,4,4}},
                                          // {8,8,16, 3, {2,2,2},{2,4,2}},
                                          // {16,8,8, 3, {2,2,2},{2,2,2}},
                                          // {16,16,16, 2, {4,4},{4,4}},
                                          // {16,16,16, 3, {4,2,2},{4,2,2}},
                                          // {16,16,16, 3, {4,2,2},{2,4,2}},
                                          // {16,16,16, 3, {8,2,1},{2,4,2}},
                                          // {16,16,16, 4, {2,2,2,2},{2,2,2,2}},
                                          // {16,16,64, 4, {2,2,2,2},{2,4,2,4}},
                                          // {256,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {256,256,256, 2, {16,16},{16,16}},
  #ifdef EVAL
                                          // {65536,1024,1024, 2, {32,32},{32,32}},
                                          // {65536,256,256, 2, {16,16},{16,16}},
                                          // {65536,512,512, 3, {8,8,8},{8,8,8}},
                                          // {100,1024,1024, 2, {32,32},{32,32}},
                                          // {10,1024,1024, 2, {32,32},{32,32}},
                                          // {1,1024,1024, 2, {32,32},{32,32}},
                                          // {100,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {10,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {1,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {100,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {1,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          
                                          // {100,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {10,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {1,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},

                                          // {100,1024,1024, 3, {16,16,4},{16,16,4}},
                                          // {100,256,256, 2, {16,16},{16,16}},
                                          // {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {1,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},

                                          // {100,512*8*8,512*8*8, 5, {8,8,8,8,8},{8,8,8,8,8}},

                                          // {100,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {10,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {1,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {1024,32*1024,32*1024, 2, {32,32,32},{32,32,32}},
  #else
                                          // {10,1024,1024, 2, {32,32},{32,32}},                                        
                                          {10,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          {10,256,256, 2, {16,16},{16,16}},
                                          // {10,256,256, 2, {16,16},{16,16}},
                                          {10,512,512, 3, {8,8,8},{8,8,8}},
                                          {10,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          {4,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {1, 4096, 4096, 2, {64,64},{64,64}},
                                          // {1, 128*128, 128*128, 2, {128,128},{128,128}}
  #endif

                                          // {1024, 1024, 1024, 2, {32,32},{32,32}}
                                          };

  // int (*fnvalues[4])(int, int) = {&one, &zeroOne, &setToI, &randMod};
  int (*fnvalues[1])(int, int) = {&randMod};
  
  #ifdef EVAL
  int Msz = 1;
  for (int i = 0; i < d; i++) {
    Msz *= twoPowerL;
  }
  MatrixSizes matrixSize {
    npoints, Msz, Msz, d, std::vector<int>(d, twoPowerL), std::vector<int>(d, twoPowerL)
  };
  matrixSizes.push_back(matrixSize);
  #endif 

  for (MatrixSizes matrixSize : matrixSizes) {
    int M = matrixSize.M;
    int N = matrixSize.N;
    int K = matrixSize.K;
    
    int NUM_KP_MATS = matrixSize.NUM_KP_MATS;
    int KP_MAT_N[NUM_KP_MATS];
    int KP_MAT_K[NUM_KP_MATS];

    printf("Matmul: %d x %d x %d, Num KP Factors: %d\n", M, N, K, NUM_KP_MATS);
    int n=1,k=1;
    for (int i = 0; i < NUM_KP_MATS; i++) {
      k *= matrixSize.KP_MAT_K[i];
      n *= matrixSize.KP_MAT_N[i];
    }
    if (n != N || k != K) {
      printf("Invalid KP Factors Sizes %d != %d, %d != %d\n", n, N, k, K);
    }

    DATA_TYPE *kpout[NUM_KP_MATS];
    DATA_TYPE *kpMats[NUM_KP_MATS];
    DATA_TYPE* kpMatmulResult[NUM_KP_MATS];

    DATA_TYPE *x = new DATA_TYPE[M*K];

    DATA_TYPE* dX;
    DATA_TYPE** dKpOut;
    DATA_TYPE** dKpMats;
    DATA_TYPE** dKpMatmulResult;
    
    CUDACHECK(hipMalloc(&dX, M*K * sizeof(DATA_TYPE)));
    
    DATA_TYPE* __dKpOut[NUM_KP_MATS];
    DATA_TYPE* __dKpMats[NUM_KP_MATS];
    DATA_TYPE* __dKpMatmulResult[2];

    for (int i = 0; i < NUM_KP_MATS; i++) {
      KP_MAT_K[i] = matrixSize.KP_MAT_K[i];
      KP_MAT_N[i] = matrixSize.KP_MAT_N[i];
      kpMats[i] = new DATA_TYPE[KP_MAT_K[i] * KP_MAT_N[i]];
      kpout[i] = new DATA_TYPE[K*N]; //TODO: larger than needed
      kpMatmulResult[i] = new DATA_TYPE[M*std::max(N,K)];

      CUDACHECK(hipMalloc(&__dKpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(DATA_TYPE)));
      // CUDACHECK(hipMalloc(&__dKpOut[i], K * N * sizeof(int)));
      

      // CUDACHECK(hipMemset(__dKpOut[i], 0, K * N * sizeof(int)));
    }

    // CUDACHECK(hipMemcpy(&dKpOut[0], &__dKpOut[0], NUM_KP_MATS * sizeof(int*), hipMemcpyHostToDevice));
    // CUDACHECK(hipMemcpy(&dKpMats[0], &__dKpMats[0], NUM_KP_MATS * sizeof(DATA_TYPE*), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&__dKpMatmulResult[0], M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMalloc(&__dKpMatmulResult[1], M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMemset(__dKpMatmulResult[0], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMemset(__dKpMatmulResult[1], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));

    DATA_TYPE* result = new DATA_TYPE[M*N];

    DATA_TYPE* dResult;

    CUDACHECK(hipMalloc(&dResult, M * N * sizeof(DATA_TYPE)));

    for (int fnvalue = 0; fnvalue < sizeof(fnvalues)/sizeof(fnvalues[0]); fnvalue++) {
      setValues(NUM_KP_MATS, kpMats, x, M, N, K, KP_MAT_N, KP_MAT_K, fnvalues[fnvalue]);

      for (int i = 0; i < NUM_KP_MATS; i++) {
        CUDACHECK(hipMemcpy(__dKpMats[i], kpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
      }
    
      CUDACHECK(hipMemcpy(dX, x, M * K * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
  #ifndef EVAL
      baselineKPThenMatmul(NUM_KP_MATS, result, x, kpout, kpMats, 
                           M, N, K, KP_MAT_N, KP_MAT_K);
  #endif
      // slicedMatmul(NUM_KP_MATS, kpMatmulResult, x, kpMats,
      //              M, N, K, KP_MAT_N, KP_MAT_K);

      for (int i = 0; i < 2; i++)
        CUDACHECK(hipMemset(__dKpMatmulResult[i], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));
  #ifdef EVAL  
      hipStream_t stream;
      hipStreamCreate(&stream);
      hipEvent_t start;
      hipEvent_t end;
      float elapsedTime = 0;
      CUDACHECK(hipEventCreate(&start));
      CUDACHECK(hipEventCreate(&end));
      for (int i = 0; i < 10; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, stream);
      CUDACHECK(hipStreamSynchronize(stream));
      CUDACHECK(hipEventRecord(start, stream));
      for (int i = 0; i < 1000; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, stream);
      CUDACHECK(hipEventRecord(end, stream));
      CUDACHECK(hipEventSynchronize(end));
      CUDACHECK(hipEventElapsedTime(&elapsedTime, start, end));
      printf("elapsedtime %f\n", elapsedTime/1000);

      for (int i = 0; i < NUM_KP_MATS; i++) {
        CUDACHECK(hipFree(__dKpMats[i]));
      }

      CUDACHECK(hipFree(__dKpMatmulResult[0]));
      CUDACHECK(hipFree(__dKpMatmulResult[1]));
      CUDACHECK(hipFree(dX));
      CUDACHECK(hipFree(dResult));
      continue;
  #else
      
      DATA_TYPE* dResult = customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, 0);
      CUDACHECK(hipDeviceSynchronize());
  #endif
      // return;
      #ifndef EVAL 
      DATA_TYPE* hKpMatMulResult = new DATA_TYPE[M*N];
      // return;
      // for (int i = 0; i < NUM_KP_MATS; i++)
      //   CUDACHECK(hipMemcpy(kpMatmulResult[i], __dKpMatmulResult[i], M*N*sizeof(int), hipMemcpyDeviceToHost));
      CUDACHECK(hipMemcpy(kpMatmulResult[NUM_KP_MATS-1], dResult, M*N*sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      // if (check(result, kpMatmulResult[NUM_KP_MATS-1], M, N))
      if (check(result, kpMatmulResult[NUM_KP_MATS-1], M,N))
        printf("Results Correct for test %d\n", fnvalue);
      else {
        // printf("\nMatmul:");
        // printMatrix(result, K, N);

        // printf("\nx:");
        // printMatrix(x, M, K);    
        // for (int kpMatId = 0; kpMatId < NUM_KP_MATS; kpMatId++) {
        //   printf("\nKP Mat %d:", kpMatId);
        //   printMatrix(kpMats[kpMatId], KP_MAT_K[kpMatId], KP_MAT_N[kpMatId]);
        // }
        // // printf("\nKP Out:");
        // // printMatrix(kpout[0], 8, 8);
        // for (int id = 0; id < NUM_KP_MATS; id++) {
        //   printf("\nKP result %d:", id);
        //   printMatrix(kpMatmulResult[id], M, N);
        // }
        // printf("\nKP result 2:");
        // printMatrix(kpMatmulResult[2], 16, 16);
        // printf("\nKP result 3:");
        // printMatrix(kpMatmulResult[3], 16, 16);
        // printf("\nKP result 1:");
        // printMatrix(kpMatmulResult[1], M, N);
        // printf("\n");
        return 0;
      }
      #endif
    }

    //Is there really a need to free anything when you have tons of RAM, am I right?
  }

  return 0;
}