#include "hip/hip_runtime.h"

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#include <iostream>
#include <string>
#include <cstdlib>
#include <cassert>
#include <vector>

#define MIN(x,y) (((x) < (y)) ? (x) : (y))
#define MAX(x,y) (((x) > (y)) ? (x) : (y))
#define DIVUP(x, y) (((x) + (y) - 1)/((y)))

template<typename T>
void setMatrix(T* mat, int M, int N, int (*fnvalue)(int i, int j)) 
{
  // #pragma omp parallel for collapse(2)
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      mat[i*N + j] = (T)fnvalue(i,j);
    }
  }
}

void printMatrix(int* mat, int M, int N) 
{
  printf("[");
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      // if (mat[i*N + j] == 18496)
        // printf("%d,%d\n",i,j);
      printf("%d, ", mat[i*N + j]);
    }
    if (i < M-1)
      printf("\n");
  }
  printf("]");
}

void baselineKPThenMatmul(int NUM_KP_MATS, int* result, int* x, int* kpout[], int* kpMats[],
                          int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int cols;
  int rows;

  for (int kp = 0; kp < NUM_KP_MATS - 1; kp++) {
    int* kpFirst = (kp == 0) ? kpMats[0] : kpout[kp - 1];
    int kpFirstRows = (kp == 0) ? KP_MAT_K[0] : rows;
    int kpFirstCols = (kp == 0) ? KP_MAT_N[0] : cols;

    cols = kpFirstCols * KP_MAT_N[kp+1];
    rows = kpFirstRows * KP_MAT_K[kp+1];
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        int v2 = kpMats[kp+1][(i%KP_MAT_K[kp+1]) * KP_MAT_N[kp+1] + j%KP_MAT_N[kp+1]];
        int v1 = kpFirst[(i/KP_MAT_K[kp+1]) * kpFirstCols + j/KP_MAT_N[kp+1]];
        kpout[kp][i*cols + j] = v1 * v2;
      }
    }
  }

  for(int i = 0; i < M; i++) {    
    for(int j = 0; j < N; j++) {    
      result[i* N + j] = 0;    
      for(int k = 0; k < K; k++) {   
        result[i * N + j] += x[i*K + k]*kpout[NUM_KP_MATS-2][k*N + j];
      }    
    }    
  }
}

/**
 * 
*/
void slicedMatmul(int NUM_KP_MATS, int* kpMatmulResult[], int* x, int* kpMats[],
                  int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[])
{
  int secFacRowMulSize = 1;
  int rowsTillNow = 1;
  int colsTillNow = 1;
  int resultCols;
  for (int kp = 0; kp < NUM_KP_MATS; kp++) {
    int* prevKPMatmul = (kp == 0) ? x : kpMatmulResult[kp - 1];
    int kpSecondK = KP_MAT_K[NUM_KP_MATS - 1 - kp];
    int kpSecondN = KP_MAT_N[NUM_KP_MATS - 1 - kp];
    int prevKPMatmulCols = (kp == 0) ? K : resultCols;

    resultCols = (prevKPMatmulCols/kpSecondK) * kpSecondN;
    secFacRowMulSize = (kp == 0) ? K/kpSecondK : rowsTillNow * K/(colsTillNow * KP_MAT_K[NUM_KP_MATS - 1 - (kp)]);

    //Number of times a column is multiplied with input matrix is equal to 
    //N/(number of column elements of this matrix * cols so far) * number of rows so far.

    rowsTillNow *= KP_MAT_N[NUM_KP_MATS - 1 - (kp)];
    colsTillNow *= KP_MAT_K[NUM_KP_MATS - 1 - (kp)];

    for (int i = 0; i < M; i++) {
      for (int j = 0; j < resultCols; j++) {
        int r = 0;

        for (int kp_k = 0; kp_k < kpSecondK; kp_k++) {
          int slice = (j / secFacRowMulSize) % kpSecondN;

          int v2 = kpMats[NUM_KP_MATS - 1 - kp][kp_k*kpSecondN + slice];
          
          r += prevKPMatmul[i* prevKPMatmulCols + (j*kpSecondK)%prevKPMatmulCols + kp_k] * v2;
        }

        kpMatmulResult[kp][i*resultCols + j] = r;
      }
    }
  }
}

#define EXTERNAL_KP_K_TILE_ 128

// #define C_IN_REG

#define C_IN_SHMEM
template<uint MAX_KP_N, uint KP_N_TILE> __device__ uint get_tile_k() {return blockIdx.x/DIVUP(MAX_KP_N, KP_N_TILE);}
template<uint MAX_KP_N, uint KP_N_TILE> __device__ uint get_external_tile_kp_n() {return blockIdx.x%DIVUP(MAX_KP_N, KP_N_TILE);}

__device__ bool isfirstIdx(dim3 idx) {return idx.x == 0 && idx.y == 0 & idx.z == 0;}

template<typename T, uint N_THREADS, uint N_COARSE_TB, uint TILE_X, uint MAX_K, uint MAX_KP_N, uint MAX_KP_K, uint KP_N_TILE_, uint K_EQUALS_VAR, uint KPK_EQUALS_VAR>
__global__ void __launch_bounds__(N_THREADS) cuda_gemm(uint M, uint NVar, uint KVar, const T * __restrict__ A, const T * __restrict__ kron_fac, T * __restrict__ C, uint kpNVar, uint kpKVar, uint kp_idx) {
  const uint KP_N_TILE = MIN(KP_N_TILE_, MAX_KP_N);
  const uint NUM_KP_N_TILES = MAX_KP_N/KP_N_TILE;
  const uint INTERNAL_KP_N_TILE = MIN(128, KP_N_TILE);
  const uint EXTERNAL_KP_K_TILE = MIN(EXTERNAL_KP_K_TILE_, MAX_KP_K);
  const uint INTERNAL_KP_K_TILE = MIN(32, EXTERNAL_KP_K_TILE);

  #ifdef EVAL
    typedef float4 LD_TYPE; 
  #else 
    typedef int4 LD_TYPE; 
  #endif 

  __shared__ __align__(128) T kron_fac_sh[INTERNAL_KP_N_TILE][INTERNAL_KP_K_TILE+1];//TODO: Change padding based on value o1, KP_K and TILE_Y
  const uint Ash_COLS = MAX_K/(MAX_KP_K/INTERNAL_KP_K_TILE);
  __shared__ __align__(128) T Ash[TILE_X][Ash_COLS];
  const uint C_ELEMS_STORE = N_THREADS * (sizeof(LD_TYPE)/sizeof(T));
  const uint Csh_COLS = MAX_K/(MAX_KP_N/KP_N_TILE);
  const uint Csh_COLS_SIZE = MIN(Csh_COLS, C_ELEMS_STORE);
#ifdef C_IN_SHMEM
  __shared__ __align__(128) T Csh[TILE_X][Csh_COLS];//Allocate Csh for only as many values that are produced
#endif

  uint wid = threadIdx.x/32;
  uint lane = threadIdx.x%32;
  uint blockWarps = blockDim.x/32;
  uint kpK;
  uint kpN;
  uint K;
  uint N;
 
  if (KPK_EQUALS_VAR) {
    kpK = MAX_KP_K;
    kpN = MAX_KP_N;
  } else {
    kpK = kpKVar;
    kpN = kpNVar;
  }

  if (K_EQUALS_VAR) {
    K = MAX_K;
    N = K;
  } else {
    K = KVar;
    N = NVar;
  }

  const uint KPK_SPLIT_SIZE = MIN(16, INTERNAL_KP_K_TILE);
  const uint NUM_KPK_SPLITS = MAX(1, INTERNAL_KP_K_TILE/KPK_SPLIT_SIZE);
  const uint ldNumElems = (sizeof(LD_TYPE)/sizeof(T));

  uint external_tile_kp_k = blockIdx.z;
  
  if (KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K) {
    #ifdef EVAL
      typedef float4 LD_TYPE; 
    #else 
      typedef int4 LD_TYPE; 
    #endif
    const int ldNumElems = sizeof(LD_TYPE)/sizeof(T);
    const int ldSize = MIN(kpN*kpK, ldNumElems);

    for (auto i = threadIdx.x*ldSize; i < (kpN * kpK); i += blockDim.x*ldSize) {
      // kron_fac_sh[i%kpN][i/kpK] = kron_fac[i];
      LD_TYPE a = *(LD_TYPE*)&kron_fac[i];
      T a1[4] = {a.x, a.y, a.z, a.w};
      for (int j = 0; j < ldSize; j++) {
        int idx = i + j;
        kron_fac_sh[idx%kpK][idx/kpK] = a1[j];
      }
    }
  } else {
  }

  
  const uint numKpColMult = MIN(MAX_K/MAX_KP_K, N_THREADS); //Threads executing in parallel to multiply one column of KP with MAX_K row elements of A, 32
  #ifdef C_IN_REG
  const uint kpMulblockWarps = MIN(MAX_KP_K, N_THREADS/numKpColMult); //
  const uint Creg_SIZE = MAX(1, Csh_COLS/N_THREADS); //
  const uint Creg_Rows = (MAX_K/MAX_KP_K)/numKpColMult; //
  const uint Creg_Cols = MAX(1, INTERNAL_KP_N_TILE/kpMulblockWarps); //
  const uint NUM_INTERNAL_KP_N_TILES = KP_N_TILE/INTERNAL_KP_N_TILE; //
  // assert(Creg_SIZE == Creg_Cols * Creg_Rows * NUM_INTERNAL_KP_N_TILES);

  register T Creg[Creg_SIZE];
  #endif

  register T kron_fac_r;

  #ifdef C_IN_SHMEM
  const uint kpMulblockWarps = N_THREADS/numKpColMult;
  #endif

  uint kpMullane = threadIdx.x%numKpColMult;
  uint kpMulwid = threadIdx.x/numKpColMult; //0
   //TODO: Names should be different

  for (uint start_row = blockIdx.y * TILE_X; start_row < gridDim.y * TILE_X * N_COARSE_TB; start_row += gridDim.y * TILE_X) {
    #ifdef C_IN_SHMEM
      for (uint a_row = 0; a_row < TILE_X; a_row += 1) {
        for (uint i = threadIdx.x; i < Csh_COLS; i += blockDim.x)
          Csh[a_row][i] = 0;
      }
    #endif
    #ifdef C_IN_REG
      #pragma unroll
      for (uint reg = 0; reg < Creg_SIZE; reg++) {
        Creg[reg] = 0;
      }
    #endif

    for (uint internal_tile_kp_k = 0; internal_tile_kp_k < EXTERNAL_KP_K_TILE; internal_tile_kp_k += INTERNAL_KP_K_TILE) {
      for (uint a_row = 0; a_row < TILE_X; a_row += 1) {
        for (uint a_col = threadIdx.x*ldNumElems; a_col < Ash_COLS; a_col += blockDim.x*ldNumElems) {
          uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
          if (INTERNAL_KP_K_TILE == MAX_KP_K) {
            LD_TYPE a = *(LD_TYPE*)&A[(a_row + start_row) * K + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + a_col];

            *(LD_TYPE*)&Ash[a_row][a_col] = a;
          } else {
            LD_TYPE a = *(LD_TYPE*)&A[(a_row + start_row) * K + (K_EQUALS_VAR ? 0 : tile_k*MAX_K) + \
                                      (a_col/INTERNAL_KP_K_TILE)*kpK + external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + a_col % INTERNAL_KP_K_TILE];
            
            *(LD_TYPE*)&Ash[a_row][a_col] = a;
          }
        }
      }
    
      //TODO: nvcc unrolls this loop, which leads to high register usage
      for (uint internal_tile_kp_n = 0; internal_tile_kp_n < KP_N_TILE; internal_tile_kp_n += INTERNAL_KP_N_TILE) {
        if (!(KP_N_TILE == MAX_KP_N && INTERNAL_KP_N_TILE == MAX_KP_N && INTERNAL_KP_K_TILE == MAX_KP_K)) {
          //Create kpK subwarps and each subwarp loads 0 to INTERNAL_KP_N_TILE elements
          #ifdef EVAL
            typedef float4 LD_TYPE; 
          #else 
            typedef int4 LD_TYPE; 
          #endif
          const uint ldNumElems = sizeof(LD_TYPE)/sizeof(T);
          const uint ldSize = MIN(INTERNAL_KP_N_TILE, ldNumElems);

          for (uint swid = threadIdx.x/(INTERNAL_KP_N_TILE/ldSize); swid < INTERNAL_KP_K_TILE; swid += blockDim.x/(INTERNAL_KP_N_TILE/ldSize)) {
            uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
            uint col = external_tile_kp_n*KP_N_TILE + internal_tile_kp_n + (threadIdx.x%(INTERNAL_KP_N_TILE/ldSize))*ldSize;
            uint row = swid;
            // kron_fac_sh[threadIdx.x%INTERNAL_KP_N_TILE][row] = kron_fac[(external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + row) * kpN + col];
            LD_TYPE a = *(LD_TYPE*)&kron_fac[(external_tile_kp_k * EXTERNAL_KP_K_TILE + internal_tile_kp_k + row) * kpN + col];
            T a1[4] = {a.x, a.y, a.z, a.w};
            for (uint i = 0; i < ldSize; i++) {
              uint idx = (threadIdx.x%(INTERNAL_KP_N_TILE/ldSize))*ldSize + i%ldSize;
              kron_fac_sh[idx][row] = a1[i];
            }
          }
        }

        __syncthreads();
        
        #ifdef C_IN_REG
        if (kpMulwid < numKpColMult)
        #endif
        for (uint a_row = 0; a_row < TILE_X; a_row++) {
          #pragma unroll
          #ifdef C_IN_REG
          for (uint a_col_start = 0, c_reg_col_start = 0; c_reg_col_start < (MAX_K/MAX_KP_K)/numKpColMult; a_col_start += numKpColMult, c_reg_col_start++) {
          #endif
          #ifdef C_IN_SHMEM
          for (uint a_col_start = 0; a_col_start < MAX_K/kpK; a_col_start += numKpColMult) {
          #endif
            const uint MAX_AR_SZ = KPK_SPLIT_SIZE;

            //Load MAX_AR_SZ elements at a time to limit the register usage
            for (uint ar_start_id = 0; ar_start_id < INTERNAL_KP_K_TILE; ar_start_id += MAX_AR_SZ) { //TODO: Shared memory bank conflicts with kpK = 32 and AR_SZ = 16
              register T Ar[MAX_AR_SZ];
              uint kpKlane = lane % MAX_AR_SZ; //
              uint ar_start = (ar_start_id + (lane/MAX_AR_SZ)*MAX_AR_SZ)%INTERNAL_KP_K_TILE;

              for (uint a_col = kpKlane, i = 0; i < MAX_AR_SZ; i++) { //
                  Ar[i] = Ash[a_row][(a_col_start+kpMullane)*INTERNAL_KP_K_TILE + ar_start + (a_col + i) % MAX_AR_SZ];//TODO: Shared memory bank conflicts here with KP_K = 4
              }
              
              #pragma unroll
              #ifdef C_IN_REG
              for (uint kp_col = kpMulwid, c_reg_idx = 0; c_reg_idx < INTERNAL_KP_N_TILE/kpMulblockWarps; kp_col += kpMulblockWarps, c_reg_idx++) {
              #endif
              #ifdef C_IN_SHMEM
              for (uint kp_col = kpMulwid; kp_col < min(kpN, INTERNAL_KP_N_TILE); kp_col += kpMulblockWarps) {
              #endif
                T c = 0;

                kron_fac_r = kron_fac_sh[kp_col][lane % INTERNAL_KP_K_TILE];
                
                #pragma unroll
                for (uint a_col = 0; a_col < MAX_AR_SZ; a_col++) {
                  //if (a_col < kpK) 
                  {
                    T a = Ar[a_col]; //Ash[a_row][a_col_start/KP_K][a_col]; //Ar[a_col];
                    uint kp_row;
                    kp_row = ar_start + (a_col + kpKlane)%KPK_SPLIT_SIZE; //kpMullane/(warpSize/kpK)
                    //} else {kp_row = (a_col+kpKlane) < kpK ? (a_col+kpKlane) : (a_col+kpKlane) - kpK;} //TODO:
                    T kp;
                    if (true){//(INTERNAL_KP_K_TILE <= 32 && kpK <= 64) {
                      // kp = kron_fac_sh[kp_col][ar_start+(a_col+kpKlane)%min(kpK, KPK_SPLIT_SIZE)];
                      kp = __shfl_sync(0xffffffff, kron_fac_r, kp_row, INTERNAL_KP_K_TILE);
                      // if (kp_col == 0 && ar_start == 16 && kpK == 128 && kp != kp1 && isfirstIdx(blockIdx))
                      //   printf("kp_col %d kp_row %d %d, %d %d, %d %d %d\n", kp_col, kp_row, ar_start + (a_col+kpKlane) % min(MAX_AR_SZ, kpK), kp, kp1, ar_start, a_col, kpKlane);
                    } else {
                      //FIXME: For 1x16384 with 128x128 Kronecker factors, the results are incorrect for __shfl_sync because numkpcolmult != 32
                      // kp_row = ar_start + kpKlane + (a_col+kpKlane < min(MAX_AR_SZ, kpK) ? a_col : a_col - min(MAX_AR_SZ, kpK));
                      kp = kron_fac_sh[kp_col][kp_row];
                      // if (a_row == 0 && kp_col == 0 && kpMullane == 0 && isfirstIdx(blockIdx))
                      //   printf("kpSplitLane %d kp_row %d kp %d internal_tile_kp_k %d\n", kpSplitLane, kp_row, kp, internal_tile_kp_k);
                    } 

                    c += a * kp;
                  }
                }

                // if (threadIdx.x == 0 && kp_col == 0 && kpMullane == 0 && isfirstIdx(blockIdx))
                //   printf("318: internal_tile_kp_n %d creg_idx1 %d c %d kp_idx %d %d\n", internal_tile_kp_n, creg_idx1, c, kp_idx, Creg[(internal_tile_kp_n/INTERNAL_KP_N_TILE)*4 + c_reg_col_start*Creg_Cols + creg_idx1]);
                #ifdef C_IN_REG
                uint __idx = (internal_tile_kp_n/INTERNAL_KP_N_TILE)*Creg_Cols*Creg_Rows + c_reg_col_start*Creg_Cols + c_reg_idx;
                Creg[__idx] += c;
                #endif 

                // if (threadIdx.x == 0 && kpMulwid == 0 && isfirstIdx(blockIdx))
                //   printf("323: internal_tile_kp_n %d creg_idx1 %d c %d kp_idx %d %d  %d\n", internal_tile_kp_n, creg_idx1, c, kp_idx, Creg[__idx], __idx);
                // __syncwarp();
                #ifdef C_IN_SHMEM
                uint csh_col = (internal_tile_kp_n + kp_col)*(MAX_K/kpK) + a_col_start +kpMullane;
                Csh[a_row][csh_col] += c;
                #endif 
              }
            }
          }
        }
      }
    }
    
    #ifdef C_IN_REG
    for (uint reg = 0; reg < Creg_SIZE; reg++) {
      uint a_row = 0;
      uint c_row = (a_row + start_row);
      uint c_idx;
      uint c_col;
      
      c_col = (reg/(Creg_Cols * Creg_Rows)) * (MAX_K/kpK) * INTERNAL_KP_N_TILE  + ((reg/Creg_Cols)%Creg_Rows)*N_THREADS + (reg%Creg_Cols) * (N_THREADS * (MAX_K/kpK)/numKpColMult) + threadIdx.x;

      if (!K_EQUALS_VAR) {
        uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
        c_col = tile_k * (MAX_K/kpK) + (c_col/(MAX_K/kpK)) * (K/kpK) + c_col%(MAX_K/kpK);
      }
      
      c_idx = start_row * N + c_col;
      if (c_col < K)
        C[c_idx] = Creg[reg];
    }
    #endif
    
    #ifdef C_IN_SHMEM
    __syncthreads();
    for (int a_row = 0; a_row < TILE_X; a_row++) {
      if (EXTERNAL_KP_K_TILE != MAX_KP_K) {
        //Atomic Store when there is an external KP_K tile
        for (uint c_col = threadIdx.x; c_col < Csh_COLS; c_col += blockDim.x) {
          uint c_row = (a_row + start_row);
          uint c_idx;
          uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
          if (K_EQUALS_VAR) 
            c_idx = c_row * N + external_tile_kp_n*Csh_COLS + c_col;
          else {
            uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
            c_idx = c_row * N + external_tile_kp_n*(K/(MAX_KP_N/KP_N_TILE)) + tile_k * (MAX_K/kpK) + (c_col/(MAX_K/kpK)) * (K/kpK) + c_col%(MAX_K/kpK);
          }
          
          atomicAdd(&C[c_idx], Csh[a_row][c_col]);
          // C[c_idx] = Csh[a_row][c_col];

          // if (kp_idx == 0 && c_idx >= 2048 && c_idx < 2048+64) {
          //   printf("Csh[a_row][%d] %d tile_k %d C[c_idx] %d\n", c_col, Csh[a_row][c_col], tile_k, C[c_idx]);
          // }
        }
      } else {
        //Normal Store
        for (uint c_col = threadIdx.x*ldNumElems; c_col < Csh_COLS; c_col += blockDim.x*ldNumElems) {
          uint c_row = (a_row + start_row);
          uint c_idx;
          uint external_tile_kp_n = get_external_tile_kp_n<MAX_KP_N, KP_N_TILE>();
          if (K_EQUALS_VAR)
            c_idx = c_row * N + external_tile_kp_n*Csh_COLS + c_col;
          else {
            uint tile_k = get_tile_k<MAX_KP_N, KP_N_TILE>();
            c_idx = c_row * N + external_tile_kp_n*(K/(MAX_KP_N/KP_N_TILE)) + tile_k * (MAX_K/kpK) + (c_col/(MAX_K/kpK)) * (K/kpK) + c_col%(MAX_K/kpK);
          }
          
          *(LD_TYPE*)&C[c_idx] = *(LD_TYPE*)&Csh[a_row][c_col];
        }
      }
    }
    #endif  
  }
}

#define N_THREADS 512
#define KP_N_TILE 128

#ifdef EVAL
    typedef float DATA_TYPE;
  #else
    typedef int DATA_TYPE;
  #endif

#define TILE_X 1

#define K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, K_EQUALS_VAR) \
(void*)cuda_gemm<DATA_TYPE,N_THREADS,N_COARSE_TB,TILE_X,MAX_K,KP_N_K,KP_N_K,KP_N_TILE,K_EQUALS_VAR,1>,
  // (void*)cuda_gemm<DATA_TYPE,N_THREADS,N_COARSE_TB,TILE_X,MAX_K,KP_N_K,KP_N_K,KP_N_TILE,K_EQUALS_VAR,0>,

#define KP_N_K_KERNELS(N_COARSE_TB, MAX_K, KP_N_K) \
  K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, 0) \
  K_EQUALS_VAR_KERNELS(N_COARSE_TB, MAX_K, KP_N_K, 1)

#define MAX_K_KERNELS(N_COARSE_TB, MAX_K) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 2) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 4) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 8) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 16) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 32) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 64) \
  KP_N_K_KERNELS(N_COARSE_TB, MAX_K, 128) 


#define COARSE_TB_KERNELS(N_COARSE_TB) \
  MAX_K_KERNELS(N_COARSE_TB, 128) \
  MAX_K_KERNELS(N_COARSE_TB, 256) \
  MAX_K_KERNELS(N_COARSE_TB, 512) \
  MAX_K_KERNELS(N_COARSE_TB, 1024) \
  MAX_K_KERNELS(N_COARSE_TB, 2048) \
  MAX_K_KERNELS(N_COARSE_TB, 4096) \
  // MAX_K_KERNELS(N_COARSE_TB, 8192) \

  // MAX_K_KERNELS(N_COARSE_TB, 16) \
  // MAX_K_KERNELS(N_COARSE_TB, 32) \
  // MAX_K_KERNELS(N_COARSE_TB, 64) \
  
#define MAX_K 4096
#define MIN_K 128
#define NUM_MAX_K_KERNELS 8
#define NUM_KP_N_K_KERNELS 7
#define NUM_COARSE_TB_KERNELS 1
#define NUM_K_EQUALS_VAR 2
#define NUM_KPK_EQUALS_VAR 1

static void* cudaGemmSpecialized[NUM_COARSE_TB_KERNELS][NUM_MAX_K_KERNELS][NUM_KP_N_K_KERNELS][NUM_K_EQUALS_VAR][NUM_KPK_EQUALS_VAR] = {
  // KP_N_K_KERNELS(8, 1024, 32)
    COARSE_TB_KERNELS(1)
    // COARSE_TB_KERNELS(2)
    // COARSE_TB_KERNELS(4)
  };

// static_assert(sizeof(cudaGemmSpecialized)/sizeof(void*) == NUM_COARSE_TB_KERNELS * NUM_KP_N_K_KERNELS * NUM_MAX_K_KERNELS*NUM_K_EQUALS_VAR*NUM_KPK_EQUALS_VAR);

int log2(int n){return 31 - __builtin_clz(n);}

template<typename T>
T* customKronGEMM(const int NUM_KP_MATS, T* kpMatmulResult[], T* x, T* kpMats[],
                    int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[], hipStream_t stream)
{
  typedef int (*cuda_gemm_ty)(int, int, int, T*, T*, T*, int kpNVar, int kpKVar);

  //Row Major Layout of all matrics
  T* resultMat = kpMatmulResult[0];
  T* prevResult = x;
  for (int i = 0; i < NUM_KP_MATS; i++) {

    const int KP_K_BATCH = 1;
    int N_COARSE_TB = (M > 100) ? 2 : 1;

    // int idx = (N_COARSE_TB/8)*NUM_MAX_K_KERNELS + (log2(K)-log2(16))*NUM_KP_N_K_KERNELS + (log2(KP_MAT_K[0])-log2(2));
    // printf("idx %d log2(K) %d log2(16) %d\n", idx, log2(K), log2(16));
    // assert(idx < sizeof(cudaGemmSpecialized)/sizeof(void*));
    
    int min_k = min(K, MAX_K);
    int k_equals_var = (min_k == K) ? 1 : 0;
    // if (min_k/KP_MAT_K[0] >= 256) {
    //   //K dimension is very high. Divide it in different threadblocks to have better parallelism
    //   min_k = min_k/KP_MAT_K[0];
    //   k_equals_var = 0;
    // }cudaGemmSpecialized[0][0][0][k_equals_var][1]; //
    cuda_gemm_ty cuda_gemm_func = (cuda_gemm_ty)cudaGemmSpecialized[N_COARSE_TB/2][log2(min_k)-log2(MIN_K)][log2(KP_MAT_K[0])-log2(2)][k_equals_var][0];
    dim3 grid = {(K/min_k) * DIVUP(KP_MAT_N[0], KP_N_TILE), DIVUP((M/TILE_X), N_COARSE_TB), DIVUP(KP_MAT_K[0], EXTERNAL_KP_K_TILE_)}; 
    dim3 block = {N_THREADS,1,1};

    void *args[] = {&M, &N, &K, &prevResult, (void*)&kpMats[NUM_KP_MATS-i-1], (void*)&resultMat, (void*)&KP_MAT_N[NUM_KP_MATS-i-1], (void*)&KP_MAT_K[NUM_KP_MATS-i-1], &i};

    CUDACHECK(hipLaunchKernel((const void*)cuda_gemm_func, grid, block, &args[0], 0, stream));

    if (i < NUM_KP_MATS - 1) {
      prevResult = resultMat;
      if (resultMat == kpMatmulResult[0]) {        
        resultMat = kpMatmulResult[1];
      } else if (resultMat == kpMatmulResult[1]) {
        resultMat = kpMatmulResult[0];
      }
    }
    
    // CUDACHECK(hipDeviceSynchronize());
  }

  return resultMat;
}

bool check(int* ref, int* computed, int M, int N) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      if (ref[i*N + j] != computed[i* N + j]) {
        printf("Mismatch for %d x %d at (%d, %d): ref = %d, computed = %d\n", M, N, i, j, ref[i*N+j], computed[i*N+j]);
        return false;
      }
    }
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int one(int i, int j) {return 1;}
int zeroOne(int i, int j) {return i % 2;}
int setToI(int i, int j) {return i;}
int randMod(int i, int j) {return rand()%5 + 1;}

template<typename T>
void setValues(int NUM_KP_MATS, T* kpMats[], T *x, int M, int N, int K, int KP_MAT_N[], int KP_MAT_K[], int (*fnvalue)(int i, int j))
{
  for (int i = 0; i < NUM_KP_MATS; i++) {
    setMatrix(kpMats[i], KP_MAT_K[i], KP_MAT_N[i], fnvalue);
  }

  setMatrix(x, M, K, fnvalue);
}

struct MatrixSizes {
  const int M, N, K;
  const int NUM_KP_MATS;
  const std::vector<int> KP_MAT_N; 
  const std::vector<int> KP_MAT_K;
};

int main(int argc, char* argv[]) 
{

  #ifdef EVAL
  if (argc < 4) {printf("invalid command args\n"); return 0;}
  int npoints = atoi(argv[1]);
  int d = atoi(argv[2]);
  int twoPowerL = atoi(argv[3]);
  #endif

  std::vector<MatrixSizes> matrixSizes = {
                                          // {4,4,4, 2, {2,2},{2,2}},
                                          // {4,4,6, 2, {1,4},{2,3}},
                                          // {4,4,8, 2, {2,2},{2,4}},
                                          // {4,4,8, 2, {2,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{4,2}},
                                          // {8,8,8, 2, {4,2},{2,4}},
                                          // {8,8,8, 3, {2,2,2},{2,2,2}},
                                          // {8,8,32, 3, {2,2,2},{2,4,4}},
                                          // {8,16,32, 3, {4,2,2},{2,4,4}},
                                          // {8,8,16, 3, {2,2,2},{2,4,2}},
                                          // {16,8,8, 3, {2,2,2},{2,2,2}},
                                          // {16,16,16, 2, {4,4},{4,4}},
                                          // {16,16,16, 3, {4,2,2},{4,2,2}},
                                          // {16,16,16, 3, {4,2,2},{2,4,2}},
                                          // {16,16,16, 3, {8,2,1},{2,4,2}},
                                          // {16,16,16, 4, {2,2,2,2},{2,2,2,2}},
                                          // {16,16,64, 4, {2,2,2,2},{2,4,2,4}},
                                          // {256,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {256,256,256, 2, {16,16},{16,16}},
  #ifdef EVAL
                                          // {65536,1024,1024, 2, {32,32},{32,32}},
                                          // {65536,256,256, 2, {16,16},{16,16}},
                                          // {65536,512,512, 3, {8,8,8},{8,8,8}},
                                          // {100,1024,1024, 2, {32,32},{32,32}},
                                          // {10,1024,1024, 2, {32,32},{32,32}},
                                          // {1,1024,1024, 2, {32,32},{32,32}},
                                          // {100,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {10,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {1,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          // {100,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {1,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          
                                          // {100,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {10,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {1,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},

                                          // {100,1024,1024, 3, {16,16,4},{16,16,4}},
                                          // {100,256,256, 2, {16,16},{16,16}},
                                          // {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          // {1,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},

                                          // {100,512*8*8,512*8*8, 5, {8,8,8,8,8},{8,8,8,8,8}},

                                          // {100,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {10,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {1,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          // {1024,32*1024,32*1024, 2, {32,32,32},{32,32,32}},
  #else
                                          // {10,1024,1024, 10, {2,2,2,2,2,2,2,2,2,2},{2,2,2,2,2,2,2,2,2,2}},
                                          {1, 128*128, 128*128, 2, {128,128},{128,128}},
                                          {1, 4096, 4096, 2, {64,64},{64,64}},
                                          {10,1024,1024, 2, {32,32},{32,32}},                                        
                                          {10,256,256, 2, {16,16},{16,16}},
                                          // {10,256,256, 2, {16,16},{16,16}},
                                          {10,512,512, 3, {8,8,8},{8,8,8}},
                                          {10,256,256, 4, {4,4,4,4},{4,4,4,4}},
                                          {10,1024,1024, 5, {4,4,4,4,4},{4,4,4,4,4}},
                                          {4,4096,4096, 6, {4,4,4,4,4,4},{4,4,4,4,4,4}},
                                          // {1, 128*128, 128*128, 2, {128,128},{128,128}}
  #endif

                                          // {1024, 1024, 1024, 2, {32,32},{32,32}}
                                          };

  // int (*fnvalues[4])(int, int) = {&one, &zeroOne, &setToI, &randMod};
  int (*fnvalues[1])(int, int) = {&randMod};
  
  #ifdef EVAL
  int Msz = 1;
  for (int i = 0; i < d; i++) {
    Msz *= twoPowerL;
  }
  MatrixSizes matrixSize {
    npoints, Msz, Msz, d, std::vector<int>(d, twoPowerL), std::vector<int>(d, twoPowerL)
  };
  matrixSizes.push_back(matrixSize);
  #endif 

  for (MatrixSizes matrixSize : matrixSizes) {
    int M = matrixSize.M;
    int N = matrixSize.N;
    int K = matrixSize.K;
    
    int NUM_KP_MATS = matrixSize.NUM_KP_MATS;
    int KP_MAT_N[NUM_KP_MATS];
    int KP_MAT_K[NUM_KP_MATS];

    printf("Matmul: %d x %d x %d, Num KP Factors: %d\n", M, N, K, NUM_KP_MATS);
    int n=1,k=1;
    for (int i = 0; i < NUM_KP_MATS; i++) {
      k *= matrixSize.KP_MAT_K[i];
      n *= matrixSize.KP_MAT_N[i];
    }
    if (n != N || k != K) {
      printf("Invalid KP Factors Sizes %d != %d, %d != %d\n", n, N, k, K);
    }

    DATA_TYPE *kpout[NUM_KP_MATS];
    DATA_TYPE *kpMats[NUM_KP_MATS];
    DATA_TYPE* kpMatmulResult[NUM_KP_MATS];

    DATA_TYPE *x = new DATA_TYPE[M*K];

    DATA_TYPE* dX;
    DATA_TYPE** dKpOut;
    DATA_TYPE** dKpMats;
    DATA_TYPE** dKpMatmulResult;
    
    CUDACHECK(hipMalloc(&dX, M*K * sizeof(DATA_TYPE)));
    
    DATA_TYPE* __dKpOut[NUM_KP_MATS];
    DATA_TYPE* __dKpMats[NUM_KP_MATS];
    DATA_TYPE* __dKpMatmulResult[2];

    for (int i = 0; i < NUM_KP_MATS; i++) {
      KP_MAT_K[i] = matrixSize.KP_MAT_K[i];
      KP_MAT_N[i] = matrixSize.KP_MAT_N[i];
      kpMats[i] = new DATA_TYPE[KP_MAT_K[i] * KP_MAT_N[i]];
      kpout[i] = new DATA_TYPE[K*N]; //TODO: larger than needed
      kpMatmulResult[i] = new DATA_TYPE[M*std::max(N,K)];

      CUDACHECK(hipMalloc(&__dKpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(DATA_TYPE)));
      // CUDACHECK(hipMalloc(&__dKpOut[i], K * N * sizeof(int)));
      

      // CUDACHECK(hipMemset(__dKpOut[i], 0, K * N * sizeof(int)));
    }

    // CUDACHECK(hipMemcpy(&dKpOut[0], &__dKpOut[0], NUM_KP_MATS * sizeof(int*), hipMemcpyHostToDevice));
    // CUDACHECK(hipMemcpy(&dKpMats[0], &__dKpMats[0], NUM_KP_MATS * sizeof(DATA_TYPE*), hipMemcpyHostToDevice));
    CUDACHECK(hipMalloc(&__dKpMatmulResult[0], M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMalloc(&__dKpMatmulResult[1], M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMemset(__dKpMatmulResult[0], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));
    CUDACHECK(hipMemset(__dKpMatmulResult[1], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));

    DATA_TYPE* result = new DATA_TYPE[M*N];

    DATA_TYPE* dResult;

    CUDACHECK(hipMalloc(&dResult, M * N * sizeof(DATA_TYPE)));

    for (int fnvalue = 0; fnvalue < sizeof(fnvalues)/sizeof(fnvalues[0]); fnvalue++) {
      setValues(NUM_KP_MATS, kpMats, x, M, N, K, KP_MAT_N, KP_MAT_K, fnvalues[fnvalue]);

      for (int i = 0; i < NUM_KP_MATS; i++) {
        CUDACHECK(hipMemcpy(__dKpMats[i], kpMats[i], KP_MAT_K[i] * KP_MAT_N[i] * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
      }
    
      CUDACHECK(hipMemcpy(dX, x, M * K * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
  #ifndef EVAL
      baselineKPThenMatmul(NUM_KP_MATS, result, x, kpout, kpMats, 
                           M, N, K, KP_MAT_N, KP_MAT_K);
  #endif
      // slicedMatmul(NUM_KP_MATS, kpMatmulResult, x, kpMats,
      //              M, N, K, KP_MAT_N, KP_MAT_K);

      for (int i = 0; i < 2; i++)
        CUDACHECK(hipMemset(__dKpMatmulResult[i], 0, M*std::max(N,K) * sizeof(DATA_TYPE)));
  #ifdef EVAL  
      hipStream_t stream;
      hipStreamCreate(&stream);
      hipEvent_t start;
      hipEvent_t end;
      float elapsedTime = 0;
      CUDACHECK(hipEventCreate(&start));
      CUDACHECK(hipEventCreate(&end));
      for (int i = 0; i < 10; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, stream);
      CUDACHECK(hipStreamSynchronize(stream));
      CUDACHECK(hipEventRecord(start, stream));
      for (int i = 0; i < 1000; i++)
        customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, stream);
      CUDACHECK(hipEventRecord(end, stream));
      CUDACHECK(hipEventSynchronize(end));
      CUDACHECK(hipEventElapsedTime(&elapsedTime, start, end));
      printf("elapsedtime %f\n", elapsedTime/1000);

      for (int i = 0; i < NUM_KP_MATS; i++) {
        CUDACHECK(hipFree(__dKpMats[i]));
      }

      CUDACHECK(hipFree(__dKpMatmulResult[0]));
      CUDACHECK(hipFree(__dKpMatmulResult[1]));
      CUDACHECK(hipFree(dX));
      CUDACHECK(hipFree(dResult));
      continue;
  #else
      
      DATA_TYPE* dResult = customKronGEMM(NUM_KP_MATS, __dKpMatmulResult, dX, __dKpMats, M, N, K, KP_MAT_N, KP_MAT_K, 0);
      CUDACHECK(hipDeviceSynchronize());
  #endif
      // return;
      #ifndef EVAL 
      DATA_TYPE* hKpMatMulResult = new DATA_TYPE[M*N];
      // return;
      // for (int i = 0; i < NUM_KP_MATS; i++)
      //   CUDACHECK(hipMemcpy(kpMatmulResult[i], __dKpMatmulResult[i], M*N*sizeof(int), hipMemcpyDeviceToHost));
      CUDACHECK(hipMemcpy(kpMatmulResult[NUM_KP_MATS-1], dResult, M*N*sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      // if (check(result, kpMatmulResult[NUM_KP_MATS-1], M, N))
      if (check(result, kpMatmulResult[NUM_KP_MATS-1], M,N))
        printf("Results Correct for test %d\n", fnvalue);
      else {
        // printf("\nMatmul:");
        // printMatrix(result, K, N);

        // printf("\nx:");
        // printMatrix(x, M, K);    
        // for (int kpMatId = 0; kpMatId < NUM_KP_MATS; kpMatId++) {
        //   printf("\nKP Mat %d:", kpMatId);
        //   printMatrix(kpMats[kpMatId], KP_MAT_K[kpMatId], KP_MAT_N[kpMatId]);
        // }
        // // printf("\nKP Out:");
        // // printMatrix(kpout[0], 8, 8);
        // for (int id = 0; id < NUM_KP_MATS; id++) {
        //   printf("\nKP result %d:", id);
        //   printMatrix(kpMatmulResult[id], M, N);
        // }
        // printf("\nKP result 2:");
        // printMatrix(kpMatmulResult[2], 16, 16);
        // printf("\nKP result 3:");
        // printMatrix(kpMatmulResult[3], 16, 16);
        // printf("\nKP result 1:");
        // printMatrix(kpMatmulResult[1], M, N);
        // printf("\n");
        return 0;
      }
      #endif
    }

    //Is there really a need to free anything when you have tons of RAM, am I right?
  }

  return 0;
}
